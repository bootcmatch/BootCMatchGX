#include "hip/hip_runtime.h"
#include <assert.h>
#include <chrono>
#include <getopt.h>
#include <mpi.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <sys/time.h>
#include <unistd.h>

#include "datastruct/CSR.h"
#include "utility/assignDeviceToProcess.h"
#include "utility/distribuite.h"
#include "utility/globals.h"
#include "utility/handles.h"
#include "utility/mpi.h"

using namespace std;

// =============================================================================

#define USAGE                                                                                       \
    "\nUsage: %s --matrix <FILE_NAME> --out <FILE_NAME>\n\n"                                        \
    "\t-m, --matrix <FILE_NAME> Read the matrix from file <FILE_NAME>.\n"                           \
    "\t-l, --log <FILE_NAME> Write log files (one per MPI process) to <FILE_NAME>_<MPI_PID>.\n"     \
    "\t-o, --out <FILE_NAME> Write out files (one per MPI process) to <FILE_NAME>_<MPI_PID>.mtx.\n" \
    "\t-c, --col-shift Use column shift.\n"                                                         \
    "\t-f, --force-multiproc Force multiproc version even with 1 process.\n"                        \
    "\n"

// =============================================================================

int main(int argc, char** argv)
{
    char* mtx_file_name = NULL;
    char* log_file_name = NULL;
    char* out_file_name = NULL;
    bool use_column_shift = false;
    bool force_multiproc = false;
    int opt;

    static struct option long_options[] = {
        { "matrix", required_argument, NULL, 'm' },
        { "log", required_argument, NULL, 'l' },
        { "out", required_argument, NULL, 'o' },
        { "col-shift", no_argument, NULL, 'c' },
        { "force-multiproc", no_argument, NULL, 'f' },
        { "help", no_argument, NULL, 'h' },
    };

    while ((opt = getopt_long(argc, argv, "m:l:o:c:f:h", long_options, NULL)) != -1) {
        switch (opt) {
        case 'm':
            mtx_file_name = strdup(optarg);
            break;
        case 'l':
            log_file_name = strdup(optarg);
            break;
        case 'o':
            out_file_name = strdup(optarg);
            break;
        case 'c':
            use_column_shift = true;
            break;
        case 'f':
            force_multiproc = true;
            break;
        case 'h':
        default:
            printf(USAGE, argv[0]);
            exit(EXIT_FAILURE);
        }
    }

    if (mtx_file_name == NULL || out_file_name == NULL) {
        printf(USAGE, argv[0]);
        exit(EXIT_FAILURE);
    }

    // Start MPI
    int myid, nprocs, device_id;
    StartMpi(&myid, &nprocs, &argc, &argv);

    // Set device
    int deviceCount = 0;
    CHECK_DEVICE(hipGetDeviceCount(&deviceCount));
    assert(deviceCount);
    device_id = assignDeviceToProcess();
    int assigned_device_id = device_id % deviceCount;
    fprintf(stderr, "Trying to set device %d. Total devices: %d. Assigned device: %d\n", device_id, deviceCount, assigned_device_id);
    CHECK_DEVICE(hipSetDevice(assigned_device_id));

    handles* h = Handles::init();

    CSR* hmA = NULL; // Host master A
    if (ISMASTER) {
        fprintf(stderr, "Read matrix: %s\n", mtx_file_name);
        hmA = read_matrix_from_file(mtx_file_name, 0, false);
        // check_and_fix_order(Alocal_master);
    }

    // Device local A
    CSR* dlA = split_matrix_mpi(hmA);
    // TODO check for errors

    if (ISMASTER) {
        CSRm::free(hmA);
    }

    if (use_column_shift && dlA->row_shift) {
        CSRm::shift_cols(dlA, -dlA->row_shift);
        dlA->col_shifted = -dlA->row_shift;
    }

    if (log_file_name) {
        char filename[255] = { 0 };
        sprintf(filename, "%s_%d", log_file_name, myid);
        log_file = fopen(filename, "w");
        if (log_file == NULL) {
            fprintf(stderr, "Error opening file <%s>\n", filename);
            exit(EXIT_FAILURE);
        }
        if (atexit(close_log_file)) {
            fprintf(stderr, "Error registering atexit\n");
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "Transposing matrix... (force_multiproc: " << force_multiproc << ")\n";
    auto t1 = std::chrono::high_resolution_clock::now();
    CSR* dlAt = (nprocs == 1 && !force_multiproc) ? CSRm::Transpose_local(dlA, log_file) : CSRm::transpose(dlA, log_file);
    auto t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> ms_double = t2 - t1;
    std::cout << "Operation took " << ms_double.count() << " ms.\n";
    // printf("dlAt->col_shifted: %d\n", dlAt->col_shifted);

    if (log_file) {
        CSRm::print(dlAt, 3, 0, log_file);
    }

    CSRm::printMM(dlAt, out_file_name);

    CSRm::free(dlA);
    CSRm::free(dlAt);

    if (log_file) {
        fflush(log_file);
        fclose(log_file);
        log_file = NULL;
    }

    Handles::free(h);
    MPI_Finalize();
    return 0;
}
