#include "basic_kernel/halo_communication/halo_communication.h"
#include "custom_cudamalloc/custom_cudamalloc.h"
#include "datastruct/CSR.h"
#include "datastruct/scalar.h"
#include "datastruct/vector.h"
#include "generator/laplacian.h"
#include "gpoweru/GPowerU.hpp"
#include "op/spspmpi.h"
#include "utility/assignDeviceToProcess.h"
#include "utility/distribuite.h"
#include "utility/globals.h"
#include "utility/handles.h"
#include "utility/memoryPools.h"
#include "utility/mpi.h"
#include "utility/utils.h"

#include <assert.h>
#include <getopt.h>
#include <mpi.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <unistd.h>

#define GLOB_MEM_ALLOC_SIZE 2000000

using namespace std;

#define USAGE                                                                                                                                                   \
    "Usage: %s [--matrix <FILE_NAME> | --laplacian <SIZE> | --laplacian-3d <FILE_NAME>] [--time] [--energy] [--processes-per-node <N_PROCS>]\n\n"              \
    "\tYou can specify only one out of the three available options: --matrix, --laplacian-3d and --laplacian.\n\n"                                              \
    "\t-m, --matrix <FILE_NAME>                    Read the matrix from file <FILE_NAME>. Please note that this option works only in a mono-process setting.\n" \
    "\t-l, --laplacian-3d <FILE_NAME>              Read generation parameters from file <FILE_NAME>.\n"                                                         \
    "\t-g, --laplacian-3d-generator [ 7p | 27p ]   Choose laplacian 3d generator (7 points or 27 points).\n"                                                    \
    "\t-a, --laplacian <SIZE>                      Generate a matrix whose size is <SIZE>^3.\n"                                                                 \
    "\t-n, --processes-per-node <N_PROCS>          Number of MPI processes per node  \n"                                                                        \
    "\t-t, --time                                  Output the execution time of the application\n"                                                              \
    "\t-e, --energy                                If set measure the energy consumption\n\n"

extern vtype* d_temp_storage_max_min;
extern vtype* min_max;

enum generator_t {
    LAP_7P,
    LAP_27P,
    INVALIG_GEN
};

generator_t get_generator(const char* str)
{
    if (!strcmp(str, "7p")) {
        return LAP_7P;
    } else if (!strcmp(str, "27p")) {
        return LAP_27P;
    } else {
        return INVALIG_GEN;
    }
}

CSR* read_local_matrix_from_mtx(const char* mtx_file)
{
    _MPI_ENV;

    CSR* Alocal_master = NULL;
    if (ISMASTER) {
        Alocal_master = read_matrix_from_file(mtx_file, 0, false);
        check_and_fix_order(Alocal_master);
    }

    taskmap = (int*)Malloc(nprocs * sizeof(*taskmap));
    if (taskmap == NULL) {
        fprintf(stderr, "Could not get %d byte for taskmap\n", nprocs * sizeof(*taskmap));
        exit(1);
    }

    itaskmap = (int*)Malloc(nprocs * sizeof(*itaskmap));
    if (itaskmap == NULL) {
        fprintf(stderr, "Could not get %d byte for itaskmap\n", nprocs * sizeof(*itaskmap));
        exit(1);
    }

    for (int i = 0; i < nprocs; i++) {
        taskmap[i] = i;
        itaskmap[i] = i;
    }

    CSR* Alocal = split_matrix_mpi(Alocal_master);
    if (ISMASTER) {
        CSRm::free(Alocal_master);
    }

    snprintf(idstring, sizeof(idstring), "1_1_1");
    CSRm::shift_cols(Alocal, -Alocal->row_shift);
    Alocal->col_shifted = -Alocal->row_shift;

    return Alocal;
}

CSR* generate_lap_local_matrix(itype n)
{
    CSR* Alocal_host = generateLocalLaplacian3D(n);
    check_and_fix_order(Alocal_host);
    CSR* Alocal = CSRm::copyToDevice(Alocal_host);
    Alocal->col_shifted = -Alocal->row_shift;
    CSRm::free(Alocal_host);
    return Alocal;
}

CSR* generate_lap3d_local_matrix(generator_t generator, const char* lap_3d_file)
{
    _MPI_ENV;
    enum lap_params { nx = 0,
        ny = 1,
        nz = 2,
        P = 3,
        Q = 4,
        R = 5 };
    int* parms = read_laplacian_file(lap_3d_file);
    if (nprocs != (parms[P] * parms[Q] * parms[R])) {
        fprintf(stderr, "Nproc must be equal to P*Q*R\n");
        exit(EXIT_FAILURE);
    }
    CSR* Alocal_host = Alocal_host = NULL;
    switch (generator) {
    case LAP_7P:
        fprintf(stderr, "Using laplacian 3d 7 points generator.\n");
        Alocal_host = generateLocalLaplacian3D_7p(parms[nx], parms[ny], parms[nz], parms[P], parms[Q], parms[R]);
        break;
    case LAP_27P:
        fprintf(stderr, "Using laplacian 3d 27 points generator.\n");
        Alocal_host = generateLocalLaplacian3D_27p(parms[nx], parms[ny], parms[nz], parms[P], parms[Q], parms[R]);
        break;
    default:
        printf("Invalid generator\n");
        exit(1);
    }
    snprintf(idstring, sizeof(idstring), "%dx%dx%d", parms[P], parms[Q], parms[R]);
    free(parms);
    check_and_fix_order(Alocal_host);
    CSR* Alocal = CSRm::copyToDevice(Alocal_host);
    Alocal->col_shifted = -Alocal->row_shift;
    CSRm::free(Alocal_host);
    return Alocal;
}

int main(int argc, char** argv)
{
    enum opts { MTX,
        LAP_3D,
        LAP,
        NONE } opt
        = NONE;
    char* mtx_file = NULL;
    char* lap_3d_file = NULL;
    signed char ch;
    itype n = 0;
    int time = 0;
    int energy = 0;
    generator_t generator = LAP_27P;
    FILE* fp_time;
    int procs_per_node = 1;
    struct timeval matrix_t, setup_t, start_kernel_t, stop_kernel_t, teardown_t;
    int mem_alloc_size = GLOB_MEM_ALLOC_SIZE;

    static struct option long_options[] = {
        { "matrix", required_argument, NULL, 'm' },
        { "laplacian-3d", required_argument, NULL, 'l' },
        { "laplacian-3d-generator", required_argument, NULL, 'g' },
        { "laplacian", required_argument, NULL, 'a' },
        { "processes-per-node", required_argument, NULL, 'n' },
        { "time", no_argument, NULL, 't' },
        { "energy", no_argument, NULL, 'e' },
        { "help", no_argument, NULL, 'h' },
        { NULL, 0, NULL, 0 }
    };

    while ((ch = getopt_long(argc, argv, "n:m:l:g:a:h:e:t", long_options, NULL)) != -1) {
        switch (ch) {
        case 'n':
            procs_per_node = atoi(optarg);
            break;
        case 't':
            time = 1;
            break;
        case 'e':
            energy = 1;
            break;
        case 'm':
            mtx_file = strdup(optarg);
            opt = MTX;
            break;
        case 'l':
            lap_3d_file = strdup(optarg);
            opt = LAP_3D;
            break;
        case 'g':
            generator = get_generator(optarg);
            break;
        case 'a':
            n = atoi(optarg);
            opt = LAP;
            break;
        case 'h':
        default:
            printf(USAGE, argv[0]);
            exit(EXIT_FAILURE);
        }
    }
    if (opt == NONE) {
        printf(USAGE, argv[0]);
        exit(EXIT_FAILURE);
    }

    int myid, nprocs, device_id;
    StartMpi(&myid, &nprocs, &argc, &argv);

    if (getenv("SCALENNZMISSING")) {
        scalennzmiss = atoi(getenv("SCALENNZMISSING"));
    }

    // SetDevice
    int deviceCount = 0;
    CHECK_DEVICE(hipGetDeviceCount(&deviceCount));
    assert(deviceCount);
    device_id = assignDeviceToProcess();
    int assigned_device_id = device_id % deviceCount;
    fprintf(stderr, "Trying to set device %d. Total devices: %d. Assigned device: %d\n", device_id, deviceCount, assigned_device_id);
    CHECK_DEVICE(hipSetDevice(assigned_device_id));

    handles* h = Handles::init();

    // *******************
    // ENERGY - gpoweru start
    // *******************
    if (energy == 1 && myid % procs_per_node == 0) {
        printf("\n####### Process %d, running GPowerU #######\n", myid);
        if (GPowerU_init() != 0) {
            fprintf(stderr, "%s: error: initializing...\n", argv[0]);
            exit(EXIT_FAILURE);
        }
    }

    // *********************
    // ENERGY - gen matrix
    // *********************
    if (energy == 1) {
        gettimeofday(&matrix_t, NULL);
    }

    CSR* Alocal = NULL;
    CSR* Plocal = NULL;
    if (opt == MTX) { // The master reads the matrix and distributes it.
        Alocal = read_local_matrix_from_mtx(mtx_file);
        Plocal = read_local_matrix_from_mtx(mtx_file);
    } else if (opt == LAP_3D) {
        Alocal = generate_lap3d_local_matrix(generator, lap_3d_file);
        Plocal = generate_lap3d_local_matrix(generator, lap_3d_file);
    } else if (opt == LAP) {
        Alocal = generate_lap_local_matrix(n);
        Plocal = generate_lap_local_matrix(n);
    }

    // *********************
    // ENERGY - setup
    // *********************
    if (energy == 1) {
        gettimeofday(&setup_t, NULL);
    }

    // init memory pool
    MemoryPool::initContext(Alocal->full_n, Alocal->n);
    iPtemp1 = NULL;
    vPtemp1 = NULL;
    MY_CUDA_CHECK(hipHostMalloc(&iAtemp1, sizeof(itype) * mem_alloc_size));
    MY_CUDA_CHECK(hipHostMalloc(&vAtemp1, sizeof(vtype) * mem_alloc_size));
    MY_CUDA_CHECK(hipMalloc(&idevtemp1, sizeof(itype) * mem_alloc_size));
    MY_CUDA_CHECK(hipMalloc(&vdevtemp1, sizeof(vtype) * mem_alloc_size));
    MY_CUDA_CHECK(hipMalloc(&idevtemp2, sizeof(itype) * mem_alloc_size));

    CustomCudaMalloc::init((Alocal->nnz) * 8, (Alocal->nnz) * 4);
    CustomCudaMalloc::init((Alocal->nnz) * 2, (Alocal->nnz) * 2, 1);
    CustomCudaMalloc::init((Alocal->nnz) * 4, (Alocal->nnz) * 4, 2);

    vector<int>* _bitcol = NULL;
    _bitcol = get_missing_col(Alocal, NULL);
    compute_rows_to_rcv_CPU(Alocal, NULL, _bitcol);
    Vector::free(_bitcol);

    // *********************
    // ENERGY - spmm start
    // *********************
    if (energy == 1) {
        gettimeofday(&start_kernel_t, NULL);
    }
    // *********************
    // TIME
    // *********************
    double TOT_TIMEM;
    if (time == 1 && ISMASTER) {
        TOT_TIMEM = MPI_Wtime();
    }

    CSR* APlocal = nsparseMGPU_commu_new(h, Alocal, Plocal, false);
    hipDeviceSynchronize();

    // *********************
    // TIME
    // *********************
    if (time == 1 && ISMASTER) {
        printf("TOTAL_TIME: %f\n", (MPI_Wtime() - TOT_TIMEM));
    }
    // *********************
    // ENERGY - spmm stop
    // *********************
    if (energy == 1) {
        gettimeofday(&stop_kernel_t, NULL);
    }

    CSRm::free(Alocal);
    CSRm::free(Plocal);
    CSRm::free(APlocal);

    if (xsize > 0) {
        hipFree(xvalstat);
    }
    CustomCudaMalloc::free(1);

    if (d_temp_storage_max_min) {
        MY_CUDA_CHECK(hipFree(d_temp_storage_max_min));
    }
    if (min_max) {
        MY_CUDA_CHECK(hipFree(min_max));
    }

    // *********************
    // ENERGY - teardown
    // *********************
    if (energy == 1) {
        gettimeofday(&teardown_t, NULL);
    }
    // *********************
    // ENERGY - write files
    // *********************
    if (energy == 1) {
        char filename[256];
        snprintf(filename, sizeof(filename), "data/spmm_%d_of_%d.time", myid, nprocs);
        mkdir("data", 0777);
        fp_time = fopen(filename, "w");

        fprintf(fp_time, "gen matrix;%ld;%ld\n", matrix_t.tv_sec, matrix_t.tv_usec);
        fprintf(fp_time, "setup;%ld;%ld\n", setup_t.tv_sec, setup_t.tv_usec);
        fprintf(fp_time, "start kernel;%ld;%ld\n", start_kernel_t.tv_sec, start_kernel_t.tv_usec);
        fprintf(fp_time, "stop kernel;%ld;%ld\n", stop_kernel_t.tv_sec, stop_kernel_t.tv_usec);
        fprintf(fp_time, "teardown;%ld;%ld\n", teardown_t.tv_sec, teardown_t.tv_usec);

        fclose(fp_time);
    }
    // *******************
    // ENERGY - gpoweru shutdown
    // *******************
    if (energy == 1) {
        if (myid % procs_per_node == 0) {
            if (GPowerU_end(5) != 0) {
                fprintf(stderr, " error: terminating...\n");
                _exit(1);
            }
        } else {
            sleep(5);
        }
    }

    MPI_Finalize();
    return 0;
}
