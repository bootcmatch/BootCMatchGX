#include "hip/hip_runtime.h"
#include "triple_inner_product.h"

#include "utility/cudamacro.h"
#include "utility/function_cnt.h"
#include "utility/metrics.h"
#include "utility/timing.h"

__global__ void _triple_innerproduct(itype n, vtype* r, vtype* w, vtype* q, vtype* v, vtype* alpha_beta_gamma, itype shift)
{
    __shared__ vtype alpha_shared[FULL_WARP];
    __shared__ vtype beta_shared[FULL_WARP];
    __shared__ vtype gamma_shared[FULL_WARP];

    itype tid = blockDim.x * blockIdx.x + threadIdx.x;
    int warp = threadIdx.x / FULL_WARP;
    int lane = tid % FULL_WARP;
    int i = tid;

    if (threadIdx.x < FULL_WARP) {
        alpha_shared[threadIdx.x] = 0.;
        beta_shared[threadIdx.x] = 0.;
        gamma_shared[threadIdx.x] = 0.;
    }
    __syncthreads();
    if (i >= n) {
        return;
    }

    vtype v_i = v[i + shift];
    vtype alpha_i = r[i] * v_i;
    vtype beta_i = w[i] * v_i;
    vtype gamma_i = q[i] * v_i;

#pragma unroll
    for (int k = FULL_WARP >> 1; k > 0; k = k >> 1) {
        alpha_i += __shfl_down_sync(FULL_MASK, alpha_i, k);
        beta_i += __shfl_down_sync(FULL_MASK, beta_i, k);
        gamma_i += __shfl_down_sync(FULL_MASK, gamma_i, k);
    }

    if (lane == 0) {
        alpha_shared[warp] = alpha_i;
        beta_shared[warp] = beta_i;
        gamma_shared[warp] = gamma_i;
    }

    __syncthreads();

    if (warp == 0) {
#pragma unroll
        for (int k = FULL_WARP >> 1; k > 0; k = k >> 1) {
            alpha_shared[lane] += __shfl_down_sync(FULL_MASK, alpha_shared[lane], k);
            beta_shared[lane] += __shfl_down_sync(FULL_MASK, beta_shared[lane], k);
            gamma_shared[lane] += __shfl_down_sync(FULL_MASK, gamma_shared[lane], k);
        }

        if (lane == 0) {
            atomicAdd(&alpha_beta_gamma[0], alpha_shared[0]);
            atomicAdd(&alpha_beta_gamma[1], beta_shared[0]);
            atomicAdd(&alpha_beta_gamma[2], gamma_shared[0]);
        }
    }
}

void triple_innerproduct(vector<vtype>* r, vector<vtype>* w, vector<vtype>* q, vector<vtype>* v, vtype* alpha, vtype* beta, vtype* gamma, itype shift)
{
    PUSH_RANGE(__func__, 4)

    _MPI_ENV;

    assert(r->n == w->n && w->n == q->n);

#if DETAILED_TIMING
    if (ISMASTER) {
        TIME::start();
    }
#endif

    Vectorinit_CNT
        vector<vtype>* alpha_beta_gamma
        = Vector::init<vtype>(3, true, true);
    Vector::fillWithValue(alpha_beta_gamma, 0.);

    GridBlock gb = gb1d(r->n, BLOCKSIZE);

    _triple_innerproduct<<<gb.g, gb.b>>>(r->n, r->val, w->val, q->val, v->val, alpha_beta_gamma->val, shift);

    vector<vtype>* alpha_beta_gamma_host = Vector::copyToHost(alpha_beta_gamma);

#if DETAILED_TIMING
    if (ISMASTER) {
        hipDeviceSynchronize();
        TOTAL_TRIPLEPROD_TIME += TIME::stop();
    }
#endif

    vtype abg[3];

#if DETAILED_TIMING
    if (ISMASTER) {
        TIME::start();
    }
#endif

    CHECK_MPI(MPI_Allreduce(
        alpha_beta_gamma_host->val,
        abg,
        3,
        MPI_DOUBLE,
        MPI_SUM,
        MPI_COMM_WORLD));

#if DETAILED_TIMING
    if (ISMASTER) {
        hipDeviceSynchronize();
        TOTAL_ALLREDUCE_TIME += TIME::stop();
    }
#endif

    *alpha = abg[0];
    *beta = abg[1];
    *gamma = abg[2];

    Vector::free(alpha_beta_gamma);
    POP_RANGE
}
