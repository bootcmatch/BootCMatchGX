#include "hip/hip_runtime.h"

#include "AMG.h"

char linebuffer[BUFSIZE+1];

int  int_get_fp(FILE *fp){
  int temp;
  char *token;
  void *out = fgets(linebuffer,BUFSIZE,fp);
  if(out == NULL){
    printf("ERROR: reading conf\n");
    exit(1);
  }
  token = strtok(linebuffer,DELIM);
  sscanf(token,"%d",&temp);
  return(temp);
}
double double_get_fp(FILE *fp){
  double temp;
  char *token;
  void *out = fgets(linebuffer,BUFSIZE,fp);
  if(out == NULL){
    printf("ERROR: reading conf\n");
    exit(1);
  }
  token = strtok(linebuffer,DELIM);
  sscanf(token,"%lf",&temp);
  return(temp);
}
char* string_get_fp(FILE *fp){
  char *token1, *token2;
  void *out =  fgets(linebuffer,BUFSIZE,fp);
  if(out == NULL){
    printf("ERROR: reading conf\n");
    exit(1);
  }
  token1 = strtok(linebuffer,DELIM);
  token2 = strtok(token1," ");
  return(strdup(token2));
}

namespace AMG{

  namespace Hierarchy{

    hierarchy* init(itype num_levels, bool allocate_mem){
      hierarchy *H = NULL;
      // on the host
      H = (hierarchy*) malloc(sizeof(hierarchy));
      CHECK_HOST(H);

      H->num_levels = num_levels;
      H->op_cmplx = 0;

      H->A_array = NULL;
      H->P_array = NULL;
      H->R_array = NULL;
      H->R_local_array = NULL;
      H->P_local_array = NULL;

      if(allocate_mem){
        H->A_array = (CSR**) malloc(num_levels * sizeof(CSR*));
        CHECK_HOST(H->A_array);

        H->P_array = (CSR**) malloc( (num_levels-1) * sizeof(CSR*) );
        CHECK_HOST(H->P_array);

        H->R_array = (CSR**) malloc( (num_levels-1) * sizeof(CSR*) );
        CHECK_HOST(H->R_array);

        H->R_local_array = (CSR**) malloc( (num_levels-1) * sizeof(CSR*) );
        CHECK_HOST(H->R_local_array);

        H->P_local_array = (CSR**) malloc( (num_levels-1) * sizeof(CSR*) );
        CHECK_HOST(H->P_local_array);

        H->D_array = (vector<vtype>**) malloc( num_levels * sizeof(vector<vtype>*));
        CHECK_HOST(H->D_array);

        H->M_array = (vector<vtype>**) malloc( num_levels * sizeof(vector<vtype>*));
        CHECK_HOST(H->M_array);

        for(int i=0; i<H->num_levels; i++){
          H->D_array[i] = NULL;
          if(i != H->num_levels - 1){
            H->R_array[i] = NULL;
            H->R_local_array[i] = NULL;
            H->P_local_array[i] = NULL;

          }
          H->M_array[i] = NULL;
        }
      }

      return H;
    }

    void free(hierarchy *H){

      for(int i=0; i<H->num_levels; i++){

        // skip the original matrix
        if(i > 0)
          CSRm::free(H->A_array[i]);

        if( H->D_array[i] != NULL )
          Vector::free(H->D_array[i]);

        if( H->M_array[i] != NULL )
          Vector::free(H->M_array[i]);


        if(i != H->num_levels - 1){
            CSRm::free(H->P_array[i]);
            if( H->R_array[i] != NULL )
              CSRm::free(H->R_array[i]);
            if( H->R_local_array[i] != NULL )
                CSRm::free(H->R_local_array[i]);
            if( H->P_local_array[i] != NULL )
                CSRm::free(H->P_local_array[i]);
        }
      }

      std::free(H->A_array);
      std::free(H->D_array);
      std::free(H->M_array);
      std::free(H->P_array);

      std::free(H);
    }

    void finalize_level(hierarchy *H, int levels_used){

      assert(levels_used > 0);

      H->num_levels = levels_used;

      H->A_array = (CSR**) realloc(H->A_array, levels_used * sizeof(CSR*));
      CHECK_HOST(H->A_array);

      H->P_array = (CSR**) realloc(H->P_array, (levels_used-1) * sizeof(CSR*) );
      CHECK_HOST(H->P_array);

      H->R_array = (CSR**) realloc(H->R_array, (levels_used-1) * sizeof(CSR*) );
      CHECK_HOST(H->R_array);

      H->R_local_array = (CSR**) realloc(H->R_local_array, (levels_used-1) * sizeof(CSR*) );
      CHECK_HOST(H->R_local_array);

      H->P_local_array = (CSR**) realloc(H->P_local_array, (levels_used-1) * sizeof(CSR*) );
      CHECK_HOST(H->P_local_array);

      H->D_array = (vector<vtype>**) realloc(H->D_array, levels_used * sizeof(vector<vtype>*));
      CHECK_HOST(H->D_array);

      H->M_array = (vector<vtype>**) realloc(H->M_array, levels_used * sizeof(vector<vtype>*));
      CHECK_HOST(H->M_array);

    }

    itype getNNZglobal(CSR *A){
      itype nnzp = 0;
      CHECK_MPI(
        MPI_Allreduce(
          &A->nnz,
          &nnzp,
          1,
          MPI_INT,
          MPI_SUM,
          MPI_COMM_WORLD
        )
      );
      return nnzp;
    }

    vtype finalize_cmplx(hierarchy *h){
      vtype cmplxfinal = 0;

      for(int i=0; i<h->num_levels; i++){
        cmplxfinal += getNNZglobal(h->A_array[i]);
      }
      cmplxfinal /= getNNZglobal(h->A_array[0]);
      h->op_cmplx = cmplxfinal;
      return cmplxfinal;
    }

    vtype finalize_wcmplx(hierarchy *h){
      vtype wcmplxfinal = 0;
      for(int i=0; i<h->num_levels; i++){
        wcmplxfinal +=  pow(2, i) * getNNZglobal(h->A_array[i]);
      }
      wcmplxfinal /= getNNZglobal(h->A_array[0]);
      h->op_wcmplx = wcmplxfinal;
      return wcmplxfinal;
    }

    void printInfo(hierarchy *h){

      for(int i=0; i<h->num_levels; i++){
        CSR *Ai = h->A_array[i];
        float avg_nnz = (float) Ai->nnz / (float) Ai->n;
        std::cout << "A" << i << " n: " << Ai->full_n << " nnz: " << Ai->nnz << " avg_nnz: " << avg_nnz << "\n";
      }
      std::cout << "\nCurrent cmplx for V-cycle: " << h->op_cmplx;
      std::cout << "\nCurrent cmplx for W-cycle: " << h->op_wcmplx;
      std::cout << "\nAverage Coarsening Ratio: " << h->avg_cratio << "\n";
    }
  }

  namespace BuildData{
    buildData* init(itype maxlevels, itype maxcoarsesize, itype sweepnumber, itype agg_interp_type, itype coarse_solver, itype CRrelax_type, vtype CRrelax_weight, itype CRit, vtype CRratio){
      buildData *bd = NULL;

      bd = (buildData*) malloc(sizeof(buildData));
      CHECK_HOST(bd);

      bd->maxlevels = maxlevels;
      bd->maxcoarsesize = maxcoarsesize;
      bd->sweepnumber = sweepnumber;
      bd->agg_interp_type = agg_interp_type;
      bd->coarse_solver = coarse_solver;
      bd->CRrelax_type = CRrelax_type;
      bd->CRrelax_weight = CRrelax_weight;
      bd->CRit = CRit;
      bd->CRratio = CRratio;

      bd->A = NULL;
      bd->w = NULL;

      bd->ftcoarse = 1;

      return bd;
    }

    void free(buildData *bd){
      std::free(bd);
    }

    buildData* initDefault(){
      buildData *bd = NULL;

      bd = (buildData*) malloc(sizeof(buildData));
      CHECK_HOST(bd);

      bd->maxlevels = 100;
      bd->maxcoarsesize = 100;
      bd->sweepnumber = 1;
      bd->agg_interp_type = 0;
      bd->coarse_solver = 9;

      bd->CRrelax_type = 0;
      bd->CRrelax_weight = 1. / 3.;
      bd->CRit = 0;
      bd->CRratio = .3;

      bd->A = NULL;
      bd->w = NULL;

      bd->ftcoarse = 1;

      return bd;
    }

    void setMaxCoarseSize(buildData *bd){
      bd->maxcoarsesize = (itype) ( 40 * pow( (double) bd->A->full_n, (double)1. / 3.) );
    }

    void print(buildData *bd){
      std::cout << "\nmaxlevels: " << bd->maxlevels << "\n";
      std::cout << "maxcoarsesize: " << bd->maxcoarsesize << "\n";
      std::cout << "sweepnumber: " << bd->sweepnumber << "\n";
      std::cout << "agg_interp_type: " << bd->agg_interp_type << "\n";
      std::cout << "coarse_solver: " << bd->coarse_solver << "\n";
      std::cout << "CRrelax_type: " << bd->CRrelax_type << "\n";
      std::cout << "CRrelax_weight: " << bd->CRrelax_weight << "\n";
      std::cout << "CRit: " << bd->CRit << "\n";
      std::cout << "CRratio: " << bd->CRratio << "\n";
      std::cout << "ftcoarse: " << bd->ftcoarse << "\n\n";
    }
  }

  namespace ApplyData{
    applyData* initDefault(){

      applyData *ad = NULL;
      ad = (applyData*) malloc(sizeof(applyData));
      CHECK_HOST(ad);

      ad->cycle_type = 0;
      ad->relax_type = 0;
      ad->relaxnumber_coarse = 1;
      ad->prerelax_number = 1;
      ad->postrelax_number = 1;
      ad->relax_weight = 1.0;
      ad->num_grid_sweeps = NULL;

      return ad;
    }

    void free(applyData *ad){
      std::free(ad->num_grid_sweeps);
      std::free(ad);
    }

    void print(applyData *ad){
      std::cout << "\ncycle_type: " << ad->cycle_type << "\n";
      std::cout << "relax_type: " << ad->relax_type << "\n";
      std::cout << "relaxnumber_coarse: " << ad->relaxnumber_coarse << "\n";
      std::cout << "prerelax_number: " << ad->prerelax_number << "\n";
      std::cout << "postrelax_number: " << ad->postrelax_number << "\n";
      std::cout << "relax_weight: " << ad->relax_weight << "\n\n";
    }

    applyData* initByParams(params p){
      applyData *amg_cycle = AMG::ApplyData::initDefault();

      amg_cycle->cycle_type = p.cycle_type;
      amg_cycle->relax_type = p.relax_type;
      amg_cycle->relaxnumber_coarse = p.relaxnumber_coarse;
      amg_cycle->prerelax_number = p.prerelax_sweeps;
      amg_cycle->postrelax_number = p.postrelax_sweeps;

      return amg_cycle;
    }

    void setGridSweeps(applyData *ad, int max_level){
      max_level--;
      ad->num_grid_sweeps = (int*) malloc( max_level * sizeof(int));
      CHECK_HOST(ad->num_grid_sweeps);

      int i, j;

      for(i=0; i<max_level; i++)
        ad->num_grid_sweeps[i] = 1;

      if(ad->cycle_type == 1){
        // H-cycle
        for(i=0; i<max_level; i++){
          j = i % 2; /*step is fixed to 2; it can be also different */
          if(j == 0)
            ad->num_grid_sweeps[i] = 2;
        }
      }else if(ad->cycle_type == 2){
        // W-cycle
        for(i=0; i<max_level-1; i++){
          ad->num_grid_sweeps[i] = 2;
        }
      }
    }
  }

  namespace BootBuildData{
    bootBuildData* initDefault(){

      bootBuildData *bd = NULL;
      bd = (bootBuildData*) malloc(sizeof(bootBuildData));
      CHECK_HOST(bd);

      bd->max_hrc = 10;
      bd->conv_ratio = 0.80;
      bd->solver_type = 1;
      bd->solver_it = 15;

      bd->amg_data = AMG::BuildData::initDefault();

      return bd;
    }

    void free(bootBuildData *ad){
      AMG::BuildData::free(ad->amg_data);
      std::free(ad);
    }

    void print(bootBuildData *ad){
      std::cout << "\nmax_hrc: " << ad->max_hrc << "\n";
      std::cout << "conv_ratio: " << ad->conv_ratio << "\n";
      std::cout << "solver_type: " << ad->solver_type << "\n";
      std::cout << "solver_it: " << ad->solver_it << "\n\n";
    }

    bootBuildData* initByParams(CSR *A, params p){
      bootBuildData *bootamg_data = AMG::BootBuildData::initDefault();
      buildData *amg_data = bootamg_data->amg_data;

      bootamg_data->solver_type = p.solver_type;
      bootamg_data->max_hrc = p.max_hrc;
      bootamg_data->conv_ratio = p.conv_ratio;

      amg_data->sweepnumber = p.aggrsweeps;
      amg_data->agg_interp_type = p.aggrtype;
      amg_data->maxlevels = p.max_levels;
      amg_data->coarse_solver = p.coarse_solver;
      amg_data->CRrelax_type = p.relax_type;


      amg_data->A = A;
      amg_data->w = Vector::init<vtype>(A->n, true, true);
      Vector::fillWithValue(amg_data->w, 1.0);

      AMG::BuildData::setMaxCoarseSize(amg_data);

      return bootamg_data;
    }
  }

  namespace Boot{

    boot* init(int n_hrc, double estimated_ratio){
      boot *b = (boot*) malloc(sizeof(boot));
      CHECK_HOST(b);

      b->n_hrc = n_hrc;
      b->estimated_ratio = estimated_ratio;
      b->H_array = (hierarchy**) malloc( n_hrc * sizeof(hierarchy*) );
      CHECK_HOST(b->H_array);

      return b;
    }

    void free(boot *b){
      for(int i=0; i<b->n_hrc; i++)
        AMG::Hierarchy::free(b->H_array[i]);
      std::free(b);
    }

    void finalize(boot *b, int num_hrc){
      assert(num_hrc > 0);
      b->n_hrc = num_hrc;
      b->H_array = (hierarchy**) realloc(b->H_array, num_hrc * sizeof(hierarchy*));
      CHECK_HOST(b->H_array);
    }
  }

  namespace Params{
    params initFromFile(const char *path){
      params inparms;

      FILE *fp = fopen(path, "r");

      if(fp == NULL){
        std::cout << "Setting file not found!" << "\n";
        exit(-1);
      }

      inparms.rhsfile           = string_get_fp(fp);
      if (strcmp(inparms.rhsfile,"NONE")==0) {
        inparms.rhsfile=NULL;
      }
      inparms.solfile           = string_get_fp(fp);
      if (strcmp(inparms.solfile,"NONE")==0) {
        inparms.solfile=NULL;
      }
      inparms.solver_type       = int_get_fp(fp);
      inparms.max_hrc           = int_get_fp(fp);
      inparms.conv_ratio        = double_get_fp(fp);
      inparms.matchtype         = int_get_fp(fp);
      inparms.aggrsweeps        = int_get_fp(fp) + 1;
      inparms.aggrtype          = int_get_fp(fp);
      inparms.max_levels        = int_get_fp(fp);
      inparms.cycle_type        = int_get_fp(fp);
      inparms.coarse_solver     = int_get_fp(fp);
      inparms.relax_type        = int_get_fp(fp);
      inparms.relaxnumber_coarse= int_get_fp(fp);
      inparms.prerelax_sweeps   = int_get_fp(fp);
      inparms.postrelax_sweeps  = int_get_fp(fp);
      inparms.itnlim            = int_get_fp(fp);
      inparms.rtol              = double_get_fp(fp);
      inparms.mem_alloc_size    = int_get_fp(fp);

      fclose(fp);
      return inparms;
    }

    /*
    void metaPrintInfo(params inparms){
      Eval::printMetaData("params;solver_type", inparms.solver_type, 0);
      Eval::printMetaData("params;max_hrc", inparms.max_hrc, 0);
      Eval::printMetaData("params;conv_ratio", inparms.conv_ratio, 1);
      Eval::printMetaData("params;matchtype", inparms.matchtype, 0);
      Eval::printMetaData("params;aggrsweeps", inparms.aggrsweeps - 1, 0);
      Eval::printMetaData("params;aggrtype", inparms.aggrtype, 0);
      Eval::printMetaData("params;max_levels", inparms.max_levels, 0);
      Eval::printMetaData("params;cycle_type", inparms.cycle_type, 0);
      Eval::printMetaData("params;coarse_solver", inparms.coarse_solver, 0);
      Eval::printMetaData("params;relax_type", inparms.relax_type, 0);
      Eval::printMetaData("params;relaxnumber_coarse", inparms.relaxnumber_coarse, 0);
      Eval::printMetaData("params;prerelax_sweeps", inparms.prerelax_sweeps, 0);
      Eval::printMetaData("params;postrelax_sweeps", inparms.postrelax_sweeps, 0);
      Eval::printMetaData("params;itnlim", inparms.itnlim, 0);
      Eval::printMetaData("params;rtol", inparms.rtol, 1);
    }
  */
  }
}
