#include "hip/hip_runtime.h"

#include "AMG.h"

#include "utility/cudamacro.h"
#include "utility/function_cnt.h"

namespace AMG {

namespace Hierarchy {

    hierarchy* init(itype num_levels, bool allocate_mem)
    {
        hierarchy* H = NULL;
        // on the host
        H = (hierarchy*)Malloc(sizeof(hierarchy));
        CHECK_HOST(H);

        H->num_levels = num_levels;
        H->op_cmplx = 0;

        H->A_array = NULL;
        H->P_array = NULL;
        H->R_array = NULL;
        H->R_local_array = NULL;
        H->P_local_array = NULL;

        if (allocate_mem) {
            H->A_array = (CSR**)Malloc(num_levels * sizeof(CSR*));
            CHECK_HOST(H->A_array);

            H->P_array = (CSR**)Malloc((num_levels - 1) * sizeof(CSR*));
            CHECK_HOST(H->P_array);

            H->R_array = (CSR**)Malloc((num_levels - 1) * sizeof(CSR*));
            CHECK_HOST(H->R_array);

            H->R_local_array = (CSR**)Malloc((num_levels - 1) * sizeof(CSR*));
            CHECK_HOST(H->R_local_array);

            H->P_local_array = (CSR**)Malloc((num_levels - 1) * sizeof(CSR*));
            CHECK_HOST(H->P_local_array);

            H->D_array = (vector<vtype>**)Malloc(num_levels * sizeof(vector<vtype>*));
            CHECK_HOST(H->D_array);

            H->M_array = (vector<vtype>**)Malloc(num_levels * sizeof(vector<vtype>*));
            CHECK_HOST(H->M_array);

            for (int i = 0; i < H->num_levels; i++) {
                H->D_array[i] = NULL;
                if (i != H->num_levels - 1) {
                    H->R_array[i] = NULL;
                    H->R_local_array[i] = NULL;
                    H->P_local_array[i] = NULL;
                }
                H->M_array[i] = NULL;
            }
        }

        return H;
    }

    void free(hierarchy* H)
    {

        for (int i = 0; i < H->num_levels; i++) {

            // skip the original matrix
            if (i > 0) {
                CSRm::free(H->A_array[i]);
            }

            if (H->D_array[i] != NULL) {
                Vector::free(H->D_array[i]);
            }

            if (H->M_array[i] != NULL) {
                Vector::free(H->M_array[i]);
            }

            if (i != H->num_levels - 1) {
                CSRm::free(H->P_array[i]);
                if (H->R_array[i] != NULL) {
                    CSRm::free(H->R_array[i]);
                }
                if (H->R_local_array[i] != NULL) {
                    CSRm::free(H->R_local_array[i]);
                }
                if (H->P_local_array[i] != NULL) {
                    CSRm::free(H->P_local_array[i]);
                }
            }
        }

        std::free(H->A_array);
        std::free(H->D_array);
        std::free(H->M_array);
        std::free(H->P_array);

        std::free(H);
    }

    void finalize_level(hierarchy* H, int levels_used)
    {

        assert(levels_used > 0);

        H->num_levels = levels_used;

        H->A_array = (CSR**)realloc(H->A_array, levels_used * sizeof(CSR*));
        CHECK_HOST(H->A_array);

        H->P_array = (CSR**)realloc(H->P_array, (levels_used - 1) * sizeof(CSR*));
        CHECK_HOST(H->P_array);

        H->R_array = (CSR**)realloc(H->R_array, (levels_used - 1) * sizeof(CSR*));
        CHECK_HOST(H->R_array);

        H->R_local_array = (CSR**)realloc(H->R_local_array, (levels_used - 1) * sizeof(CSR*));
        CHECK_HOST(H->R_local_array);

        H->P_local_array = (CSR**)realloc(H->P_local_array, (levels_used - 1) * sizeof(CSR*));
        CHECK_HOST(H->P_local_array);

        H->D_array = (vector<vtype>**)realloc(H->D_array, levels_used * sizeof(vector<vtype>*));
        CHECK_HOST(H->D_array);

        H->M_array = (vector<vtype>**)realloc(H->M_array, levels_used * sizeof(vector<vtype>*));
        CHECK_HOST(H->M_array);
    }

    long getNNZglobal(CSR* A)
    {
        PUSH_RANGE(__func__, 7)

        unsigned long nnzp = 0;
        unsigned long lnnz = A->nnz;
        CHECK_MPI(
            MPI_Allreduce(
                &lnnz,
                &nnzp,
                1,
                MPI_LONG,
                MPI_SUM,
                MPI_COMM_WORLD));

        POP_RANGE
        return nnzp;
    }

    vtype finalize_cmplx(hierarchy* h)
    {
        vtype cmplxfinal = 0;

        for (int i = 0; i < h->num_levels; i++) {
            cmplxfinal += getNNZglobal(h->A_array[i]);
        }
        cmplxfinal /= getNNZglobal(h->A_array[0]);
        h->op_cmplx = cmplxfinal;
        return cmplxfinal;
    }

    vtype finalize_wcmplx(hierarchy* h)
    {
        vtype wcmplxfinal = 0;
        for (int i = 0; i < h->num_levels; i++) {
            wcmplxfinal += pow(2, i) * getNNZglobal(h->A_array[i]);
        }
        wcmplxfinal /= getNNZglobal(h->A_array[0]);
        h->op_wcmplx = wcmplxfinal;
        return wcmplxfinal;
    }

    void printInfo(hierarchy* h)
    {

        for (int i = 0; i < h->num_levels; i++) {
            CSR* Ai = h->A_array[i];
            float avg_nnz = (float)Ai->nnz / (float)Ai->n;
            std::cout << "A" << i << " n: " << Ai->full_n << " nnz: " << Ai->nnz << " avg_nnz: " << avg_nnz << "\n";
        }
        std::cout << "\nCurrent cmplx for V-cycle: " << h->op_cmplx;
        std::cout << "\nCurrent cmplx for W-cycle: " << h->op_wcmplx;
        std::cout << "\nAverage Coarsening Ratio: " << h->avg_cratio << "\n";
    }
}

namespace BuildData {
    buildData* init(itype maxlevels, itype maxcoarsesize, itype sweepnumber, itype agg_interp_type, CoarseSolverType coarse_solver, RelaxType CRrelax_type, vtype CRrelax_weight, itype CRit, vtype CRratio)
    {
        buildData* bd = NULL;

        bd = (buildData*)Malloc(sizeof(buildData));
        CHECK_HOST(bd);

        bd->maxlevels = maxlevels;
        bd->maxcoarsesize = maxcoarsesize;
        bd->sweepnumber = sweepnumber;
        bd->agg_interp_type = agg_interp_type;
        bd->coarse_solver = coarse_solver;
        bd->CRrelax_type = CRrelax_type;
        bd->CRrelax_weight = CRrelax_weight;
        bd->CRit = CRit;
        bd->CRratio = CRratio;

        bd->A = NULL;
        bd->w = NULL;

        bd->ftcoarse = 1;

        return bd;
    }

    void free(buildData* bd)
    {
        std::free(bd);
    }

    buildData* initDefault()
    {
        buildData* bd = NULL;

        bd = (buildData*)Malloc(sizeof(buildData));
        CHECK_HOST(bd);

        bd->maxlevels = 100;
        bd->maxcoarsesize = 100;
        bd->sweepnumber = 1;
        bd->agg_interp_type = 0;
        bd->coarse_solver = CoarseSolverType::L1_JACOBI;

        bd->CRrelax_type = RelaxType::L1_JACOBI;
        bd->CRrelax_weight = 1. / 3.;
        bd->CRit = 0;
        bd->CRratio = .3;

        bd->A = NULL;
        bd->w = NULL;

        bd->ftcoarse = 1;

        return bd;
    }

    void setMaxCoarseSize(buildData* bd)
    {
        bd->maxcoarsesize = (itype)(40 * pow((double)bd->A->full_n, (double)1. / 3.));
    }

    void print(buildData* bd)
    {
        std::cout << "\nmaxlevels: " << bd->maxlevels << "\n";
        std::cout << "maxcoarsesize: " << bd->maxcoarsesize << "\n";
        std::cout << "sweepnumber: " << bd->sweepnumber << "\n";
        std::cout << "agg_interp_type: " << bd->agg_interp_type << "\n";
        std::cout << "coarse_solver: " << coarse_solver_type_to_string(bd->coarse_solver) << "\n";
        std::cout << "CRrelax_type: " << relax_type_to_string(bd->CRrelax_type) << "\n";
        std::cout << "CRrelax_weight: " << bd->CRrelax_weight << "\n";
        std::cout << "CRit: " << bd->CRit << "\n";
        std::cout << "CRratio: " << bd->CRratio << "\n";
        std::cout << "ftcoarse: " << bd->ftcoarse << "\n\n";
    }
}

namespace ApplyData {
    applyData* initDefault()
    {

        applyData* ad = NULL;
        ad = (applyData*)Malloc(sizeof(applyData));
        CHECK_HOST(ad);

        ad->cycle_type = CycleType::V_CYCLE;
        ad->relax_type = RelaxType::L1_JACOBI;
        ad->relaxnumber_coarse = 1;
        ad->prerelax_number = 1;
        ad->postrelax_number = 1;
        ad->relax_weight = 1.0;
        ad->num_grid_sweeps = NULL;

        return ad;
    }

    void free(applyData* ad)
    {
        std::free(ad->num_grid_sweeps);
        std::free(ad);
    }

    void print(applyData* ad)
    {
        std::cout << "\ncycle_type: " << cycle_type_to_string(ad->cycle_type) << "\n";
        std::cout << "relax_type: " << relax_type_to_string(ad->relax_type) << "\n";
        std::cout << "relaxnumber_coarse: " << ad->relaxnumber_coarse << "\n";
        std::cout << "prerelax_number: " << ad->prerelax_number << "\n";
        std::cout << "postrelax_number: " << ad->postrelax_number << "\n";
        std::cout << "relax_weight: " << ad->relax_weight << "\n\n";
    }

    applyData* initByParams(const params& p)
    {
        applyData* amg_cycle = AMG::ApplyData::initDefault();

        amg_cycle->cycle_type = p.cycle_type;
        amg_cycle->relax_type = p.relax_type;
        amg_cycle->relaxnumber_coarse = p.relaxnumber_coarse;
        amg_cycle->prerelax_number = p.prerelax_sweeps;
        amg_cycle->postrelax_number = p.postrelax_sweeps;

        return amg_cycle;
    }

    void setGridSweeps(applyData* ad, int max_level)
    {
        max_level--;
        ad->num_grid_sweeps = (int*)Malloc(max_level * sizeof(int));
        CHECK_HOST(ad->num_grid_sweeps);

        int i, j;

        for (i = 0; i < max_level; i++) {
            ad->num_grid_sweeps[i] = 1;
        }
    }
}

namespace BootBuildData {
    bootBuildData* initDefault()
    {

        bootBuildData* bd = NULL;
        bd = (bootBuildData*)Malloc(sizeof(bootBuildData));
        CHECK_HOST(bd);

        bd->max_hrc = 10;
        bd->conv_ratio = 0.80;
        bd->bootstrap_composition_type = BootstrapCompositionType::MULTIPLICATIVE;
        bd->solver_it = 15;

        bd->amg_data = AMG::BuildData::initDefault();

        return bd;
    }

    void free(bootBuildData* ad)
    {
        AMG::BuildData::free(ad->amg_data);
        std::free(ad);
    }

    void print(bootBuildData* ad)
    {
        std::cout << "\nmax_hrc: " << ad->max_hrc << "\n";
        std::cout << "conv_ratio: " << ad->conv_ratio << "\n";
        std::cout << "bootstrap_composition_type: " << bootstrap_composition_type_to_string(ad->bootstrap_composition_type) << "\n";
        std::cout << "solver_it: " << ad->solver_it << "\n\n";
    }

    bootBuildData* initByParams(CSR* A, params p)
    {
        bootBuildData* bootamg_data = AMG::BootBuildData::initDefault();
        buildData* amg_data = bootamg_data->amg_data;

        bootamg_data->bootstrap_composition_type = p.bootstrap_composition_type;
        bootamg_data->max_hrc = p.max_hrc;
        bootamg_data->conv_ratio = p.conv_ratio;

        amg_data->sweepnumber = p.aggrsweeps;
        amg_data->agg_interp_type = p.aggrtype;
        amg_data->maxlevels = p.max_levels;
        amg_data->coarse_solver = p.coarse_solver;
        amg_data->CRrelax_type = p.relax_type;

        amg_data->A = A;
        Vectorinit_CNT
            amg_data->w
            = Vector::init<vtype>(A->n, true, true);
        Vector::fillWithValue(amg_data->w, 1.0);

        AMG::BuildData::setMaxCoarseSize(amg_data);

        return bootamg_data;
    }
}

namespace Boot {

    boot* init(int n_hrc, double estimated_ratio)
    {
        boot* b = (boot*)Malloc(sizeof(boot));
        CHECK_HOST(b);

        b->n_hrc = n_hrc;
        b->estimated_ratio = estimated_ratio;
        b->H_array = (hierarchy**)Malloc(n_hrc * sizeof(hierarchy*));
        CHECK_HOST(b->H_array);

        return b;
    }

    void free(boot* b)
    {
        for (int i = 0; i < b->n_hrc; i++) {
            AMG::Hierarchy::free(b->H_array[i]);
        }
        std::free(b);
    }

    void finalize(boot* b, int num_hrc)
    {
        assert(num_hrc > 0);
        b->n_hrc = num_hrc;
        b->H_array = (hierarchy**)realloc(b->H_array, num_hrc * sizeof(hierarchy*));
        CHECK_HOST(b->H_array);
    }
}
}
