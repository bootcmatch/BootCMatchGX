#include "hip/hip_runtime.h"
#include "basic_kernel/halo_communication/halo_communication.h"
#include "basic_kernel/halo_communication/local_permutation.h"

#include "utility/function_cnt.h"
#include "utility/cudamacro.h"

#define USE_GETMCT
#define NUM_THR 1024
extern int scalennzmiss;
__global__
void _getMissingMask(itype nnz, itype *A_col, itype *missing, itype row_shift, itype n){
  itype i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i >= nnz)
    return;

  itype col = A_col[i];

  if(col < row_shift || col >= row_shift+n)
    missing[i] = col;
  else
    missing[i] = -1;
}

__global__
void _count(itype nnz, itype *missing, itype *c){
  itype i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i >= nnz)
    return;

  if(missing[i])
    atomicAdd(c, 1);
}

void getMissing(CSR *A, vector<itype> **missing, CSR *R, gstype *row_shift){
  _MPI_ENV;

  stype row_ns[nprocs];
  gstype ends[nprocs];
  
  CHECK_MPI(
    MPI_Allgather(
      &A->n,
      sizeof(stype),
      MPI_BYTE,
      row_ns,
      sizeof(stype),
      MPI_BYTE,
      MPI_COMM_WORLD
    )
  );

  ends[0] = row_ns[0];
  row_shift[0]=0;
  for(itype i=1; i<nprocs; i++) {
      ends[i] = row_ns[i] + ends[i-1];
      row_shift[i]=row_shift[i-1]+ends[i-1];
  }

  assert(ends[nprocs-1] == A->full_n);

#ifndef USE_GETMCT
  itype nnz = 0;

  if(R != NULL)
    nnz = R->nnz;
  else
    nnz = A->nnz;

  vector<itype> *mask, *mask_sorted;
  Vectorinit_CNT
  mask = Vector::init<itype>(nnz, true, true);
  scalar<itype> *d_num_selected_out = Scalar::init<itype>(0, true);

  Vectorinit_CNT
  mask_sorted = Vector::init<itype>(nnz, true, true);

  gridblock gb;

  if(R != NULL){
    gb = gb1d(nnz, BLOCKSIZE);
    _getMissingMask<<<gb.g, gb.b>>>(nnz, R->col, mask->val, A->row_shift, A->n);
  }else{
    gb = gb1d(nnz, BLOCKSIZE);
    _getMissingMask<<<gb.g, gb.b>>>(nnz, A->col, mask->val, A->row_shift, A->n);
  }

  // Determine temporary device storage requirements
  void     *d_temp_storage = NULL;
  size_t   temp_storage_bytes = 0;

  hipcub::DeviceRadixSort::SortKeys(
    d_temp_storage,
    temp_storage_bytes,
    mask->val,
    mask_sorted->val,
    nnz
  );

  cudaMalloc_CNT
  CHECK_DEVICE( hipMalloc(&d_temp_storage, temp_storage_bytes) );

  hipcub::DeviceRadixSort::SortKeys(
    d_temp_storage,
    temp_storage_bytes,
    mask->val,
    mask_sorted->val,
    nnz
  );

  MY_CUDA_CHECK( hipFree(d_temp_storage) );
  d_temp_storage = NULL;
  temp_storage_bytes = 0;

  hipcub::DeviceSelect::Unique(
    d_temp_storage,
    temp_storage_bytes,
    mask_sorted->val,
    mask->val,
    d_num_selected_out->val,
    nnz
  );

  cudaMalloc_CNT
  CHECK_DEVICE( hipMalloc(&d_temp_storage, temp_storage_bytes) );

  hipcub::DeviceSelect::Unique(
    d_temp_storage,
    temp_storage_bytes,
    mask_sorted->val,
    mask->val,
    d_num_selected_out->val,
    nnz
  );

  itype *cp = Scalar::getvalueFromDevice(d_num_selected_out);
  itype c = (*cp)-1;
  free(cp);
  
  for(int i=0; i<nprocs; i++){
    if(i != myid)
      missing[i]->n = 0;
  }

  if(c > 0){
    itype *missing_flat = NULL;
    missing_flat = (itype*)  malloc(sizeof(itype) * c);
    CHECK_HOST(missing_flat);
    hipMemcpy(missing_flat, mask->val+1, c * sizeof(itype), hipMemcpyDeviceToHost);

    itype J = 0, I = 0;
    for(itype i=0; i<c; i++){
      itype j = missing_flat[i];
      assert( (I+1)<(A->nnz/nprocs) );

      CHECK_AGAIN:
      if(j >= ends[J]){

        if(J != myid)
          missing[J]->n = I;

        J++;
        I = 0;
        goto CHECK_AGAIN;
      }
      missing[J]->val[I] = j;
      I++;
    }

    if(I){
      missing[J]->n = I;
    }

    free(missing_flat);
  }
  
  Vector::free(mask);
  Vector::free(mask_sorted);
  Scalar::free(d_num_selected_out);
  
#else
  // -----------------------------

  gstype mypfirstrow;
  mypfirstrow = A->row_shift; // (R != NULL) ? R->row_shift : A->row_shift;
        
  int uvs;
  int *getmct(itype *,itype,itype,itype,int *,int**,int*,int);
  int *ptr;
  if(R != NULL) {
    ptr = getmct( R->col, R->nnz, 0, A->n-1, &uvs, &(R->bitcol), &(R->bitcolsize), NUM_THR);
  } else {
    ptr = getmct( A->col, A->nnz, 0, A->n-1, &uvs, &(A->bitcol), &(A->bitcolsize), NUM_THR);
  }

  vector<int> *_bitcol;
  if(uvs == 0){ 
    _bitcol = Vector::init<int>(1, true, false);
  } else {
    _bitcol = Vector::init<int>(uvs, false, false);
    _bitcol->val=ptr;
  }
  
//   vector<itype> **missing2 = (vector<itype>**)malloc(sizeof(vector<itype>*)*nprocs);
//   for(int i=0; i<nprocs; i++){
//     missing2[i] = (uvs > 0) ? Vector::init<itype>(uvs, true, false) : Vector::init<int>(1, true, false);
//     if(i != myid)
//       missing2[i]->n = 0;
//   }

  for(int i=0; i<nprocs; i++){
    if(i != myid)
      missing[i]->n = 0;
  }

  if(uvs > 0){
#if 0
    itype cum_p_n_per_process[nprocs];
    cum_p_n_per_process[0]=ends[0]-1;
    for(int i=1; i<nprocs; i++){
     cum_p_n_per_process[i]=cum_p_n_per_process[i-1] + ends[i];
    }
#endif
    stype *missing_flat = NULL;
    missing_flat = (stype*)  malloc(sizeof(stype) * uvs);
    CHECK_HOST(missing_flat);
    memcpy(missing_flat, _bitcol->val, uvs * sizeof(stype));
    mypfirstrow = A->row_shift;
    stype J = 0, I = 0;
    for(itype i=0; i<uvs; i++){
      itype j = missing_flat[i];
      //assert( (I+1)<((A->nnz/nprocs)*scalennzmiss) );
      assert( (I+1)< ( (((long)(A->nnz)*(long)scalennzmiss))/((long)nprocs )) );
#if 1    
      CHECK_AGAIN:
      if((j+mypfirstrow) >= ends[J]){

        if(J != myid)
          missing[J]->n = I;

        J++;
        I = 0;
        goto CHECK_AGAIN;
      }
#else
      int bswhichprocess(itype *, int, itype);
      J = bswhichprocess(cum_p_n_per_process, nprocs, j+mypfirstrow);
      if(J > (nprocs-1)){
         J=nprocs-1;
      }
#endif
      if(J>=nprocs) { fprintf(stderr,"Task id %d: unexpected missing source :%d\n",myid,J); exit(1); }
      //if(J<0 || J>=nprocs) { fprintf(stderr,"Task id %d: unexpected missing source :%d\n",myid,J); exit(1); }
//      if(I>=(((A->nnz)*scalennzmiss)/nprocs) ) { fprintf(stderr,"Task id: unexpected I:%d, should be not greater than %d\n",myid,I, (((A->nnz)*scalennzmiss)/nprocs) ); exit(1); }
      missing[J]->val[I] = j+(mypfirstrow-(J?ends[J-1]:0));
      I++;
    }
    if(I){
      missing[J]->n = I;
    }
    free(missing_flat);
  }
  
//   for (int j=0; j<nprocs; j++) {
//     if (j == myid) {
//         printf("------------ proc %d ------------------\n", myid);
// 
//         printf("getmct:\n");
//         Vector::print(_bitcol, -1);
//         
//         printf("mask:\n");
//         Vector::print(mask, -1);
//         
//         for (int i=0; i<nprocs; i++) {
//             if (i!=myid) {
//                 printf("missing[%d]:\n", i);
//                 Vector::print(missing[i], -1);
//                 
//                 printf("missing2[%d]:\n", i);
//                 Vector::print(missing2[i], -1);
//     //             
//     //             printf("mask_sorted:\n");
//     //             Vector::print(mask_sorted, -1);
//     //             
//     //             vector<int> *temp_storage_vec = Vector::init<int>((temp_storage_bytes)/(sizeof(int)), false, true);
//     //             temp_storage_vec->val = (int*)d_temp_storage;
//     //             printf("d_temp_storage:\n");
//     //             Vector::print(temp_storage_vec, -1);
//             }
//         }
//         Vector::free(_bitcol);
//     }
//     MPI_Barrier(MPI_COMM_WORLD);
//   }
//   
//   for(int i=0; i<nprocs; i++){
//     Vector::free(missing2[i]);
//   }
//   std::free(missing2);
// //   exit(0);
  // -----------------------------
#endif
  
}

__global__
void _getToSendMask(itype n, itype *to_send, itype *to_send_mask, itype shift){
  itype i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i >= n)
    return;

  to_send_mask[to_send[i]-shift] = i;
}

halo_info haloSetup(CSR *A, CSR *R=NULL){
  PUSH_RANGE(__func__, 5)
    
  _MPI_ENV;
  if ( R != NULL ) {    // product compatibility check
      assert( R->m == A->full_n );
  } else {
      if(A->m != A->full_n) {
	      fprintf(stderr,"Task %d, in haloSetup: A->m=%lu, A->full_n=%lu\n",myid,A->m,A->full_n);
	      fflush(stderr);
      }
      assert( A->m == A->full_n );
  }
  vector<itype> **my_missing;
  int *sendcounts = (int*) malloc(sizeof(int)*nprocs);
  int *sdispls = (int*) malloc(sizeof(int)*nprocs);
  CHECK_HOST(sendcounts);
  CHECK_HOST(sdispls);
  int *recvcounts = (int*) malloc(sizeof(int)*nprocs);
  int *rdispls = (int*) malloc(sizeof(int)*nprocs);
  CHECK_HOST(recvcounts);
  CHECK_HOST(rdispls);
  vector<itype> *my_missing_flat = NULL;
  vector<itype> *their_missing_flat = NULL;
  itype their_missing_flat_total_n = 0;
  itype total_n = 0;
  gstype row_shift[nprocs];
  halo_info hi;
  
  if(1 || (A->rows_to_get==NULL || R!=NULL)) {
    my_missing = (vector<itype>**) malloc(sizeof(vector<itype> *) * nprocs);
    CHECK_HOST(my_missing);

    for(int i=0; i<nprocs; i++){
      //if ( ((A->nnz)*scalennzmiss)/(nprocs) == 0 ) {
      //	fprintf(stderr, "A->nnz = %d, nprocs = %d, scalennzmiss = %d \n", A->nnz, nprocs, scalennzmiss);
      //}	    
      if(i != myid)
	//my_missing[i] = Vector::init<itype>( ((A->nnz)/(nprocs)*scalennzmiss), true, false);
	my_missing[i] = Vector::init<itype>( ((long)(A->nnz)*(long)(scalennzmiss))/((long)(nprocs)), true, false);
      else
	my_missing[i] = NULL;
    }
    
    getMissing(A, my_missing, R,  row_shift);

    for(itype i=0; i<nprocs; i++){
      if(myid == i)
        continue;
      total_n += my_missing[i]->n;
    }

    if(total_n > 0){
      my_missing_flat = Vector::init<itype>(total_n, true, false);
      hi.to_receive = Vector::init<gstype>(total_n, true, false);      
    }

    itype shift = 0;
    for(itype i=0; i<nprocs; i++){

      if(myid == i){
        sendcounts[i] = 0;
        sdispls[i] = shift;
        continue;
      }

      if(my_missing[i]->n > 0)
        memcpy(my_missing_flat->val+shift, my_missing[i]->val, my_missing[i]->n*sizeof(itype));

      sendcounts[i] = my_missing[i]->n;
      sdispls[i] = shift;
      shift += my_missing[i]->n;
    }

    CHECK_MPI(
	      MPI_Alltoall(
			   sendcounts,
			   1,
			   ITYPE_MPI,
			   recvcounts,
			   1,
			   ITYPE_MPI,
			   MPI_COMM_WORLD
			   )
	      );

    shift = 0;
    for(itype i=0; i<nprocs; i++){
      rdispls[i] = shift;
      shift += recvcounts[i];
      their_missing_flat_total_n += recvcounts[i];
    }

    if(their_missing_flat_total_n > 0)
      their_missing_flat = Vector::init<itype>(their_missing_flat_total_n, true, false);

    CHECK_MPI(
	      MPI_Alltoallv(
			    my_missing_flat != NULL ? my_missing_flat->val : NULL,
			    sendcounts,
			    sdispls,
			    ITYPE_MPI,
			    their_missing_flat != NULL ? their_missing_flat->val : NULL,
			    recvcounts,
			    rdispls,
			    ITYPE_MPI,
			    MPI_COMM_WORLD
			    )
	      );
  }

  hi.init = true;
  if(1 || (A->rows_to_get==NULL || R!=NULL)) {
    hi.to_receive_n = total_n;
    int k=0;
    for(int i=0; i<nprocs; i++) {
    	    for(int j=0; j<sendcounts[i]; j++) {
	        	    hi.to_receive->val[k]=my_missing_flat->val[k]+row_shift[i];
			    k++;
	    }
    }
    hi.to_receive_counts = sendcounts;
    hi.to_receive_spls = sdispls;
  } else {
    for(int i=0; i<nprocs; i++) {
        A->halo.to_receive_counts[i]=A->rows_to_get->rcounts2[i]/sizeof(itype);
        A->halo.to_receive_spls[i]=A->rows_to_get->displr2[i]/sizeof(itype);
    }
    hi.to_receive = Vector::init<gstype>(A->rows_to_get->countall, true, false);
    hi.to_receive_n = A->rows_to_get->countall;
    hi.to_receive->val=A->rows_to_get->whichprow;
  }

  hi.what_to_receive = NULL;
  hi.to_receive_d = NULL;
  if(hi.to_receive_n > 0){
    hi.what_to_receive = (vtype*)malloc(sizeof(vtype)*hi.to_receive_n);
    CHECK_HOST(hi.what_to_receive);
    cudaMalloc_CNT
    CHECK_DEVICE( hipMalloc( (void**) &hi.what_to_receive_d , sizeof(vtype)*hi.to_receive_n ) );
    VectorcopyToDevice_CNT
    hi.to_receive_d = Vector::copyToDevice(hi.to_receive);
  }
  if(1 || (A->rows_to_get==NULL || R!=NULL)) {
    hi.to_send = their_missing_flat;
    hi.to_send_n = their_missing_flat_total_n;
    hi.to_send_counts = recvcounts;
    hi.to_send_spls = rdispls;
  } else {
    for(int i=0; i<nprocs; i++) {
        A->halo.to_send_counts[i]=A->rows_to_get->scounts2[i]/sizeof(itype);
        A->halo.to_send_spls[i]=A->rows_to_get->displs2[i]/sizeof(itype);
	their_missing_flat_total_n+=A->halo.to_send_counts[i];
    }
    hi.to_send = Vector::init<itype>(their_missing_flat_total_n, true, false);  
    hi.to_send_n = their_missing_flat_total_n;
    hi.to_send->val = A->rows_to_get->rcvprow;
  }

  hi.what_to_send_d = NULL;
  hi.what_to_send = NULL;

  if(hi.to_send_n > 0){

    cudaMalloc_CNT
    CHECK_DEVICE(  hipHostMalloc((void**)&hi.what_to_send, sizeof(vtype)*hi.to_send_n) );

    CHECK_HOST(hi.what_to_send);
    cudaMalloc_CNT
    CHECK_DEVICE( hipMalloc( (void**) &hi.what_to_send_d , sizeof(vtype)*hi.to_send_n ) );
    VectorcopyToDevice_CNT
    hi.to_send_d = Vector::copyToDevice(hi.to_send);
  }

  if(1 || (A->rows_to_get==NULL || R!=NULL)) {
  for(itype i=0; i<nprocs; i++){
    if(my_missing[i] != NULL)
      Vector::free(my_missing[i]);
  }
  std::free(my_missing);
  if(my_missing_flat != NULL) 
     Vector::free(my_missing_flat);  
  }
  POP_RANGE
  return hi;
}

__global__
void _getToSend(itype n, vtype *x, vtype *what_to_send, itype *to_send, itype shift){
  itype i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i >= n)
    return;

  itype j = to_send[i];
  what_to_send[i] = x[j+shift];
}

__global__
void _getToSend_new(itype n, vtype *x, vtype *what_to_send, itype *to_send, itype shift){
  itype i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i >= n)
    return;

  itype j = to_send[i];
  what_to_send[i] = x[j /* - shift */];
}

__global__
void setReceivedWithMask(itype n, vtype *x, vtype *received, gstype *receive_map, itype shift){
  itype i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i >= n)
    return;

  itype j = receive_map[i];
  vtype val = received[i];
  x[j /* +shift */] = val;
}

__global__
void setReceivedWithMask_new(itype n, vtype *x, vtype *received, gstype *receive_map, itype shift){
  itype i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i >= n)
    return;

  //itype j = receive_map[i];
  //vtype val = received[i];
}

#define SYNCSOL_TAG 4321
#define MAXNTASKS 4096

void halo_sync(halo_info hi, CSR *A, vector<vtype> *x, bool local_flag){
  _MPI_ENV;

  assert(A->on_the_device);
  assert(x->on_the_device);
  static MPI_Request requests[MAXNTASKS];
  static hipStream_t sync_stream; 
  static int first=1;
  if(first) {
	  first=0;
	  CHECK_DEVICE( hipStreamCreate(&sync_stream) );
  }

  gridblock gb;

  if(hi.to_send_n){
#if SMART_AGGREGATE_GETSET_GPU == 1
    gridblock gb = gb1d(hi.to_send_n, BLOCKSIZE);
    if (local_flag)
        _getToSend_new<<<gb.g, gb.b>>>(hi.to_send_d->n, x->val, hi.what_to_send_d, hi.to_send_d->val, A->row_shift);
    else
        _getToSend<<<gb.g, gb.b>>>(hi.to_send_d->n, x->val, hi.what_to_send_d, hi.to_send_d->val, A->row_shift);
    CHECK_DEVICE( hipMemcpyAsync(hi.what_to_send, hi.what_to_send_d, hi.to_send_n*sizeof(vtype), hipMemcpyDeviceToHost, sync_stream) );
#else
    vector<vtype> *x_host = Vector::copyToHost(x);
    int start = 0;
    for(int i=0; i<nprocs; i++){
      int end = start + hi.to_send_counts[i];
      for(int j=start; j<end; j++){
        itype v = hi.to_send->val[j];
        hi.what_to_send[j] = x_host->val[v];
      }
      start = end;
    }
#endif
  }
    int j=0, ntr;
    for(int t=0; t<nprocs; t++) {
        if(t==myid) continue;
        if(hi.to_receive_counts[t]>0) {
            CHECK_MPI (
                MPI_Irecv(hi.what_to_receive+(hi.to_receive_spls[t]),hi.to_receive_counts[t],VTYPE_MPI,t,SYNCSOL_TAG,MPI_COMM_WORLD,requests+j));
            j++;
            if(j==MAXNTASKS) {
                fprintf(stderr,"Too many tasks in halo_sync, max is %d\n",MAXNTASKS);
                exit(1);
            }
        }
    }
    ntr=j;
    if(hi.to_send_n){
        hipStreamSynchronize(sync_stream);
    }

    for(int t=0; t<nprocs; t++) {
        if(t==myid) continue;
        if(hi.to_send_counts[t]>0) {
            CHECK_MPI (MPI_Send(hi.what_to_send+(hi.to_send_spls[t]),hi.to_send_counts[t],VTYPE_MPI,t,SYNCSOL_TAG,MPI_COMM_WORLD));
        }
    }

    if(!hi.to_receive_n)
        return;
    if(ntr>0) { CHECK_MPI(MPI_Waitall(ntr,requests,MPI_STATUSES_IGNORE)); }

    #if SMART_AGGREGATE_GETSET_GPU == 1
        CHECK_DEVICE( hipMemcpy(hi.what_to_receive_d, hi.what_to_receive, hi.to_receive_n * sizeof(vtype), hipMemcpyHostToDevice) );
        
        gb = gb1d(hi.to_receive_n , BLOCKSIZE);
        if (local_flag) {
            if (x->n == A->full_n)
                setReceivedWithMask<<<gb.g, gb.b>>>(hi.to_receive_n , x->val, hi.what_to_receive_d, hi.to_receive_d->val, A->row_shift);
        } else {
            setReceivedWithMask<<<gb.g, gb.b>>>(hi.to_receive_n , x->val, hi.what_to_receive_d, hi.to_receive_d->val, A->row_shift);
        }
    #else
        if (x->n == A->full_n) {        // PICO
            vector<vtype> *x_host = Vector::copyToHost(x);
            int start = 0;
            for(int i=0; i<nprocs; i++){
                int end = start + hi.to_receive_counts[i];
                for(int j=start; j<end; j++){
                    gstype v = hi.to_receive->val[j];
                    x_host->val[v] = hi.what_to_receive[j];
                }
                start = end;
            }
            CHECK_DEVICE( hipMemcpy(x->val, x_host->val, x_host->n * sizeof(vtype), hipMemcpyHostToDevice) );
            Vector::free(x_host);
        }
    #endif
    
}

bool checkSync(CSR *_A, vector<vtype> *_x0, vector<vtype> *_x1, int level){
  _MPI_ENV;
  CSR *A = CSRm::copyToHost(_A);
  vector<vtype> *x0 = Vector::copyToHost(_x0);
  vector<vtype> *x1 = Vector::copyToHost(_x1);

  bool flag = true;
  for(int i=0; i<A->n; i++){
    for(int j=A->row[i]; j<A->row[i+1]; j++){
      itype col = A->col[j];
      if(x0->val[col] != x1->val[col]){
        printf("n %d] %d} -- col: %d | ", myid, level, col);
        std::cout << x0->val[col] << " ---- " << x1->val[col] << "\n";
        flag = false;
      }    
    }
  }

  CSRm::free(A);
  Vector::free(x0);
  Vector::free(x1);

  return flag;
}


void halo_sync_stream(halo_info hi, CSR *A, vector<vtype> *x, hipStream_t stream=0, bool local_flag = false){
  _MPI_ENV;

  assert(A->on_the_device);
  assert(x->on_the_device);

  gridblock gb;

  if(hi.to_send_n){
    gridblock gb = gb1d(hi.to_send_n, BLOCKSIZE);
    
    _getToSend<<<gb.g, gb.b, 0, stream>>>(hi.to_send_d->n, x->val, hi.what_to_send_d, hi.to_send_d->val, A->row_shift);
    CHECK_DEVICE( hipMemcpyAsync(hi.what_to_send, hi.what_to_send_d, hi.to_send_n*sizeof(vtype), hipMemcpyDeviceToHost, stream) );
  }

  hipStreamSynchronize(stream);

  CHECK_MPI(
    MPI_Alltoallv(
      hi.what_to_send,
      hi.to_send_counts,
      hi.to_send_spls,
      VTYPE_MPI,
      hi.what_to_receive,
      hi.to_receive_counts,
      hi.to_receive_spls,
      VTYPE_MPI,
      MPI_COMM_WORLD
    )
  );


  if(!hi.to_receive_n)
    return;

    CHECK_DEVICE( hipMemcpyAsync(hi.what_to_receive_d, hi.what_to_receive, hi.to_receive_n * sizeof(vtype), hipMemcpyHostToDevice, stream) );
    if (x->n == A->full_n) {        // PICO
        gb = gb1d(hi.to_receive_n , BLOCKSIZE);
        setReceivedWithMask<<<gb.g, gb.b, 0, stream>>>(hi.to_receive_n , x->val, hi.what_to_receive_d, hi.to_receive_d->val, A->row_shift);
    }
}

