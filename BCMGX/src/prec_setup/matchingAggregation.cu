#include "hip/hip_runtime.h"
float SUITOR_TIME = 0;
float TOTAL_MUL_TIME = 0;
float TOTAL_MATCH_TIME = 0;
float TOTAL_SETUP_TIME = 0;
float TOTAL_MEM_TIME = 0;
float TOTAL_RELAX_TIME = 0;
float TOTAL_SHIFTED_CSRVEC=0;
float TOTAL_MAKE_P=0;
float TOTAL_TRA_P=0;
float TOTAL_MAKEAHW_TIME=0;
float TOTAL_MATCHINGPAIR_TIME=0;
float TOTAL_OTHER_TIME=0;
int   DETAILED_TIMING=0;
extern char idstring[];
#include "matchingAggregation.h"

#include "utility/memoryPools.cu"
#include "prec_setup/suitor.cu"
#include "prec_setup/matching.cu"
#include "matchingPairAggregation.cu"

#include "utility/cudamacro.h"
int MUL_NUM = 0;
int I = 0;

#include "utility/function_cnt.h"

#define FTCOARSE_INC 100
#define COARSERATIO_THRSLD 1.2

#include "basic_kernel/halo_communication/local_permutation.h"
#include "basic_kernel/custom_cudamalloc/custom_cudamalloc.h"
#include "utility/timing.h"

itype *iPtemp1;
vtype *vPtemp1;
itype *iAtemp1;
vtype *vAtemp1;
itype *idevtemp1;
vtype *vdevtemp1;
itype *idevtemp2;
// --------- TEST ----------
itype * dev_rcvprow_stat;
vtype * completedP_stat_val;
itype * completedP_stat_col;
itype * completedP_stat_row;
// -------- AH glob --------
itype * AH_glob_row;
itype * AH_glob_col;
vtype * AH_glob_val;
// -------------------------
int * buffer_4_getmct;
int sizeof_buffer_4_getmct = 0;
unsigned int * idx_4shrink;
bool alloced_idx = false;
// ------ cuCompactor ------
int * glob_d_BlocksCount;
int * glob_d_BlocksOffset;
// -------------------------

void relaxPrepare(handles *h, int level, CSR *A, hierarchy *hrrch, buildData *amg_data, int force_relax_type=-1){
  PUSH_RANGE(__func__, 5)
    
  int relax_type;

  if(force_relax_type != -1)
    relax_type = force_relax_type;
  else
    relax_type = amg_data->CRrelax_type;

  if(relax_type == 0){
    // jacobi
    if(hrrch->D_array[level] != NULL)
      Vector::free(hrrch->D_array[level]);
    hrrch->D_array[level] = CSRm::diag(A);

  }else if(relax_type == 4){
    // L1 smoother
    if(hrrch->D_array[level] != NULL)
      Vector::free(hrrch->D_array[level]);
    hrrch->D_array[level] = CSRm::diag(A);

    if(hrrch->M_array[level] != NULL)
      Vector::free(hrrch->M_array[level]);
    hrrch->M_array[level] = CSRm::absoluteRowSum(A, NULL);
  }
  
  POP_RANGE
}

vector<itype>* makePCol_CPU(vector<itype> *mask, itype *ncolc){

  vector<itype> *col = Vector::init<itype>(mask->n, true, false);

  for(itype v=0; v<mask->n; v++){
    itype u = mask->val[v];
    if((u>=0) && (v != u) && (v < u)){
      col->val[v] = ncolc[0];
      col->val[u] = ncolc[0];
      ncolc[0]++;
    }
  }

  for(itype v=0; v<mask->n; v++){
    if(mask->val[v] == -2){
	     col->val[v] = ncolc[0]-1;
	  }else if (mask->val[v] == -1){
	     col->val[v] = ncolc[0];
	     ncolc[0]++;
	   }
  }
  return col;
}


__global__
void __setPsRow4prod(itype n, itype *row, itype nnz, itype start, itype stop){

  itype i = blockDim.x * blockIdx.x + threadIdx.x;

  if ( i >= n ) {
     return;
  }

  if(i < start){
    row[i] = 0;
  }

  if(i > stop){
    row[i] = nnz;
  }
}

CSR* matchingAggregation(handles *h, buildData *amg_data, CSR *A, vector<vtype> **w, CSR **P, CSR **R, int level){
  PUSH_RANGE(__func__, 5)
    
  _MPI_ENV;
  TIMER_DEF;
  static int cnt=0;
  CSR *Ai_ = A, *Ai = NULL;

  CSR *Ri_ = NULL;
  vector<vtype> *wi_ = *w, *wi = NULL;

  double size_coarse, size_precoarse;
  double coarse_ratio;

  for(int i=0; i<amg_data->sweepnumber; i++){
    CSR *Pi_;
    if (0 && myid==0) fprintf(stderr,"Task %d reached line %d \n",myid,__LINE__);

    if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TIME::start();
      TIMER_START;
    }
    matchingPairAggregation(h, Ai_, wi_, &Pi_, &Ri_, (i==0)); /* routine with the real work. It calls the suitor procedure */
    if (0 && myid==0) fprintf(stderr,"Task %d reached line %d \n",myid,__LINE__);
    char MName[256];
    sprintf(MName,"Pi%d_%s",cnt,idstring);
    CSRm::printMM(Pi_,MName);
    sprintf(MName,"Ri%d_%s",cnt,idstring);
    
    if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TOTAL_MATCHINGPAIR_TIME += TIME::stop();
      TIMER_STOP;
      TOTAL_MATCHINGPAIR_TIME += TIMER_ELAPSED;
    }

    if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TIME::start();
      TIMER_START;
    }
    char AiName[256], APName[256];
    sprintf(APName,"AP%d_%s",cnt,idstring);
    sprintf(AiName,"Ai%d_%s",cnt++,idstring);
    // --------------- PICO ------------------
    CSR *AP;
    AP  = nsparseMGPU_commu_new(h, Ai_, Pi_, false);
    CSRm::shift_cols(Ri_, -AP->row_shift);
    Ri_->col_shifted=-AP->row_shift;  
    Ai = nsparseMGPU_noCommu_new(h, Ri_, AP);  //, (i+1 < amg_data->sweepnumber ? false : true));
    if(myid!=0 && Ai->col_shifted==0) {
     CSRm::shift_cols(Ai, -(Ai->row_shift) );
     Ai->col_shifted=-(Ai->row_shift);
    }
    if(myid!=0 && AP->col_shifted==0) { /* This is only for debugging */
     CSRm::shift_cols(AP, -(AP->row_shift) );
     AP->col_shifted=-(AP->row_shift);
    }
    CSRm::printMM(Ri_,MName);
    CSRm::printMM(Ai,AiName);
    CSRm::printMM(AP,APName);
	       
    if (0 && myid==0) fprintf(stderr,"Task %d reached line %d \n",myid,__LINE__);
    // ---------------------------------------
        
    if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TOTAL_MUL_TIME += TIME::stop();
      TIMER_STOP;
      TOTAL_MUL_TIME += TIMER_ELAPSED;
      MUL_NUM += 2;
    }

    if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TIME::start();
      TIMER_START;
    }
    CSRm::free(AP);
    if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TOTAL_OTHER_TIME += TIME::stop();
      TIMER_STOP;
      TOTAL_OTHER_TIME += TIMER_ELAPSED;
    }
    // ------------- custom hipMalloc -------------
    // Vectorinit_CNT
    // wi = Vector::init<vtype>(Ai->n, true, true);
    // >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
    wi = Vector::init<vtype>(Ai->n, false, true);
    wi->val = CustomCudaMalloc::alloc_vtype(Ai->n, 1);
    // ---------------------------------------------

    if(DETAILED_TIMING && ISMASTER){
         hipDeviceSynchronize();
//          TIME::start();
         TIMER_START;
    }
    CSRm::shifted_CSRVector_product_adaptive_miniwarp2(Ri_, wi_, wi, 0 /* Ai_->row_shift */);
    if(DETAILED_TIMING && ISMASTER){
        hipDeviceSynchronize();
//         TOTAL_SHIFTED_CSRVEC += TIME::stop();
        TIMER_STOP;
        TOTAL_SHIFTED_CSRVEC += TIMER_ELAPSED;
    }

    size_precoarse = Ai_->full_n;
    size_coarse = Ai->full_n;
    coarse_ratio = size_precoarse / size_coarse;
    
    if (coarse_ratio <= COARSERATIO_THRSLD){
      amg_data->ftcoarse = FTCOARSE_INC;
    }
    
    bool brk_flag = (i+1 >= amg_data->sweepnumber) || (size_coarse <= amg_data->ftcoarse * amg_data->maxcoarsesize);

    if(i == 0){
      *P = Pi_;
    }else{

      if(DETAILED_TIMING && ISMASTER){
         hipDeviceSynchronize();
//          TIME::start();
         TIMER_START;
      }

      CSRm::shift_cols(*P, -(Pi_->row_shift) );      
      (*P)->m = (unsigned long)Pi_->n;
      csrlocinfo Pinfo1p; 
      Pinfo1p.fr=0;
      Pinfo1p.lr=Pi_->n;
      Pinfo1p.row=Pi_->row;
      Pinfo1p.col=NULL;
      Pinfo1p.val=Pi_->val;

      CSR *tmpP = *P; 
      *P = nsparseMGPU(*P, Pi_, &Pinfo1p, brk_flag);
      CSRm::free(tmpP);
      if(DETAILED_TIMING && ISMASTER){
        hipDeviceSynchronize();
//         TOTAL_MUL_TIME += TIME::stop();
        TIMER_STOP;
        TOTAL_MUL_TIME += TIMER_ELAPSED;
        MUL_NUM += 1;
      }

      if(DETAILED_TIMING && ISMASTER){
     	hipDeviceSynchronize();
//         TIME::start();
        TIMER_START;
      }
      //Ri_->row -= Ri_->row_shift;
      CSRm::free(Ri_);
      Ri_ = NULL;
      CSRm::free(Pi_);
      CSRm::free(Ai_);
      if(DETAILED_TIMING && ISMASTER){
         hipDeviceSynchronize();
//          TOTAL_OTHER_TIME += TIME::stop();
         TIMER_STOP;
         TOTAL_OTHER_TIME += TIMER_ELAPSED;
      }
    }
    // ------------- custom hipMalloc -------------
    //Vector::free(wi_);
    std::free(wi_);
    // ---------------------------------------------

    if(size_coarse <= amg_data->ftcoarse * amg_data->maxcoarsesize){
      break;
    }

    Ai_ = Ai;
    wi_ = wi;
    if(myid!=0 && Ai_->col_shifted==0) {
    	CSRm::shift_cols(Ai_, -(Ai_->row_shift) );
        Ai_->col_shifted=-(Ai_->row_shift);
    }
  }

  *w = wi;
  if (0 && myid==0) fprintf(stderr,"Task %d reached line %d \n",myid,__LINE__);
  if(Ri_ == NULL){

    if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TIME::start();
      TIMER_START;
    }
    //*R = CSRm::T(h->cusparse_h0, *P);
    if(nprocs > 1){
      //itype ms[nprocs];
      gstype  m_shifts[nprocs];
      // send columns numbers to each process
      m_shifts[myid]=Ai->row_shift;
      CSRm::shift_cols(*P, -m_shifts[myid]);
      
      gstype swp_m = (*P)->m;
      if (myid == nprocs-1){
          (*P)->m = Ai->n;
      }else{
          (*P)->m = Ai->n /* m_shifts[myid+1]-m_shifts[myid] */;
      }

      *R = CSRm::T_multiproc(h->cusparse_h0, *P, Ai->n, true);
    
      (*P)->m = swp_m;
      CSRm::shift_cols(*P, m_shifts[myid]);
      
      //(*R)->row += m_shifts[myid];
      //(*R)->n = Ai->n;
      (*R)->m = (*P)->full_n;
      (*R)->full_n = (*P)->m;
      CSRm::shift_cols(*R, (*P)->row_shift);
      (*R)->row_shift = m_shifts[myid];
    }else{
      *R = CSRm::T(h->cusparse_h0, *P);
    }
    if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TOTAL_TRA_P += TIME::stop();
      TIMER_STOP;
      TOTAL_TRA_P += TIMER_ELAPSED;
    }
  }else{
    *R = Ri_;
  }
  
  POP_RANGE
  
  if(myid!=0 && Ai->col_shifted==0) {
     CSRm::shift_cols(Ai, -(Ai->row_shift) );
     Ai->col_shifted=-(Ai->row_shift);
  }

  return Ai;
}


hierarchy* adaptiveCoarsening(handles *h, buildData *amg_data, const params p, bool precondition_flag){
  PUSH_RANGE(__func__,4)
    
  _MPI_ENV;
  TIMER_DEF;
  
  if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TIME::start();
      TIMER_START;
  }
  CSR *A = amg_data->A;

  // init memory pool
  MemoryPool::initContext(A->full_n, A->n);
  iPtemp1 = NULL;
  vPtemp1 = NULL;
 
  MY_CUDA_CHECK( hipHostMalloc(&iAtemp1, sizeof (itype) * p.mem_alloc_size ) );
  MY_CUDA_CHECK( hipHostMalloc(&vAtemp1, sizeof (vtype) * p.mem_alloc_size ) );
  cudaMalloc_CNT
  MY_CUDA_CHECK( hipMalloc(&idevtemp1, sizeof (itype) * p.mem_alloc_size ) ); 
  cudaMalloc_CNT
  MY_CUDA_CHECK( hipMalloc(&vdevtemp1, sizeof (vtype) * p.mem_alloc_size ) ); 
  cudaMalloc_CNT
  MY_CUDA_CHECK( hipMalloc(&idevtemp2, sizeof (itype) * p.mem_alloc_size ) );
  // -------- AH glob --------
  cudaMalloc_CNT
  MY_CUDA_CHECK( hipMalloc(&AH_glob_row, sizeof (itype) * (A->n +1) ) ); 
  cudaMalloc_CNT
  MY_CUDA_CHECK( hipMalloc(&AH_glob_col, sizeof (itype) * A->nnz ) );
  cudaMalloc_CNT
  MY_CUDA_CHECK( hipMalloc(&AH_glob_val, sizeof (vtype) * A->nnz ) );;
  // -------------------------
  
  vector<vtype> *w = amg_data->w;
  //vector<vtype> *w_temp = Vector::clone(w);
  vector<vtype> *w_temp;

  // -----  CustomCudaMalloc ---- //
  if( w->on_the_device ) {
    w_temp = Vector::init<vtype>(w->n, false, true);
    w_temp->val = CustomCudaMalloc::alloc_vtype(w->n, 1);
    hipError_t err;
    err = hipMemcpy(w_temp->val, w->val, w_temp->n * sizeof(vtype), hipMemcpyDeviceToDevice);
    CHECK_DEVICE(err);
  }else{
    w_temp = Vector::clone(w);
  }    
  // ----------------------------//

  CSR *P = NULL, *R = NULL;
  hierarchy *hrrch = AMG::Hierarchy::init(amg_data->maxlevels + 1);
  hrrch->A_array[0] = A;
  
  if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TOTAL_MEM_TIME += TIME::stop();
      TIMER_STOP;
      TOTAL_MEM_TIME += TIMER_ELAPSED;
  }

  // compute comunication patterns for solver
  if(nprocs > 1){
    if(DETAILED_TIMING && ISMASTER){
//       TIME::start();
        TIMER_START;
    }
    halo_info hi = haloSetup(hrrch->A_array[0], NULL);
//    printf("Task %d, halo done for %x level=%d\n",myid,hrrch->A_array[0],0);
    
    if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TOTAL_SETUP_TIME += TIME::stop();
      TIMER_STOP;
      TOTAL_SETUP_TIME += TIMER_ELAPSED;
    }
    hrrch->A_array[0]->halo = hi;
  }

  vtype avcoarseratio = 0.;
  int level = 0;
  if (precondition_flag) {
    if(DETAILED_TIMING && ISMASTER){
  //      TIME::start();
        TIMER_START;
    }
    relaxPrepare(h, level, hrrch->A_array[level], hrrch, amg_data);
    if(DETAILED_TIMING && ISMASTER){
        hipDeviceSynchronize();
  //       TOTAL_RELAX_TIME += TIME::stop();
        TIMER_STOP;
        TOTAL_RELAX_TIME += TIMER_ELAPSED;
    }
  }

  amg_data->ftcoarse = 1;

  if(precondition_flag){
    for(level=1; level < amg_data->maxlevels;){
      if(0 && myid==0) fprintf(stderr,"Task %d entering level %d\n",myid,level);
      if(DETAILED_TIMING && ISMASTER){
//           TIME::start();
          TIMER_START;
      }
      hrrch->A_array[level] = matchingAggregation(h, amg_data, hrrch->A_array[level-1], &w_temp, &P, &R, level-1);
      if(0 && myid==0) fprintf(stderr,"Task %d out of matchingAggregation\n",myid);
      if(DETAILED_TIMING && ISMASTER){
        hipDeviceSynchronize();
//         TOTAL_MATCH_TIME += TIME::stop();
        TIMER_STOP;
        TOTAL_MATCH_TIME += TIMER_ELAPSED;
      }
 
      if(nprocs > 1){
        if(DETAILED_TIMING && ISMASTER){
//           TIME::start();
            TIMER_START;
        }
	if(myid!=0 && hrrch->A_array[level]->col_shifted==0) {
  	      CSRm::shift_cols(hrrch->A_array[level], -(hrrch->A_array[level]->row_shift) );
	      hrrch->A_array[level]->col_shifted=-(hrrch->A_array[level]->row_shift);
	}
        halo_info hi = haloSetup(hrrch->A_array[level], NULL);
//        printf("Task %d, halo done for %x, level=%d\n",myid,hrrch->A_array[level],level);
        if(0 && myid==0) fprintf(stderr,"Task %d out of haloSetup\n",myid);
        if(DETAILED_TIMING && ISMASTER){
            hipDeviceSynchronize();
//             TOTAL_SETUP_TIME += TIME::stop();
            TIMER_STOP;
            TOTAL_SETUP_TIME += TIMER_ELAPSED;
        }
        hrrch->A_array[level]->halo = hi;
      }

      if(!amg_data->agg_interp_type){
       if(DETAILED_TIMING && ISMASTER){
//           TIME::start();
           TIMER_START;
       }
       relaxPrepare(h, level, hrrch->A_array[level], hrrch, amg_data);
       if(0 && myid==0)  fprintf(stderr,"Task %d out of relaxPrepare\n",myid);
       if(DETAILED_TIMING && ISMASTER){
          hipDeviceSynchronize();
//           TOTAL_RELAX_TIME += TIME::stop();
          TIMER_STOP;
          TOTAL_RELAX_TIME += TIMER_ELAPSED;
       }
      }

      hrrch->P_array[level-1] = P;
      hrrch->R_array[level-1] = R;
      
      // --------------- PICO ------------------
      bool shrink_col(CSR*, CSR*);
//      printf("Task %d, shrinking col of matrix %x, level %d\n",myid,hrrch->A_array[level-1],level-1);    
      shrink_col(hrrch->A_array[level-1], NULL);
      if(myid!=0 && hrrch->P_array[level-1]->col_shifted==0) {
           CSRm::shift_cols(hrrch->P_array[level-1], -(hrrch->A_array[level]->row_shift));
	   hrrch->P_array[level-1]->col_shifted=-(hrrch->A_array[level]->row_shift);
      }
      shrink_col(hrrch->P_array[level-1], hrrch->A_array[level]);
      
      if (level != hrrch->num_levels-1) {
        if(myid!=0 && hrrch->R_array[level-1]->col_shifted==0) {
	   hrrch->R_array[level-1]->bitcol=NULL;
	   hrrch->R_array[level-1]->bitcolsize=0; 
           CSRm::shift_cols(hrrch->R_array[level-1], -(hrrch->A_array[level-1]->row_shift));
     	   hrrch->R_array[level-1]->col_shifted=-(hrrch->A_array[level-1]->row_shift);
        }

        shrink_col(hrrch->R_array[level-1], hrrch->A_array[level-1]);
      } 
      // ---------------------------------------

      if(nprocs > 1){

        if(DETAILED_TIMING && ISMASTER){
//             TIME::start();
            TIMER_START;
        }
        halo_info hi = haloSetup(hrrch->A_array[level], hrrch->P_array[level-1]);
        if(0 && myid==0) fprintf(stderr,"Task %d out of haloSetup 2\n",myid);
	
        if(DETAILED_TIMING && ISMASTER){
            hipDeviceSynchronize();
//             TOTAL_SETUP_TIME += TIME::stop();
            TIMER_STOP;
            TOTAL_SETUP_TIME += TIMER_ELAPSED;
        }
        hrrch->P_array[level-1]->halo = hi;
        
      }
      
      if(nprocs > 1 && (level != hrrch->num_levels-1)) {
        
        if(DETAILED_TIMING && ISMASTER){
//             TIME::start();
            TIMER_START;
        }
        halo_info hi = haloSetup(hrrch->A_array[level-1], hrrch->R_array[level-1]); // BUG: haloSetup(hrrch->R_array[level-1], hrrch->A_array[level-1]);
        if(0 && myid==0) fprintf(stderr,"Task %d out of haloSetup 3\n",myid);
        if(DETAILED_TIMING && ISMASTER){
            hipDeviceSynchronize();
//             TOTAL_SETUP_TIME += TIME::stop();
            TIMER_STOP;
            TOTAL_SETUP_TIME += TIMER_ELAPSED;
        }
        hrrch->R_array[level-1]->halo = hi;
        
      }
      
      vtype size_coarse = hrrch->A_array[level]->full_n;

      vtype coarse_ratio = hrrch->A_array[level-1]->full_n / size_coarse;
      avcoarseratio = avcoarseratio + coarse_ratio;
      level++;

      if(size_coarse <= amg_data->ftcoarse * amg_data->maxcoarsesize){
        break;
      }
      if(0 && myid==0) fprintf(stderr,"Task %d end level %d\n",myid,level);
    }
//    printf("Task %d, shrinking col of matrix %x, level %d\n",myid,hrrch->A_array[level-1],level-1);    
//#if LOCAL_COARSEST==0
    if( p.coarsesolver_type == 0){
      shrink_col(hrrch->A_array[level-1], NULL);
    }
//#endif
  } else {
    bool shrink_col(CSR*, CSR*);
    shrink_col(hrrch->A_array[level], NULL);
  }
  if(0 && myid==0) fprintf(stderr,"Task %d end loop on levels\n",myid);

  if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TIME::start();
      TIMER_START;
  }

// ### Start Free ASYNC
//  MY_CUDA_CHECK( hipFreeAsync(idevtemp1, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(idevtemp2, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(vdevtemp1, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(dev_rcvprow_stat, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(completedP_stat_val, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(completedP_stat_col, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(completedP_stat_row, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(AH_glob_row, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(AH_glob_col, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(AH_glob_val, h->stream_free));
//  MY_CUDA_CHECK( hipFreeAsync(buffer_4_getmct, h->stream_free));
//  if (alloced_idx == true) {
//    MY_CUDA_CHECK( hipFreeAsync(idx_4shrink, h->stream_free));
//  }
// ### End Free ASYNC  


  if (precondition_flag) {
    AMG::Hierarchy::finalize_level(hrrch, level);
    AMG::Hierarchy::finalize_cmplx(hrrch);
    AMG::Hierarchy::finalize_wcmplx(hrrch);
    hrrch->avg_cratio = avcoarseratio / (level-1);

    if(ISMASTER){
      AMG::Hierarchy::printInfo(hrrch);
      Eval::printMetaData("agg;level_number", level, 0);
      Eval::printMetaData("agg;avg_coarse_ratio", hrrch->avg_cratio, 1);
      Eval::printMetaData("agg;OpCmplx", hrrch->op_cmplx, 1);
      Eval::printMetaData("agg;OpCmplxW", hrrch->op_wcmplx, 1);
      Eval::printMetaData("agg;coarsest_size", hrrch->A_array[level-1]->full_n, 0);
      Eval::printMetaData("agg;total_mul_num", MUL_NUM, 0);
    }
  }
  MY_CUDA_CHECK( hipHostFree(iPtemp1));
  free(vPtemp1);
  MY_CUDA_CHECK( hipHostFree(iAtemp1));
  MY_CUDA_CHECK( hipHostFree(vAtemp1));
  MY_CUDA_CHECK( hipFree(idevtemp1));
  MY_CUDA_CHECK( hipFree(idevtemp2));
  MY_CUDA_CHECK( hipFree(vdevtemp1));
 // ----------------- TEST --------------------
  MY_CUDA_CHECK( hipFree(dev_rcvprow_stat));
  MY_CUDA_CHECK( hipFree(completedP_stat_val));
  MY_CUDA_CHECK( hipFree(completedP_stat_col));
  MY_CUDA_CHECK( hipFree(completedP_stat_row));
 // --------------- AH glob -------------------
  MY_CUDA_CHECK( hipFree(AH_glob_row));
  MY_CUDA_CHECK( hipFree(AH_glob_col));
  MY_CUDA_CHECK( hipFree(AH_glob_val));
 // -------------------------------------------
  MY_CUDA_CHECK( hipFree(buffer_4_getmct));
  if (alloced_idx == true) {
    MY_CUDA_CHECK( hipFree(idx_4shrink));
  }
  // ------------ cuCompactor ------------------
  MY_CUDA_CHECK( hipFree(glob_d_BlocksCount));
  MY_CUDA_CHECK( hipFree(glob_d_BlocksOffset));
  // -------------------------------------------
  
  // ------------- custom hipMalloc -------------
//   Vector::free(w_temp);
  std::free(w_temp);
  // ---------------------------------------------

  MemoryPool::freeContext();
  if(DETAILED_TIMING && ISMASTER){
      hipDeviceSynchronize();
//       TOTAL_MEM_TIME += TIME::stop();
      TIMER_STOP;
      TOTAL_MEM_TIME += TIMER_ELAPSED;
  }
  if(DETAILED_TIMING && ISMASTER){
    Eval::printMetaData("agg;SUITOR_TIME", SUITOR_TIME / 1000.0, 1);
    Eval::printMetaData("agg;total_mul_time", TOTAL_MUL_TIME / 1000.0, 1);
    Eval::printMetaData("agg;total_setup_time", TOTAL_SETUP_TIME / 1000.0, 1);
    Eval::printMetaData("agg;total_mem_time", TOTAL_MEM_TIME / 1000.0, 1);
    Eval::printMetaData("agg;total_relax_time", TOTAL_RELAX_TIME / 1000.0, 1);
    Eval::printMetaData("agg;total_shifted_csrvec", TOTAL_SHIFTED_CSRVEC/ 1000.0, 1);
    Eval::printMetaData("agg;total_make_p", TOTAL_MAKE_P/ 1000.0, 1);
    Eval::printMetaData("agg;total_traspose_p", TOTAL_TRA_P/ 1000.0, 1);
    Eval::printMetaData("agg;total_makeAH_W", TOTAL_MAKEAHW_TIME/ 1000.0, 1);
    Eval::printMetaData("agg;total_matchingPairAggregation", TOTAL_MATCHINGPAIR_TIME/ 1000.0, 1);
    Eval::printMetaData("agg;total_matchingAggregation", TOTAL_MATCH_TIME/ 1000.0, 1);
    Eval::printMetaData("agg;total_OtherTime", TOTAL_OTHER_TIME/ 1000.0, 1);
    
  }

  POP_RANGE
  return hrrch;
}

