#include "hip/hip_runtime.h"
#include "datastruct/CSR.h"
#include "datastruct/scalar.h"
#include "utility/cudamacro.h"
#include "utility/memory.h"
#include "utility/mpi.h"

#include <hip/hip_runtime.h>

extern int *taskmap, *itaskmap;

template <typename T>
__inline__ void chop_array_MPI_same(int nprocs, unsigned n, unsigned* chunks, unsigned* chunkn)
{
    int i;
    int e4chunk = n / nprocs * sizeof(T);
    for (i = 0; i < nprocs - 1; i++) {
        chunkn[i] = e4chunk;
        chunks[i] = (i)*e4chunk;
    }
    chunkn[nprocs - 1] = (n * sizeof(T)) - i * e4chunk;
    chunks[i] = e4chunk * (nprocs - 1);
}

template <typename T>
__inline__ void chop_array_MPI(int nprocs, int n, int n_local, int* chunks, int* chunkn)
{
    itype ns[nprocs];
    itype tmpns[nprocs], tmpchunks[nprocs];

    // std::cerr << "Before MPI_Allgather\n";
    CHECK_MPI(
        MPI_Allgather(
            &n_local,
            sizeof(itype),
            MPI_BYTE,
            tmpns,
            sizeof(itype),
            MPI_BYTE,
            MPI_COMM_WORLD));
    // std::cerr << "After MPI_Allgather\n";

    // std::cerr << "1\n";
    int i;
    for (i = 0; i < nprocs - 1; i++) {
        ns[i] = tmpns[i];
        chunkn[i] = ns[i] * sizeof(T);
    }

    // std::cerr << "2\n";
    itype tot = 0;
    for (i = 0; i < nprocs - 1; i++) {
        tmpchunks[i] = tot;
        tot += (tmpns[itaskmap[i]] * sizeof(T));
    }

    // std::cerr << "3\n";
    for (i = 0; i < nprocs - 1; i++) {
        chunks[i] = tmpchunks[taskmap[i]];
    }

    // std::cerr << "4\n";
    chunkn[nprocs - 1] = (n * sizeof(T)) - tot;
    chunks[i] = tot;
}

vector<vtype>* aggregate_vector(vector<vtype>* u_local, itype full_n)
{
    _MPI_ENV;

    vector<vtype>* h_u_local = Vector::copyToHost(u_local);
    vector<vtype>* h_u = Vector::init<vtype>(full_n, true, false);

    // std::cerr << "Before chop_array_MPI\n";
    int chunks[nprocs], chunkn[nprocs];
    chop_array_MPI<vtype>(nprocs, full_n, u_local->n, chunks, chunkn);

    CHECK_MPI(
        MPI_Allgatherv(
            h_u_local->val,
            u_local->n * sizeof(vtype),
            MPI_BYTE,
            h_u->val,
            chunkn,
            chunks,
            MPI_BYTE,
            MPI_COMM_WORLD));

    Vector::free(h_u_local);
    return h_u;
}

vector<vtype>* aggregate_vector(vector<vtype>* u_local, itype full_n, vector<vtype>* u)
{
    _MPI_ENV;

    vector<vtype>* h_u_local = Vector::copyToHost(u_local);

    vector<vtype>* h_u = Vector::init<vtype>(full_n, true, false);

    int chunks[nprocs], chunkn[nprocs];
    chop_array_MPI<vtype>(nprocs, full_n, u_local->n, chunks, chunkn);

    CHECK_MPI(
        MPI_Allgatherv(
            h_u_local->val,
            u_local->n * sizeof(vtype),
            MPI_BYTE,
            h_u->val,
            chunkn,
            chunks,
            MPI_BYTE,
            MPI_COMM_WORLD));

    if (u == NULL) {
        u = Vector::copyToDevice(h_u);
    } else {
        CHECK_DEVICE(hipMemcpy(u->val, h_u->val, h_u->n * sizeof(vtype), hipMemcpyHostToDevice));
    }

    Vector::free(h_u_local);
    Vector::free(h_u);

    return u;
}

void aggregateFullPartialVector(vector<vtype>* u, itype local_n, itype shift)
{
    _MPI_ENV;
    // get your slice
    vtype* u_val = u->val + shift;
    itype full_n = u->n;

    vector<vtype>* h_u_local = Vector::init<vtype>(local_n, true, false);
    vector<vtype>* h_u = Vector::init<vtype>(full_n, true, false);

    // cpy slice to host
    CHECK_DEVICE(hipMemcpy(h_u_local->val, u_val, local_n * sizeof(vtype), hipMemcpyDeviceToHost));

    int chunks[nprocs], chunkn[nprocs];

    chop_array_MPI<vtype>(nprocs, u->n, local_n, chunks, chunkn);

    CHECK_MPI(
        MPI_Allgatherv(
            h_u_local->val,
            local_n * sizeof(vtype),
            MPI_BYTE,
            h_u->val,
            chunkn,
            chunks,
            MPI_BYTE,
            MPI_COMM_WORLD));

    CHECK_DEVICE(hipMemcpy(u->val, h_u->val, full_n * sizeof(vtype), hipMemcpyHostToDevice));
    Vector::free(h_u_local);
    Vector::free(h_u);
}

__global__ void _split_local(itype nstart, itype nrow, itype* Arow, vtype* Aval, itype* Acol, itype* Alocal_row, vtype* Alocal_val, itype* Alocal_col, itype* nnz)
{
    itype i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= nrow) {
        return;
    }

    itype shift = Arow[nstart];
    itype is = i + nstart;
    itype j_start = Arow[is];
    itype j_stop = Arow[is + 1];

    int j;
    Alocal_row[i] = Arow[is] - shift;
    for (j = j_start; j < j_stop; j++) {
        Alocal_val[j - shift] = Aval[j];
        Alocal_col[j - shift] = Acol[j];
    }

    if (i == nrow - 1) {
        *nnz = Arow[nrow + nstart] - shift;
        Alocal_row[nrow] = Arow[is + 1] - shift;
    }
}

CSR* split_local(CSR* A)
{
    _MPI_ENV;
    assert(A->on_the_device && A->n == A->full_n);

    itype rowsxproc = 0;
    // Split A

    int nrows[nprocs];
    rowsxproc = A->n / nprocs;
    for (itype i = 0; i < nprocs - 1; i++) {
        nrows[i] = rowsxproc;
    }
    nrows[nprocs - 1] = A->n - (rowsxproc * (nprocs - 1));

    int nstart = 0;
    for (int j = 0; j < myid; j++) {
        nstart += nrows[j];
    }

    CSR* Alocal = CSRm::init(nrows[myid], A->m, A->nnz, true, true, false, A->n, nstart);

    scalar<itype>* nnz = Scalar::init<itype>(-1, true);

    GridBlock gb = gb1d(nrows[myid], BLOCKSIZE);
    _split_local<<<gb.g, gb.b>>>(nstart, nrows[myid], A->row, A->val, A->col, Alocal->row, Alocal->val, Alocal->col, nnz->val);

    int* h_nnz = Scalar::getvalueFromDevice(nnz);
    Scalar::free(nnz);

    Alocal->nnz = *h_nnz;

    return Alocal;
}

CSR* split_matrix_mpi_host(CSR* A)
{
    _MPI_ENV;

    gstype colxproc[nprocs];
    stype rowsxproc = 0;

    if (ISMASTER) {
        assert(!A->on_the_device);
        // Split A
        rowsxproc = A->full_n / nprocs;
        for (itype i = 1; i < nprocs; i++) {
            colxproc[i - 1] = A->row[i * rowsxproc] - A->row[(i - 1) * rowsxproc];
        }
        colxproc[nprocs - 1] = A->row[A->full_n] - A->row[(nprocs - 1) * rowsxproc];
    }

    gstype n, m;
    if (ISMASTER) {
        n = A->n;
        m = A->m;
    }

    CHECK_MPI(
        MPI_Bcast(&n, sizeof(gstype), MPI_BYTE, 0, MPI_COMM_WORLD));

    CHECK_MPI(
        MPI_Bcast(&m, sizeof(gstype), MPI_BYTE, 0, MPI_COMM_WORLD));

    if ((nprocs > 1) && myid == (nprocs - 1)) {
        // compute the number of rows for the last process
        rowsxproc = n - ((n / nprocs) * (nprocs - 1));
    } else {
        // compute the number of rows for the process
        rowsxproc = n / nprocs;
    }

    gstype mycol = 0;
    // send columns numbers to each process
    CHECK_MPI(
        MPI_Scatter(
            colxproc,
            sizeof(gstype),
            MPI_BYTE,
            &mycol,
            sizeof(gstype),
            MPI_BYTE,
            0,
            MPI_COMM_WORLD));

    stype chunks[nprocs], chunkn[nprocs];
    chop_array_MPI_same<stype>(nprocs, (unsigned)n, chunks, chunkn);
    stype rows_shift = chunks[myid] / sizeof(stype);

    CSR* Alocal = CSRm::init(rowsxproc, m, (stype)mycol, true, false, false, n, rows_shift);

    // get row pointers
    CHECK_MPI(
        MPI_Scatterv(
            myid ? NULL : A->row,
            (int*)chunkn,
            (int*)chunks,
            MPI_BYTE,
            Alocal->row,
            sizeof(itype) * rowsxproc,
            MPI_BYTE,
            0,
            MPI_COMM_WORLD));
    // set the last pointer in the row array
    Alocal->row[rowsxproc] = Alocal->row[0] + mycol;

    // get columns
    for (int i = 0; i < nprocs; i++) {
        chunkn[i] = (int)(colxproc[i] * sizeof(itype));
        chunks[i] = ((i == 0) ? 0 : (chunks[i - 1] + chunkn[i - 1]));
    }

    CHECK_MPI(
        MPI_Scatterv(
            myid ? NULL : A->col,
            (int*)chunkn,
            (int*)chunks,
            MPI_BYTE,
            Alocal->col,
            sizeof(itype) * mycol,
            MPI_BYTE,
            0,
            MPI_COMM_WORLD));

    // get values
    for (int i = 0; i < nprocs; i++) {
        chunkn[i] = (int)(colxproc[i] * sizeof(vtype));
        chunks[i] = ((i == 0) ? 0 : (chunks[i - 1] + chunkn[i - 1]));
    }
    CHECK_MPI(
        MPI_Scatterv(
            myid ? NULL : A->val,
            (int*)chunkn,
            (int*)chunks,
            MPI_BYTE,
            Alocal->val,
            sizeof(vtype) * mycol,
            MPI_BYTE,
            0,
            MPI_COMM_WORLD));

    // shift row pointers
    if (myid > 0) {
        itype shift = Alocal->row[0];
        for (int i = 0; i <= Alocal->n; i++) {
            Alocal->row[i] -= shift;
        }
    }
    return Alocal;
}

CSR* split_matrix_mpi(CSR* A)
{
    CSR* Alocal = split_matrix_mpi_host(A);
    CSR* d_Alocal = CSRm::copyToDevice(Alocal);
    CSRm::free(Alocal);
    return d_Alocal;
}

CSR* join_matrix_mpi(CSR* Alocal)
{
    _MPI_ENV;

    assert(nprocs > 1);
    assert(!Alocal->on_the_device);

    itype row_ns[nprocs];

    // send rows sizes
    CHECK_MPI(
        MPI_Allgather(
            &Alocal->n,
            sizeof(itype),
            MPI_BYTE,
            row_ns,
            sizeof(itype),
            MPI_BYTE,
            MPI_COMM_WORLD));

    itype nnzs[nprocs];

    // send nnz sizes
    CHECK_MPI(
        MPI_Allgather(
            &Alocal->nnz,
            sizeof(itype),
            MPI_BYTE,
            nnzs,
            sizeof(itype),
            MPI_BYTE,
            MPI_COMM_WORLD));

    itype full_n = 0;
    itype full_nnz = 0;
    CSR* A;
    int chunkn[nprocs], chunks[nprocs];

    if (ISMASTER) {

        for (int i = 0; i < nprocs; i++) {
            full_n += row_ns[i];
            full_nnz += nnzs[i];
        }

        assert(full_n == Alocal->full_n);

        A = CSRm::init(full_n, Alocal->m, full_nnz, true, false, false, full_n, 0);

        // gather rows
        for (int i = 0; i < nprocs; i++) {
            chunkn[i] = row_ns[i] * sizeof(itype);
            chunks[i] = ((i == 0) ? 0 : (chunks[i - 1] + chunkn[i - 1]));
        }
        chunkn[nprocs - 1] += 1 * sizeof(itype);
    }

    itype rn = Alocal->n * sizeof(itype);
    if (myid == nprocs - 1) {
        rn += 1; // +1 for the last process
    }

    CHECK_MPI(
        MPI_Gatherv(
            Alocal->row,
            rn,
            MPI_BYTE,
            myid ? NULL : A->row,
            chunkn,
            chunks,
            MPI_BYTE,
            0,
            MPI_COMM_WORLD));

    if (ISMASTER) {
        /* reset the row number */
        itype rowoffset = 0;
        itype th = row_ns[0];
        int j = 0;
        for (int i = 0; i < Alocal->full_n; i++) {
            // next piece
            if (i >= th && (j < (nprocs))) {
                rowoffset += nnzs[j];
                j++;
                th += row_ns[j];
            }
            A->row[i] += rowoffset;
        }

        A->row[A->full_n] = nnzs[0];
        for (int i = 1; i < nprocs; i++) {
            A->row[A->full_n] += nnzs[i];
        }
    }
    // gather columns
    for (int i = 0; i < nprocs; i++) {
        chunkn[i] = nnzs[i] * sizeof(itype);
        chunks[i] = ((i == 0) ? 0 : (chunks[i - 1] + chunkn[i - 1]));
    }
    CHECK_MPI(
        MPI_Gatherv(
            Alocal->col,
            Alocal->nnz * sizeof(itype),
            MPI_BYTE,
            myid ? NULL : A->col,
            chunkn,
            chunks,
            MPI_BYTE,
            0,
            MPI_COMM_WORLD));

    // gather value
    for (int i = 0; i < nprocs; i++) {
        chunkn[i] = nnzs[i] * sizeof(vtype);
        chunks[i] = ((i == 0) ? 0 : (chunks[i - 1] + chunkn[i - 1]));
    }
    CHECK_MPI(
        MPI_Gatherv(
            Alocal->val,
            Alocal->nnz * sizeof(vtype),
            MPI_BYTE,
            myid ? NULL : A->val,
            chunkn,
            chunks,
            MPI_BYTE,
            0,
            MPI_COMM_WORLD));

    return A;
}

int stringCmp(const void* a, const void* b)
{
    return strcmp((const char*)a, (const char*)b);
}

void checkMatrixMPI(CSR* A, bool check_diagonal = true)
{
    _MPI_ENV;
    assert(A->on_the_device);
    CSR* h_Alocal = CSRm::copyToHost(A);
    CSR* h_Afull = join_matrix_mpi(h_Alocal);

    if (ISMASTER) {
        CSRm::checkMatrix(h_Afull, check_diagonal);
    }

    CSRm::free(h_Alocal);
    if (ISMASTER) {
        CSRm::free(h_Afull);
    }
}

bool _check_in_A(CSR* A, int i, int J)
{
    for (int j = A->row[i]; j < A->row[i + 1]; j++) {
        int c = A->col[j];
        if (c == J) {
            return true;
        }
    }
    return false;
}

void check_A_P_MPI(CSR* A_local, CSR* P_)
{
    _MPI_ENV;
    assert(A_local->on_the_device);
    CSR* h_Alocal = CSRm::copyToHost(A_local);
    CSR* A = join_matrix_mpi(h_Alocal);

    if (ISMASTER) {
        CSR* P = CSRm::copyToHost(P_);
        CSRm::checkMatrix(A);

        for (int i = 0; i < P->n; i++) {
            for (int j = P->row[i]; j < P->row[i + 1]; j++) {
                if (!_check_in_A(A, i, P->col[j])) {
                    printf("AP_ERROR %d %d\n", P->col[j], i);
                }
            }
        }
        CSRm::free(A);
        CSRm::free(P);
    }

    CSRm::free(h_Alocal);
}

CSR* broadcast_FullMatrix(CSR* A)
{
    _MPI_ENV;

    if (ISMASTER) {
        assert(!A->on_the_device);
    }

    itype n, m, nnz;
    if (ISMASTER) {
        n = A->n;
        m = A->m;
        nnz = A->nnz;
    }

    CHECK_MPI(
        MPI_Bcast(&n, sizeof(itype), MPI_BYTE, 0, MPI_COMM_WORLD));

    CHECK_MPI(
        MPI_Bcast(&m, sizeof(itype), MPI_BYTE, 0, MPI_COMM_WORLD));

    CHECK_MPI(
        MPI_Bcast(&nnz, sizeof(itype), MPI_BYTE, 0, MPI_COMM_WORLD));

    if (!ISMASTER) {
        A = CSRm::init(n, m, nnz, true, false, false, n, 0);
    }

    CHECK_MPI(
        MPI_Bcast(A->row, sizeof(itype) * (A->n + 1), MPI_BYTE, 0, MPI_COMM_WORLD));

    CHECK_MPI(
        MPI_Bcast(A->col, sizeof(itype) * A->nnz, MPI_BYTE, 0, MPI_COMM_WORLD));

    CHECK_MPI(
        MPI_Bcast(A->val, sizeof(vtype) * A->nnz, MPI_BYTE, 0, MPI_COMM_WORLD));

    CSR* d_A = CSRm::copyToDevice(A);
    CSRm::free(A);

    return d_A;
}

CSR* join_matrix_mpi_all(CSR* Alocal)
{
    _MPI_ENV;

    assert(nprocs > 1);
    assert(!Alocal->on_the_device);

    itype row_ns[nprocs];

    // send rows sizes
    CHECK_MPI(
        MPI_Allgather(
            &Alocal->n,
            sizeof(itype),
            MPI_BYTE,
            row_ns,
            sizeof(itype),
            MPI_BYTE,
            MPI_COMM_WORLD));

    itype nnzs[nprocs];

    // send nnz sizes
    CHECK_MPI(
        MPI_Allgather(
            &Alocal->nnz,
            sizeof(itype),
            MPI_BYTE,
            nnzs,
            sizeof(itype),
            MPI_BYTE,
            MPI_COMM_WORLD));
    //  if(myid==0) {
    //        for(int i=0; i<nprocs; i++){
    //		 printf("n[%d]=%d, nnzs[%d]=%d\n",i,row_ns[i],i,nnzs[i]);
    // 	 }
    // }

    itype full_n = 0;
    itype full_nnz = 0;
    CSR* A;
    int chunkn[nprocs], chunks[nprocs], tmpchunkn[nprocs], tmpchunks[nprocs];

    for (int i = 0; i < nprocs; i++) {
        full_n += row_ns[i];
        full_nnz += nnzs[i];
    }

    assert(full_n == Alocal->full_n);

    A = CSRm::init(full_n, Alocal->m, full_nnz, true, false, false, full_n, 0);

    // gather rows
    for (int i = 0; i < nprocs; i++) {
        chunkn[i] = tmpchunkn[i] = row_ns[i] * sizeof(itype);
    }
    chunkn[nprocs - 1] += sizeof(itype);

    itype tot = 0;
    for (int i = 0; i < nprocs; i++) {
        tmpchunks[i] = tot;
        tot += tmpchunkn[itaskmap[i]];
    }

    for (int i = 0; i < nprocs; i++) {
        chunks[i] = tmpchunks[taskmap[i]];
    }

    itype rn = Alocal->n * sizeof(itype);
    if (myid == nprocs - 1) {
        rn += 1; // +1 for the last process
    }

    CHECK_MPI(
        MPI_Allgatherv(
            Alocal->row,
            rn,
            MPI_BYTE,
            A->row,
            chunkn,
            chunks,
            MPI_BYTE,
            MPI_COMM_WORLD));

    itype rowoffset = 0;
    itype th = row_ns[0];
    int j = 0;
    for (int i = 0; i < Alocal->full_n; i++) {
        // next piece
        if (i >= th && (j < (nprocs))) {
            rowoffset += nnzs[taskmap[j]];
            j++;
            th += row_ns[taskmap[j]];
        }
        A->row[i] += rowoffset;
    }

    A->row[A->full_n] = nnzs[0];
    for (int i = 1; i < nprocs; i++) {
        A->row[A->full_n] += nnzs[i];
    }

    // gather columns
    for (int i = 0; i < nprocs; i++) {
        chunkn[i] = tmpchunkn[i] = nnzs[i] * sizeof(itype);
    }

    tot = 0;
    for (int i = 0; i < nprocs; i++) {
        tmpchunks[i] = tot;
        tot += tmpchunkn[itaskmap[i]];
    }

    for (int i = 0; i < nprocs; i++) {
        chunks[i] = tmpchunks[taskmap[i]];
    }
    CHECK_MPI(
        MPI_Allgatherv(
            Alocal->col,
            Alocal->nnz * sizeof(itype),
            MPI_BYTE,
            A->col,
            chunkn,
            chunks,
            MPI_BYTE,
            MPI_COMM_WORLD));

    // gather value
    for (int i = 0; i < nprocs; i++) {
        chunkn[i] = chunkn[i] * (sizeof(vtype) / sizeof(itype));
        chunks[i] = chunks[i] * (sizeof(vtype) / sizeof(itype));
    }

    CHECK_MPI(
        MPI_Allgatherv(
            Alocal->val,
            Alocal->nnz * sizeof(vtype),
            MPI_BYTE,
            A->val,
            chunkn,
            chunks,
            MPI_BYTE,
            MPI_COMM_WORLD));

    return A;
}
