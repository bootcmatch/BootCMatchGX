#include "utility/handles.h"

handles* Handles::init()
{

    handles* h = (handles*)malloc(sizeof(handles));
    CHECK_HOST(h);

    CHECK_CUBLAS(hipblasCreate(&(h->cublas_h)));

    CHECK_DEVICE(hipStreamCreate(&(h->stream1)));
    CHECK_DEVICE(hipStreamCreate(&(h->stream2)));
    CHECK_DEVICE(hipStreamCreate(&(h->stream3)));
    CHECK_DEVICE(hipStreamCreate(&(h->stream4)));
    CHECK_DEVICE(hipStreamCreate(&(h->stream_free)));

    return h;
}

void Handles::free(handles* h)
{
    CHECK_CUBLAS(hipblasDestroy(h->cublas_h));

    CHECK_DEVICE(hipStreamDestroy(h->stream1));
    CHECK_DEVICE(hipStreamDestroy(h->stream2));
    CHECK_DEVICE(hipStreamDestroy(h->stream3));
    CHECK_DEVICE(hipStreamDestroy(h->stream4));
    CHECK_DEVICE(hipStreamDestroy(h->stream_free));

    std::free(h);
}
