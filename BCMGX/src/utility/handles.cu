#include "utility/handles.h"
#include "utility/memory.h"

handles* Handles::init()
{
    handles* h = MALLOC(handles, 1);
    CHECK_HOST(h);

    CHECK_CUBLAS(hipblasCreate(&(h->cublas_h)));

    CHECK_DEVICE(hipStreamCreate(&(h->stream1)));
    CHECK_DEVICE(hipStreamCreate(&(h->stream2)));
    CHECK_DEVICE(hipStreamCreate(&(h->stream3)));
    CHECK_DEVICE(hipStreamCreate(&(h->stream4)));
    CHECK_DEVICE(hipStreamCreate(&(h->stream_free)));

    return h;
}

void Handles::free(handles* h)
{
    CHECK_CUBLAS(hipblasDestroy(h->cublas_h));

    CHECK_DEVICE(hipStreamDestroy(h->stream1));
    CHECK_DEVICE(hipStreamDestroy(h->stream2));
    CHECK_DEVICE(hipStreamDestroy(h->stream3));
    CHECK_DEVICE(hipStreamDestroy(h->stream4));
    CHECK_DEVICE(hipStreamDestroy(h->stream_free));

    FREE(h);
}
