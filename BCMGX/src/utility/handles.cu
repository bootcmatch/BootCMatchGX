#include "utility/handles.h"

  handles* Handles::init(){

    handles *h = (handles*) malloc(sizeof(handles));
    CHECK_HOST(h);

    CHECK_CUSPARSE( hipsparseCreate(&(h->cusparse_h0)) );
    CHECK_CUSPARSE( hipsparseCreate(&(h->cusparse_h1)) );

    CHECK_CUBLAS( hipblasCreate(&(h->cublas_h)) );

    CHECK_DEVICE( hipStreamCreate(&(h->stream1)) );
    CHECK_DEVICE( hipStreamCreate(&(h->stream2)) );
    CHECK_DEVICE( hipStreamCreate(&(h->stream3)) );
    CHECK_DEVICE( hipStreamCreate(&(h->stream4)) );

    CHECK_CUSPARSE( hipsparseSetStream(h->cusparse_h1, h->stream1) );

    return h;
  }

  void Handles::free(handles *h){
    CHECK_CUSPARSE( hipsparseDestroy(h->cusparse_h0) );
    CHECK_CUSPARSE( hipsparseDestroy(h->cusparse_h1) );

    CHECK_CUBLAS( hipblasDestroy(h->cublas_h) );

    CHECK_DEVICE( hipStreamDestroy(h->stream1) );
    CHECK_DEVICE( hipStreamDestroy(h->stream2) );
    CHECK_DEVICE( hipStreamDestroy(h->stream3) );
    CHECK_DEVICE( hipStreamDestroy(h->stream4) );
    std::free(h);
  }
