#include "scalar.h"

#include "utility/function_cnt.h"
#include "utility/utils.h"

namespace Scalar {

template <typename T>
scalar<T>* init(T val, bool on_the_device)
{
    scalar<T>* v = NULL;
    // on the host
    v = (scalar<T>*)Malloc(sizeof(scalar<T>));
    CHECK_HOST(v);

    v->on_the_device = on_the_device;

    if (on_the_device) {
        // on the device
        hipError_t err;
        cudaMalloc_CNT
            err
            = hipMalloc((void**)&v->val, sizeof(T));
        CHECK_DEVICE(err);

        err = hipMemcpy(v->val, &val, sizeof(T), hipMemcpyHostToDevice);
        CHECK_DEVICE(err);
    } else {
        // on the host
        v->val = (T*)Malloc(sizeof(T));
        CHECK_HOST(v->val);
        v->val[0] = val;
    }
    return v;
}

template <typename T>
scalar<T>* copyToDevice(scalar<T>* v)
{

    assert(!v->on_the_device);

    // alocate scalar on the device memory
    scalar<T>* v_d = init<T>(0, true);

    hipError_t err = hipMemcpy(v_d->val, v->val, sizeof(T), hipMemcpyHostToDevice);
    CHECK_DEVICE(err);

    return v_d;
}

template <typename T>
scalar<T>* copyToHost(scalar<T>* v_d)
{

    assert(v_d->on_the_device);

    // alocate scalar on the host memory
    scalar<T>* v = init<T>(0, false);

    hipError_t err;

    err = hipMemcpy(v->val, v_d->val, sizeof(T), hipMemcpyDeviceToHost);
    CHECK_DEVICE(err);

    return v;
}

template <typename T>
void free(scalar<T>* v)
{
    if (v->on_the_device) {
        hipError_t err;
        err = hipFree(v->val);
        CHECK_DEVICE(err);
    } else {
        std::free(v->val);
    }
    std::free(v);
}

template <typename T>
void print(scalar<T>* v)
{
    scalar<T>* v_;

    if (v->on_the_device) {
        v_ = Scalar::copyToHost<T>(v);
    } else {
        v_ = v;
    }

    std::cout << v_->val[0] << "\n";

    if (v->on_the_device) {
        Scalar::free<T>(v_);
    }
}

// like copyToHost but with less overhead
template <typename T>
T* getvalueFromDevice(scalar<T>* v_d)
{

    assert(v_d->on_the_device);

    // alocate scalar on the host memory
    T* v = (int*)Malloc(sizeof(T));
    CHECK_HOST(v);

    hipError_t err;

    err = hipMemcpy(v, v_d->val, sizeof(T), hipMemcpyDeviceToHost);
    CHECK_DEVICE(err);

    return v;
}
}

namespace Scalar {
template scalar<itype>* init<itype>(itype, bool);
template scalar<vtype>* init<vtype>(vtype, bool);

template scalar<itype>* copyToDevice<itype>(scalar<itype>*);
template scalar<vtype>* copyToDevice<vtype>(scalar<vtype>*);

template scalar<itype>* copyToHost<itype>(scalar<itype>*);
template scalar<vtype>* copyToHost<vtype>(scalar<vtype>*);

template void free<itype>(scalar<itype>*);
template void free<vtype>(scalar<vtype>*);

template void print<itype>(scalar<itype>*);
template void print<vtype>(scalar<vtype>*);

template itype* getvalueFromDevice<itype>(scalar<itype>*);
}
