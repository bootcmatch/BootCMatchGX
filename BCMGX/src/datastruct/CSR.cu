#include "hip/hip_runtime.h"
#include "CSR.h"

#include "halo_communication/extern2.h"
#include "halo_communication/halo_communication.h"

#include "datastruct/matrixItem.h"
#include "utility/MatrixItemSender.h"
#include "utility/cuCompactorXT.cuh"
#include "utility/cudamacro.h"
#include "utility/devicePartition.h"
#include "utility/devicePrefixSum.h"
#include "utility/deviceSort.h"
#include "utility/hostPartition.h"
#include "utility/hostSort.h"
#include "utility/profiling.h"

#include <hipcub/hipcub.hpp>
#include <string.h>

#define MAXMATRIXFILENAME 256

int CSRm::choose_mini_warp_size(CSR* A)
{
    int density = A->nnz / A->n;

    if (density < MINI_WARP_THRESHOLD_2) {
        return 2;
    } else if (density < MINI_WARP_THRESHOLD_4) {
        return 4;
    } else if (density < MINI_WARP_THRESHOLD_8) {
        return 8;
    } else if (density < MINI_WARP_THRESHOLD_16) {
        return 16;
    } else {
        return 32;
    }
}

CSR* CSRm::init(stype n, gstype m, stype nnz, bool allocate_mem, bool on_the_device, bool is_symmetric, gstype full_n, gstype row_shift)
{
    // ---------- Pico ----------
    if (n <= 0 || m <= 0 || nnz <= 0) {
        fprintf(stderr, "error in CSRm::init:\n\tint  n: %d  m: %lu  nnz: %d\n", n, m, nnz);
    }
    assert(n > 0);
    assert(m > 0);
    assert(nnz > 0);
    // --------------------------

    CSR* A = NULL;

    // on the host
    A = MALLOC(CSR, 1, true);

    A->nnz = nnz;
    A->n = n;
    A->m = m;
    A->full_m = m;

    A->on_the_device = on_the_device;
    A->is_symmetric = false;
    A->custom_alloced = false;

    A->full_n = full_n;
    A->row_shift = row_shift;

    A->rows_to_get = NULL;

    A->shrinked_flag = false;
    A->shrinked_col = NULL;
    A->shrinked_m = m;
    A->halo.init = false;
    A->col_shifted = 0;

    A->post_local = 0;
    A->bitcolsize = 0;
    A->bitcol = NULL;

    if (allocate_mem) {
        if (on_the_device) {
            // on the device
            A->val = CUDA_MALLOC(vtype, nnz, true);
            A->col = CUDA_MALLOC(itype, nnz, true);
            A->row = CUDA_MALLOC(itype, n + 1, true);
        } else {
            // on the host
            A->val = MALLOC(vtype, nnz, true);
            A->col = MALLOC(itype, nnz, true);
            A->row = MALLOC(itype, n + 1, true);
        }
    }

    return A;
}

void CSRm::printMM(CSR* A, char* name, bool appendMyIdAndNprocs)
{
    _MPI_ENV;
    CSR* A_ = NULL;
    if (A->on_the_device) {
        A_ = CSRm::copyToHost(A);
    } else {
        A_ = A;
    }

    char localname[MAXMATRIXFILENAME];
    if (appendMyIdAndNprocs) {
        snprintf(localname, sizeof(localname), "%s_%d_%d", name, myid, nprocs);
    } else {
        snprintf(localname, sizeof(localname), "%s", name);
    }
    FILE* fp = fopen(localname, "w");
    if (fp == NULL) {
        fprintf(stderr, "Could not open %s", localname);
        exit(1);
    }
    fprintf(fp, "%%%%MatrixMarket matrix coordinate real general\n");
    fprintf(fp, "%d %lu %d "
                "%ld %ld %ld\n",
        A_->n, A_->m, A_->nnz,
        A_->row_shift, A_->full_n, A_->col_shifted);
    for (int i = 0; i < A_->n; i++) {
        for (int j = A_->row[i]; j < A_->row[i + 1]; j++) {
            fprintf(fp, "%lu %ld %lf\n",
                i + 1 + A_->row_shift,
                A_->col[j] + 1 - A_->col_shifted,
                A_->val[j]);
        }
    }
    fclose(fp);

    if (A->on_the_device) {
        CSRm::free(A_);
    }
}

void CSRm::print(CSR* A, int type, int limit, FILE* fp)
{
    CSR* A_ = NULL;

    if (A->on_the_device) {
        A_ = CSRm::copyToHost(A);
    } else {
        A_ = A;
    }

    switch (type) {
    case 0:
        fprintf(fp, "ROW: %d (%lu)\n\t", A_->n, A_->full_n);
        if (limit == 0) {
            limit = A_->full_n + 1;
        }
        for (int i = 0; i < limit; i++) {
            fprintf(fp, "%3d ", A_->row[i]);
        }
        break;
    case 1:
        fprintf(fp, "COL:\n");
        if (limit == 0) {
            limit = A_->nnz;
        }
        for (int i = 0; i < limit; i++) {
            fprintf(fp, "%d\n", A_->col[i]);
        }
        break;
    case 2:
        fprintf(fp, "VAL:\n");
        if (limit == 0) {
            limit = A_->nnz;
        }
        for (int i = 0; i < limit; i++) {
            fprintf(fp, "%14.12g\n", A_->val[i]);
        }
        break;
    case 3:
        fprintf(fp, "MATRIX_Form:\n");
        for (int i = 0; i < A_->n; i++) {
            fprintf(fp, "\t");
            for (int j = 0; j < A_->m; j++) {
                int flag = 0, temp = A_->row[i];
                for (temp = A_->row[i]; flag == 0 && (i != (A_->n) - 1 ? temp < (A_->row[i + 1]) : temp < A_->nnz); temp++) {
                    if (A_->col[temp] == j) {
                        fprintf(fp, "%g ", A_->val[temp]);
                        flag = 1;
                    }
                }
                if (flag == 0) {
                    fprintf(fp, "%g ", 0.0);
                }
            }
            fprintf(fp, "\n");
        }
        break;
    case 4:
        fprintf(fp, "boolMATRIX_Form:\n");
        for (int i = 0; i < A_->n; i++) {
            fprintf(fp, "\t");
            for (int j = 0; j < A_->m; j++) {
                if (j % 32 == 0) {
                    fprintf(fp, "| ");
                }
                int flag = 0, temp = A_->row[i];
                for (temp = A_->row[i]; flag == 0 && (i != (A_->n) - 1 ? temp < (A_->row[i + 1]) : temp < A_->nnz); temp++) {
                    if (A_->col[temp] == j) {
                        fprintf(fp, "\033[0;31mX\033[0m ");
                        flag = 1;
                    }
                }
                if (flag == 0) {
                    fprintf(fp, "O ");
                }
            }
            fprintf(fp, "\n");
        }
        break;
    case 5:
        fprintf(fp, "SHRINKED COL:\n");
        if (limit == 0) {
            limit = A_->shrinked_m;
        }
        for (int i = 0; i < limit; i++) {
            fprintf(fp, "%d\n", A_->shrinked_col[i]);
        }
        break;
    }
    fprintf(fp, "\n\n");

    if (A->on_the_device) {
        CSRm::free(A_);
    }
}

void CSRm::free_rows_to_get(CSR* A)
{
    if (A->rows_to_get != NULL) {
        FREE(A->rows_to_get->rcvprow);
        FREE(A->rows_to_get->whichprow);
        FREE(A->rows_to_get->rcvpcolxrow);
        FREE(A->rows_to_get->scounts);
        FREE(A->rows_to_get->displs);
        FREE(A->rows_to_get->displr);
        FREE(A->rows_to_get->rcounts2);
        FREE(A->rows_to_get->scounts2);
        FREE(A->rows_to_get->displs2);
        FREE(A->rows_to_get->displr2);
        FREE(A->rows_to_get->rcvcntp);
        FREE(A->rows_to_get->P_n_per_process);
        if (A->rows_to_get->nnz_per_row_shift != NULL) {
            Vector::free(A->rows_to_get->nnz_per_row_shift);
        }
        FREE(A->rows_to_get);
    }
    A->rows_to_get = NULL;
}

void CSRm::free(CSR* A)
{
    if (A->on_the_device) {
        CUDA_FREE(A->val);
        CUDA_FREE(A->col);
        CUDA_FREE(A->row);
        CUDA_FREE(A->shrinked_col);
    } else {
        FREE(A->val);
        FREE(A->col);
        FREE(A->row);
    }
    if (A->rows_to_get != NULL) {
        FREE(A->rows_to_get->rcvprow);
        FREE(A->rows_to_get->whichprow);
        FREE(A->rows_to_get->rcvpcolxrow);
        FREE(A->rows_to_get->scounts);
        FREE(A->rows_to_get->displs);
        FREE(A->rows_to_get->displr);
        FREE(A->rows_to_get->rcounts2);
        FREE(A->rows_to_get->scounts2);
        FREE(A->rows_to_get->displs2);
        FREE(A->rows_to_get->displr2);
        FREE(A->rows_to_get->rcvcntp);
        FREE(A->rows_to_get->P_n_per_process);
        if (A->rows_to_get->nnz_per_row_shift != NULL) {
            Vector::free(A->rows_to_get->nnz_per_row_shift);
        }
        FREE(A->rows_to_get);
        A->rows_to_get = NULL;
    }

    CUDA_FREE(A->bitcol);
    A->bitcol = NULL;

    if (A->halo.init == true) {
        // Free the halo_info halo halo_info halo;
        Vector::free(A->halo.to_receive);
        Vector::free(A->halo.to_receive_d);
        FREE(A->halo.to_receive_counts);
        FREE(A->halo.to_receive_spls);
        CUDA_FREE_HOST(A->halo.what_to_receive);
        CUDA_FREE(A->halo.what_to_receive_d);
        Vector::free(A->halo.to_send);
        Vector::free(A->halo.to_send_d);
        FREE(A->halo.to_send_counts);
        FREE(A->halo.to_send_spls);
        CUDA_FREE_HOST(A->halo.what_to_send);
        CUDA_FREE(A->halo.what_to_send_d);
        A->halo.init = false;
    }

    Vector::free(A->os.loc_rows);
    A->os.loc_rows = NULL;
    Vector::free(A->os.needy_rows);
    A->os.needy_rows = NULL;
}

void shift_cpucol(itype* Arow, itype* Acol, unsigned int n, stype row_shift)
{
    for (unsigned int i = 0; i < n; i++) {
        for (unsigned int j = Arow[i]; j < Arow[i + 1]; j++) {
            Acol[j] += row_shift;
        }
    }
}

CSR* CSRm::copyToDevice(CSR* A)
{
    assert(!A->on_the_device);

    itype n, nnz;
    gstype m;
    n = A->n;
    m = A->m;

    nnz = A->nnz;

    // allocate CSR matrix on the device memory
    CSR* A_d = CSRm::init(n, m, nnz, true, true, A->is_symmetric, A->full_n, A->row_shift);
    A_d->full_m = A->full_m;
    A_d->col_shifted = A->col_shifted;

    hipError_t err;
    err = hipMemcpy(A_d->val, A->val, nnz * sizeof(vtype), hipMemcpyHostToDevice);
    CHECK_DEVICE(err);
    err = hipMemcpy(A_d->row, A->row, (n + 1) * sizeof(itype), hipMemcpyHostToDevice);
    CHECK_DEVICE(err);
    err = hipMemcpy(A_d->col, A->col, nnz * sizeof(itype), hipMemcpyHostToDevice);
    CHECK_DEVICE(err);

    return A_d;
}

CSR* CSRm::copyToHost(CSR* A_d)
{
    assert(A_d->on_the_device);

    itype n, m, nnz;

    n = A_d->n;
    m = A_d->m;

    nnz = A_d->nnz;

    // allocate CSR matrix on the device memory
    CSR* A = CSRm::init(n, m, nnz, true, false, A_d->is_symmetric, A_d->full_n, A_d->row_shift);
    A->full_m = A_d->full_m;
    A->col_shifted = A_d->col_shifted;

    hipError_t err;

    assert(A->val);
    assert(A_d->val);
    err = hipMemcpy(A->val, A_d->val, nnz * sizeof(vtype), hipMemcpyDeviceToHost);
    CHECK_DEVICE(err);

    assert(A->row);
    assert(A_d->row);
    err = hipMemcpy(A->row, A_d->row, (n + 1) * sizeof(itype), hipMemcpyDeviceToHost);
    CHECK_DEVICE(err);

    assert(A->col);
    assert(A_d->col);
    err = hipMemcpy(A->col, A_d->col, nnz * sizeof(itype), hipMemcpyDeviceToHost);
    CHECK_DEVICE(err);

    if (A_d->shrinked_m && A_d->shrinked_col) {
        A->shrinked_col = MALLOC(itype, A_d->shrinked_m, false);
        err = hipMemcpy(A->shrinked_col, A_d->shrinked_col, A_d->shrinked_m * sizeof(itype), hipMemcpyDeviceToHost);
        CHECK_DEVICE(err);
    } else {
        A->shrinked_col = NULL;
    }

    return A;
}

__global__ void _shift_cols(itype n, itype* col, gsstype shift)
{
    itype i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= n) {
        return;
    }
    gsstype scratch = col[i];
    scratch += shift;
    col[i] = scratch;
}

void CSRm::shift_cols(CSR* A, gsstype shift)
{
    assert(A->on_the_device);
    GridBlock gb = gb1d(A->nnz, BLOCKSIZE);
    _shift_cols<<<gb.g, gb.b>>>(A->nnz, A->col, shift);
}

void CSRm::shift_cols_nogpu(CSR* A, gsstype shift)
{
    assert(!A->on_the_device);

    itype n = A->nnz;
    for (itype i = 0; i < n; i++) {
        gsstype scratch = A->col[i];
        scratch += shift;
        A->col[i] = scratch;
    }
}

__global__ void _prepare_column_ptr(stype A_nrows, itype* A_row, itype* A_col, itype* T_row)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    while (tid < A_nrows) {
        itype start_idx = A_row[tid];
        itype end_idx = A_row[tid + 1];
        // Count the number of nnz per column
        for (itype i = start_idx; i < end_idx; i++) {
            atomicAdd(&T_row[A_col[i] + 1], 1);
        }
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void _write_row_indices(stype A_nrows, itype* A_row, itype* T_col)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    while (tid < A_nrows) {
        itype start_idx = A_row[tid];
        itype end_idx = A_row[tid + 1];
        for (itype i = start_idx; i < end_idx; i++) {
            T_col[i] = tid;
        }
        tid += blockDim.x * gridDim.x;
    }
}

template <int OP_TYPE>
__global__ void CSRm::_CSR_vector_mul_mini_warp(itype n, int MINI_WARP_SIZE, vtype alpha, vtype beta, vtype* A_val, itype* A_row, itype* A_col, vtype* x, vtype* y)
{
    itype tid = blockDim.x * blockIdx.x + threadIdx.x;

    int warp = tid / MINI_WARP_SIZE;

    if (warp >= n) {
        return;
    }

    int lane = tid % MINI_WARP_SIZE;
    int mask_id = (tid % FULL_WARP) / MINI_WARP_SIZE;
    int warp_mask = getMaskByWarpID(MINI_WARP_SIZE, mask_id);

    vtype T_i = 0.;

    for (int j = A_row[warp] + lane; j < A_row[warp + 1]; j += MINI_WARP_SIZE) {
        if (OP_TYPE == 0) {
            T_i += (alpha * A_val[j]) * __ldg(&x[A_col[j]]);
        } else if (OP_TYPE == 1) {
            T_i += A_val[j] * __ldg(&x[A_col[j]]);
        } else if (OP_TYPE == 2) {
            T_i += -A_val[j] * __ldg(&x[A_col[j]]);
        }
    }

    for (int k = MINI_WARP_SIZE >> 1; k > 0; k = k >> 1) {
        T_i += __shfl_down_sync(warp_mask, T_i, k);
    }

    if (lane == 0) {
        if (OP_TYPE == 0) {
            y[warp] = T_i + (beta * y[warp]);
        } else if (OP_TYPE == 1) {
            y[warp] = T_i;
        } else if (OP_TYPE == 2) {
            y[warp] = T_i + y[warp];
        }
    }
}

template <int OP_TYPE>
__global__ void CSRm::_CSR_vector_mul_mini_warp_indirect(itype n, itype* rows, unsigned offset, int MINI_WARP_SIZE, vtype alpha, vtype beta, vtype* A_val, itype* A_row, itype* A_col, vtype* x, vtype* y)
{
    itype tid = blockDim.x * blockIdx.x + threadIdx.x;

    int warp = tid / MINI_WARP_SIZE;

    if (warp >= n) {
        return;
    }

    warp = rows[warp];

    int lane = tid % MINI_WARP_SIZE;
    int mask_id = (tid % FULL_WARP) / MINI_WARP_SIZE;
    int warp_mask = getMaskByWarpID(MINI_WARP_SIZE, mask_id);

    vtype T_i = 0.;

    for (int j = A_row[warp] + lane; j < A_row[warp + 1]; j += MINI_WARP_SIZE) {
        if (OP_TYPE == 0) {
            T_i += (alpha * A_val[j]) * __ldg(&x[A_col[j] - offset]);
        } else if (OP_TYPE == 1) {
            T_i += A_val[j] * __ldg(&x[A_col[j] - offset]);
        } else if (OP_TYPE == 2) {
            T_i += -A_val[j] * __ldg(&x[A_col[j] - offset]);
        }
    }

    for (int k = MINI_WARP_SIZE >> 1; k > 0; k = k >> 1) {
        T_i += __shfl_down_sync(warp_mask, T_i, k);
    }

    if (lane == 0) {
        if (OP_TYPE == 0) {
            y[warp] = T_i + (beta * y[warp]);
        } else if (OP_TYPE == 1) {
            y[warp] = T_i;
        } else if (OP_TYPE == 2) {
            y[warp] = T_i + y[warp];
        }
    }
}

template <int OP_TYPE>
__global__ void CSRm::_CSR_scale_mini_warp(itype n, int MINI_WARP_SIZE, vtype alpha, vtype beta, vtype* A_val, itype* A_row, itype* A_col, vtype* x, vtype* y)
{
    itype tid = blockDim.x * blockIdx.x + threadIdx.x;

    int warp = tid / MINI_WARP_SIZE;

    if (warp >= n) {
        return;
    }

    int lane = tid % MINI_WARP_SIZE;
    int mask_id = (tid % FULL_WARP) / MINI_WARP_SIZE;
    int warp_mask = getMaskByWarpID(MINI_WARP_SIZE, mask_id);

    vtype T_i = 0.;

    for (int j = A_row[warp] + lane; j < A_row[warp + 1]; j += MINI_WARP_SIZE) {
        if (OP_TYPE == 0) {
            T_i = (alpha * A_val[j]) * __ldg(&x[A_col[j]]);
        } else if (OP_TYPE == 1) {
            T_i = A_val[j] * __ldg(&x[A_col[j]]);
        } else if (OP_TYPE == 2) {
            T_i = -A_val[j] * __ldg(&x[A_col[j]]);
        }
        y[j] = T_i * __ldg(&x[warp]);
    }
}

__global__ void CSRm::_CSR_vector_mul_mini_indexed_warp(itype n, int MINI_WARP_SIZE, vtype alpha, vtype beta, vtype* A_val, itype* A_row, itype* A_col, vtype* x, vtype* y, itype* to_comp, itype shift, int op_type)
{
    itype tid = blockDim.x * blockIdx.x + threadIdx.x;

    int warp = tid / MINI_WARP_SIZE;
    if (warp >= n) {
        return;
    }

    int target = warp;
    int lane = tid % MINI_WARP_SIZE;
    int mask_id = (tid % FULL_WARP) / MINI_WARP_SIZE;
    int warp_mask = getMaskByWarpID(MINI_WARP_SIZE, mask_id);

    vtype T_i = 0.;
    warp = to_comp[warp] /* -shift */;
    if (op_type == 0) {
        for (int j = A_row[warp] + lane; j < A_row[warp + 1]; j += MINI_WARP_SIZE) {
            T_i += (alpha * A_val[j]) * __ldg(&x[A_col[j]]);
        }
    }
    if (op_type == 1) {
        for (int j = A_row[warp] + lane; j < A_row[warp + 1]; j += MINI_WARP_SIZE) {
            T_i += A_val[j] * __ldg(&x[A_col[j]]);
        }
    }
    if (op_type == 2) {
        for (int j = A_row[warp] + lane; j < A_row[warp + 1]; j += MINI_WARP_SIZE) {
            T_i += -A_val[j] * __ldg(&x[A_col[j]]);
        }
    }

    for (int k = MINI_WARP_SIZE >> 1; k > 0; k = k >> 1) {
        T_i += __shfl_down_sync(warp_mask, T_i, k);
    }

    if (lane == 0) {
        if (op_type == 0) {
            y[target] = T_i + (beta * y[warp]);
        } else if (op_type == 1) {
            y[target] = T_i;
        } else if (op_type == 2) {
            y[target] = T_i + y[warp];
        }
    }
}

vector<vtype>* CSRm::CSRVector_product_adaptive_miniwarp(CSR* A, vector<vtype>* x, vector<vtype>* y, vtype alpha, vtype beta)
{
    BEGIN_PROF(__FUNCTION__);

    itype n = A->n;

    int density = A->nnz / A->n;

    int min_w_size;

    if (density < MINI_WARP_THRESHOLD_2) {
        min_w_size = 2;
    } else if (density < MINI_WARP_THRESHOLD_4) {
        min_w_size = 4;
    } else if (density < MINI_WARP_THRESHOLD_8) {
        min_w_size = 4;
    } else {
        min_w_size = 16;
    }

    if (y == NULL) {
        assert(beta == 0.);
        y = Vector::init<vtype>(n, true, true); // OK perchè vettore di output
    }

    GridBlock gb = gb1d(n, BLOCKSIZE, true, min_w_size);

    if (alpha == 1. && beta == 0.) {
        CSRm::_CSR_vector_mul_mini_warp<1><<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val);
    } else if (alpha == -1. && beta == 1.) {
        CSRm::_CSR_vector_mul_mini_warp<2><<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val);
    } else {
        CSRm::_CSR_vector_mul_mini_warp<0><<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val);
    }
    hipDeviceSynchronize();

    END_PROF(__FUNCTION__);
    return y;
}

vector<vtype>* CSRm::CSRVector_product_adaptive_indirect_row_miniwarp(CSR* A, vector<vtype>* x, vector<vtype>* y, itype n, itype* rows, hipStream_t stream, unsigned int offset, vtype alpha, vtype beta)
{

    int density = A->nnz / A->n;

    int min_w_size;

    if (density < MINI_WARP_THRESHOLD_2) {
        min_w_size = 2;
    } else if (density < MINI_WARP_THRESHOLD_4) {
        min_w_size = 4;
    } else if (density < MINI_WARP_THRESHOLD_8) {
        min_w_size = 4;
    } else {
        min_w_size = 16;
    }

    if (y == NULL) {
        assert(beta == 0.);
        y = Vector::init<vtype>(n, true, true); // OK perchè vettore di output
    }

    GridBlock gb = gb1d(n, BLOCKSIZE, true, min_w_size);

    if (alpha == 1. && beta == 0.) {
        CSRm::_CSR_vector_mul_mini_warp_indirect<1><<<gb.g, gb.b, 0, stream>>>(n, rows, offset, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val);
    } else if (alpha == -1. && beta == 1.) {
        CSRm::_CSR_vector_mul_mini_warp_indirect<2><<<gb.g, gb.b, 0, stream>>>(n, rows, offset, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val);
    } else {
        CSRm::_CSR_vector_mul_mini_warp_indirect<0><<<gb.g, gb.b, 0, stream>>>(n, rows, offset, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val);
    }
    hipStreamSynchronize(stream);
    return y;
}

vector<vtype>* CSRm::CSRscale_adaptive_miniwarp(CSR* A, vector<vtype>* x, vector<vtype>* y, vtype alpha, vtype beta)
{
    itype n = A->n;

    int density = A->nnz / A->n;

    int min_w_size;

    if (density < MINI_WARP_THRESHOLD_2) {
        min_w_size = 2;
    } else if (density < MINI_WARP_THRESHOLD_4) {
        min_w_size = 4;
    } else if (density < MINI_WARP_THRESHOLD_8) {
        min_w_size = 4;
    } else {
        min_w_size = 16;
    }

    if (y == NULL) {
        assert(beta == 0.);
        y = Vector::init<vtype>(n, true, true); // OK perchè vettore di output
    }

    GridBlock gb = gb1d(n, BLOCKSIZE, true, min_w_size);

    if (alpha == 1. && beta == 0.) {
        CSRm::_CSR_scale_mini_warp<1><<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val);
    } else if (alpha == -1. && beta == 1.) {
        CSRm::_CSR_scale_mini_warp<2><<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val);
    } else {
        CSRm::_CSR_scale_mini_warp<0><<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val);
    }
    hipDeviceSynchronize();
    return y;
}

__global__ void _vector_sync(vtype* local_x, itype local_n, vtype* what_to_receive_d, itype receive_n, itype post_local, vtype* x, itype x_n)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id < x_n) {
        if (id < post_local) {
            x[id] = what_to_receive_d[id];
        } else {
            if (id < post_local + local_n) {
                x[id] = local_x[id - post_local];
            } else {
                x[id] = what_to_receive_d[id - local_n];
            }
        }
    }
}

vector<vtype>* CSRm::CSRVector_product_adaptive_miniwarp_new(CSR* A, vector<vtype>* local_x, vector<vtype>* w, vtype alpha, vtype beta)
{
    BEGIN_PROF(__FUNCTION__);

    _MPI_ENV;

    if (nprocs == 1) {
        vector<vtype>* w_ = NULL;
        if (w == NULL) {
            w_ = Vector::init<vtype>(A->n, true, true);
            Vector::fillWithValue(w_, 0.);
        } else {
            w_ = w;
        }
        CSRm::CSRVector_product_adaptive_miniwarp(A, local_x, w_, alpha, beta);
        END_PROF(__FUNCTION__);
        return (w_);
    }

    assert(A->shrinked_flag == 1);

    CSR* A_ = CSRm::init(A->n, (gstype)A->shrinked_m, A->nnz, false, A->on_the_device, A->is_symmetric, A->full_n, A->row_shift);
    A_->row = A->row;
    A_->val = A->val;
    A_->col = A->shrinked_col;

    // ----------------------------------------- temp check -----------------------------------------
    //  assert( A->halo.to_receive_n + local_x->n == A_->m ); /* Massimo March 13 2024. To fix problem with Xtent */
    // ----------------------------------------------------------------------------------------------
    int post_local = A->post_local;

    vector<vtype>* x_ = NULL;
    if (A->halo.to_receive_n > 0) {
        x_ = Vector::init<vtype>(A_->m, false, true);
        if (A_->m > xsize) {
            CUDA_FREE(xvalstat);
            xsize = A_->m;
            xvalstat = CUDA_MALLOC(vtype, xsize, true);
        }
        x_->val = xvalstat;
        GridBlock gb = gb1d(A_->m, BLOCKSIZE);
        _vector_sync<<<gb.g, gb.b>>>(local_x->val, A->n, A->halo.what_to_receive_d, A->halo.to_receive_d->n, post_local, x_->val, x_->n);
    } else {
        x_ = local_x;
    }

    vector<vtype>* w_ = NULL;
    if (w == NULL) {
        w_ = Vector::init<vtype>(A->n, true, true);
        Vector::fillWithValue(w_, 1.);
    } else {
        w_ = w;
    }
    CSRm::CSRVector_product_adaptive_miniwarp(A_, x_, w_, alpha, beta);

    // --------------------------------------- print -----------------------------------------
    //   vector<vtype> *what_to_receive_d = Vector::init<vtype>(A->halo.to_receive_n, false, true);
    //   what_to_receive_d->val = A->halo.what_to_receive_d;
    //
    //   PICO_PRINT(  \
    //     fprintf(fp, "A->halo:\n\tto_receive: "); Vector::print(A->halo.to_receive, -1, fp); \
    //     fprintf(fp, "\tto_send: "); Vector::print(A->halo.to_send, -1, fp); \
    //     fprintf(fp, "post_local = %d\n", post_local); \
    //     fprintf(fp, "what_to_receive_d: "); Vector::print(what_to_receive_d, -1, fp); \
    //     fprintf(fp, "local_x: "); Vector::print(local_x, -1, fp); \
    //     fprintf(fp, "x_: "); Vector::print(x_, -1, fp); \
    //   )
    //
    //   FREE(what_to_receive_d);
    // ---------------------------------------------------------------------------------------

    if (A->halo.to_receive_n > 0) {
        FREE(x_);
    }
    A_->col = NULL;
    A_->row = NULL;
    A_->val = NULL;
    FREE(A_);

    END_PROF(__FUNCTION__);
    return (w_);
}

#define SYNCSOL_TAG 4321
#define MAXNTASKS 4096
#define USESTREAM 1

vector<vtype>* CSRm::CSRVector_product_adaptive_miniwarp_witho(CSR* A, vector<vtype>* local_x, vector<vtype>* w, vtype alpha, vtype beta)
{
    BEGIN_PROF(__FUNCTION__);

    _MPI_ENV;

    if (nprocs == 1) {
        vector<vtype>* w_ = NULL;
        if (w == NULL) {
            w_ = Vector::init<vtype>(A->n, true, true);
            Vector::fillWithValue(w_, 0.);
        } else {
            w_ = w;
        }
        CSRm::CSRVector_product_adaptive_miniwarp(A, local_x, w_, alpha, beta);
        END_PROF(__FUNCTION__);
        return (w_);
    }

    if (A->os.loc_n == 0 && A->os.needy_n == 0) {
        setupOverlapped(A);
    }

    assert(A->shrinked_flag == 1);

    assert(A->halo.init);

    if (A->halo.to_receive_n == 0 && A->halo.to_send_n == 0) {
        vector<vtype>* ret = CSRm::CSRVector_product_adaptive_miniwarp_new(A, local_x, w, alpha, beta);
        END_PROF(__FUNCTION__);
        return ret;
    }

    CSR* A_ = CSRm::init(A->n, (gstype)A->shrinked_m, A->nnz, false, A->on_the_device, A->is_symmetric, A->full_n, A->row_shift);
    A_->row = A->row;
    A_->val = A->val;
    A_->col = A->shrinked_col;

    overlapped os = A->os;

    assert(os.loc_n != 0 || os.needy_n != 0);

    halo_info hi = A->halo;
    static MPI_Request requests[MAXNTASKS];
    static int ntr = 0;

    int post_local = A->post_local;

    vector<vtype>* x_ = NULL;
    if (A->halo.to_receive_n > 0) {
        x_ = Vector::init<vtype>(A_->m, false, true);
        if (A_->m > xsize) {
            CUDA_FREE(xvalstat);
            xsize = A_->m;
            xvalstat = CUDA_MALLOC(vtype, xsize, true);
        }
        x_->val = xvalstat;
    } else {
        x_ = local_x;
    }

    vector<vtype>* w_ = NULL;
    if (w == NULL) {
        w_ = Vector::init<vtype>(A->n, true, true);
        Vector::fillWithValue(w_, 1.);
    } else {
        w_ = w;
    }

    hipStreamSynchronize(*(os.streams->comm_stream));
    if (hi.to_send_n) {
        assert(hi.what_to_send != NULL);
        assert(hi.what_to_send_d != NULL);
        GridBlock gb = gb1d(hi.to_send_n, BLOCKSIZE);
#if defined(USESTREAM)
        _getToSend_new<<<gb.g, gb.b, 0, *(os.streams->comm_stream)>>>(hi.to_send_d->n, local_x->val, hi.what_to_send_d, hi.to_send_d->val, A->row_shift);
        CHECK_DEVICE(hipMemcpyAsync(hi.what_to_send, hi.what_to_send_d, hi.to_send_n * sizeof(vtype), hipMemcpyDeviceToHost, *(os.streams->comm_stream)));
#else
        _getToSend_new<<<gb.g, gb.b>>>(hi.to_send_d->n, local_x->val, hi.what_to_send_d, hi.to_send_d->val, A->row_shift);
        CHECK_DEVICE(hipMemcpy(hi.what_to_send, hi.what_to_send_d, hi.to_send_n * sizeof(vtype), hipMemcpyDeviceToHost));
#endif
    }

    if (os.loc_n) {
        // start compute local
        CSRm::CSRVector_product_adaptive_indirect_row_miniwarp(A_, local_x, w_,
            os.loc_n, os.loc_rows->val, *(os.streams->local_stream), post_local, alpha, beta);
    }

    int j = 0;
    for (int t = 0; t < nprocs; t++) {
        if (t == myid) {
            continue;
        }
        if (hi.to_receive_counts[t] > 0) {
            CHECK_MPI(
                MPI_Irecv(hi.what_to_receive + (hi.to_receive_spls[t]), hi.to_receive_counts[t], VTYPE_MPI, t, SYNCSOL_TAG, MPI_COMM_WORLD, requests + j));
            j++;
            if (j == MAXNTASKS) {
                fprintf(stderr, "Too many tasks in matrix-vector product, max is %d\n",
                    MAXNTASKS);
                exit(1);
            }
        }
    }

    ntr = j;
    if (hi.to_send_n) {
        hipStreamSynchronize(*(os.streams->comm_stream));
    }

    for (int t = 0; t < nprocs; t++) {
        if (t == myid) {
            continue;
        }
        if (hi.to_send_counts[t] > 0) {
            CHECK_MPI(MPI_Isend(hi.what_to_send + (hi.to_send_spls[t]), hi.to_send_counts[t], VTYPE_MPI, t, SYNCSOL_TAG, MPI_COMM_WORLD, requests + ntr + t));
        }
    }

    // copy received data
    if (hi.to_receive_n) {
        if (ntr > 0) {
            CHECK_MPI(MPI_Waitall(ntr, requests, MPI_STATUSES_IGNORE));
        }
        assert(hi.what_to_receive != NULL);
        assert(hi.what_to_receive_d != NULL);
        GridBlock gb = gb1d(A_->m, BLOCKSIZE);
#if defined(USESTREAM)
        CHECK_DEVICE(hipMemcpyAsync(hi.what_to_receive_d, hi.what_to_receive, hi.to_receive_n * sizeof(vtype), hipMemcpyHostToDevice, *(os.streams->comm_stream)));
        _vector_sync<<<gb.g, gb.b, 0, *(os.streams->comm_stream)>>>(local_x->val, A->n, A->halo.what_to_receive_d, A->halo.to_receive_d->n, post_local, x_->val, x_->n);
#else
        CHECK_DEVICE(hipMemcpy(hi.what_to_receive_d, hi.what_to_receive, hi.to_receive_n * sizeof(vtype), hipMemcpyHostToDevice));
        _vector_sync<<<gb.g, gb.b>>>(local_x->val, A->n, A->halo.what_to_receive_d, A->halo.to_receive_d->n, post_local, x_->val, x_->n);
#endif
        // complete computation for halo
        if (os.needy_n) {
            CSRm::CSRVector_product_adaptive_indirect_row_miniwarp(A_, x_, w_,
                os.needy_n, os.needy_rows->val, *(os.streams->comm_stream), 0, alpha, beta);
        }
    }

    hipStreamSynchronize(*(os.streams->local_stream));
    hipStreamSynchronize(*(os.streams->comm_stream));

    if (A->halo.to_receive_n > 0) {
        FREE(x_);
    }

    A_->col = NULL;
    A_->row = NULL;
    A_->val = NULL;
    FREE(A_);

    END_PROF(__FUNCTION__);
    return (w_);
}

vector<vtype>* CSRm::CSRscaleA_0(CSR* A, vector<vtype>* local_x, vector<vtype>* w, vtype alpha, vtype beta)
{
    _MPI_ENV;

    if (nprocs == 1) {
        vector<vtype>* w_ = NULL;
        if (w == NULL) {
            w_ = Vector::init<vtype>(A->n, true, true);
            Vector::fillWithValue(w_, 0.);
        } else {
            w_ = w;
        }
        CSRm::CSRscale_adaptive_miniwarp(A, local_x, w_, alpha, beta);
        return (w_);
    }

    assert(A->shrinked_flag == 1);

    CSR* A_ = CSRm::init(A->n, (gstype)A->shrinked_m, A->nnz, false, A->on_the_device, A->is_symmetric, A->full_n, A->row_shift);
    A_->row = A->row;
    A_->val = A->val;
    A_->col = A->shrinked_col;

    // ----------------------------------------- temp check -----------------------------------------
    assert(A->halo.to_receive_n + local_x->n == A_->m);
    // ----------------------------------------------------------------------------------------------
    int post_local = A->post_local;

    vector<vtype>* x_ = NULL;
    if (A->halo.to_receive_n > 0) {
        x_ = Vector::init<vtype>(A_->m, false, true);
        if (A_->m > xsize) {
            CUDA_FREE(xvalstat);
            xsize = A_->m;
            xvalstat = CUDA_MALLOC(vtype, xsize, true);
        }
        x_->val = xvalstat;
        GridBlock gb = gb1d(A_->m, BLOCKSIZE);
        _vector_sync<<<gb.g, gb.b>>>(local_x->val, A->n, A->halo.what_to_receive_d, A->halo.to_receive_d->n, post_local, x_->val, x_->n);
    } else {
        x_ = local_x;
    }

    vector<vtype>* w_ = NULL;
    if (w == NULL) {
        w_ = Vector::init<vtype>(A->n, true, true);
        Vector::fillWithValue(w_, 1.);
    } else {
        w_ = w;
    }
    CSRm::CSRscale_adaptive_miniwarp(A_, x_, w_, alpha, beta);

    if (A->halo.to_receive_n > 0) {
        FREE(x_);
    }
    A_->col = NULL;
    A_->row = NULL;
    A_->val = NULL;
    FREE(A_);

    return (w_);
}

vector<vtype>* CSRm::CSRscaleA_0IP(CSR* A, vector<vtype>* local_x, vtype alpha, vtype beta)
{
    _MPI_ENV;

    vector<vtype>* w = Vector::init<vtype>(A->n, false, true);
    w->val = A->val;

    if (nprocs == 1) {
        vector<vtype>* w_ = NULL;
        if (w == NULL) {
            w_ = Vector::init<vtype>(A->n, true, true);
            Vector::fillWithValue(w_, 0.);
        } else {
            w_ = w;
        }
        CSRm::CSRscale_adaptive_miniwarp(A, local_x, w_, alpha, beta);
        return (w_);
    }

    assert(A->shrinked_flag == 1);

    CSR* A_ = CSRm::init(A->n, (gstype)A->shrinked_m, A->nnz, false, A->on_the_device, A->is_symmetric, A->full_n, A->row_shift);
    A_->row = A->row;
    A_->val = A->val;
    A_->col = A->shrinked_col;

    // ----------------------------------------- temp check -----------------------------------------
    assert(A->halo.to_receive_n + local_x->n == A_->m);
    // ----------------------------------------------------------------------------------------------
    int post_local = A->post_local;

    vector<vtype>* x_ = NULL;
    if (A->halo.to_receive_n > 0) {
        x_ = Vector::init<vtype>(A_->m, false, true);
        if (A_->m > xsize) {
            CUDA_FREE(xvalstat);
            xsize = A_->m;
            xvalstat = CUDA_MALLOC(vtype, xsize, true);
        }
        x_->val = xvalstat;
        GridBlock gb = gb1d(A_->m, BLOCKSIZE);
        _vector_sync<<<gb.g, gb.b>>>(local_x->val, A->n, A->halo.what_to_receive_d, A->halo.to_receive_d->n, post_local, x_->val, x_->n);
    } else {
        x_ = local_x;
    }

    vector<vtype>* w_ = NULL;
    if (w == NULL) {
        w_ = Vector::init<vtype>(A->n, true, true);
        Vector::fillWithValue(w_, 1.);
    } else {
        w_ = w;
    }
    CSRm::CSRscale_adaptive_miniwarp(A_, x_, w_, alpha, beta);

    if (A->halo.to_receive_n > 0) {
        FREE(x_);
    }
    A_->col = NULL;
    A_->row = NULL;
    A_->val = NULL;
    FREE(A_);

    return (w_);
}

template <int OP_TYPE>
__global__ void _shifted_CSR_vector_mul_mini_warp(itype n, int MINI_WARP_SIZE, vtype alpha, vtype beta, vtype* A_val, itype* A_row, itype* A_col, vtype* x, vtype* y, itype shift)
{

    itype tid = blockDim.x * blockIdx.x + threadIdx.x;

    int warp = tid / MINI_WARP_SIZE;

    if (warp >= n) {
        return;
    }

    int lane = tid % MINI_WARP_SIZE;
    int mask_id = (tid % FULL_WARP) / MINI_WARP_SIZE;
    int warp_mask = getMaskByWarpID(MINI_WARP_SIZE, mask_id);

    vtype T_i = 0.;

    for (int j = A_row[warp] + lane; j < A_row[warp + 1]; j += MINI_WARP_SIZE) {
        if (OP_TYPE == 0) {
            T_i += (alpha * A_val[j]) * __ldg(&x[A_col[j]]);
        } else if (OP_TYPE == 1) {
            T_i += A_val[j] * __ldg(&x[A_col[j]]);
        } else if (OP_TYPE == 2) {
            T_i += -A_val[j] * __ldg(&x[A_col[j]]);
        }
    }

    for (int k = MINI_WARP_SIZE >> 1; k > 0; k = k >> 1) {
        T_i += __shfl_down_sync(warp_mask, T_i, k);
    }

    if (lane == 0) {
        if (OP_TYPE == 0) {
            y[shift + warp] = T_i + (beta * y[shift + warp]);
        } else if (OP_TYPE == 1) {
            y[shift + warp] = T_i;
        } else if (OP_TYPE == 2) {
            y[shift + warp] = T_i + y[shift + warp];
        }
    }
}

vector<vtype>* CSRm::shifted_CSRVector_product_adaptive_miniwarp(CSR* A, vector<vtype>* x, vector<vtype>* y, itype shift, vtype alpha, vtype beta)
{
    itype n = A->n;

    int density = A->nnz / A->n;

    int min_w_size;

    if (density < MINI_WARP_THRESHOLD_2) {
        min_w_size = 2;
    } else if (density < MINI_WARP_THRESHOLD_4) {
        min_w_size = 4;
    } else if (density < MINI_WARP_THRESHOLD_8) {
        min_w_size = 8;
    } else {
        min_w_size = 16;
    }

    if (y == NULL) {
        assert(beta == 0.);
        y = Vector::init<vtype>(n, true, true);
    }

    GridBlock gb = gb1d(n, BLOCKSIZE, true, min_w_size);

    if (alpha == 1. && beta == 0.) {
        _shifted_CSR_vector_mul_mini_warp<1><<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val, shift);
    } else if (alpha == -1. && beta == 1.) {
        _shifted_CSR_vector_mul_mini_warp<2><<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val, shift);
    } else {
        _shifted_CSR_vector_mul_mini_warp<0><<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val, shift);
    }
    hipDeviceSynchronize();
    return y;
}

__global__ void _shifted_CSR_vector_mul_mini_warp2(itype n, int MINI_WARP_SIZE, vtype alpha, vtype beta, vtype* A_val, itype* A_row, itype* A_col, vtype* x, vtype* y, itype shift)
{

    itype tid = blockDim.x * blockIdx.x + threadIdx.x;

    int warp = tid / MINI_WARP_SIZE;

    if (warp >= n) {
        return;
    }

    int lane = tid % MINI_WARP_SIZE;
    int mask_id = (tid % FULL_WARP) / MINI_WARP_SIZE;
    int warp_mask = getMaskByWarpID(MINI_WARP_SIZE, mask_id);

    vtype T_i = 0.;

    for (int j = A_row[warp] + lane; j < A_row[warp + 1]; j += MINI_WARP_SIZE) {
        T_i += A_val[j] * __ldg(&x[A_col[j] - shift]);
    }

    for (int k = MINI_WARP_SIZE >> 1; k > 0; k = k >> 1) {
        T_i += __shfl_down_sync(warp_mask, T_i, k);
    }

    if (lane == 0) {
        y[warp] = T_i;
    }
}

vector<vtype>* CSRm::shifted_CSRVector_product_adaptive_miniwarp2(CSR* A, vector<vtype>* x, vector<vtype>* y, itype shift, vtype alpha, vtype beta)
{
    itype n = A->n;

    int density = A->nnz / A->n;

    int min_w_size;

    if (density < MINI_WARP_THRESHOLD_2) {
        min_w_size = 2;
    } else if (density < MINI_WARP_THRESHOLD_4) {
        min_w_size = 4;
    } else if (density < MINI_WARP_THRESHOLD_8) {
        min_w_size = 8;
    } else {
        min_w_size = 16;
    }

    if (y == NULL) {
        assert(beta == 0.);
        y = Vector::init<vtype>(n, true, true);
    }

    GridBlock gb = gb1d(n, BLOCKSIZE, true, min_w_size);

    if (alpha == 1. && beta == 0.) {
        _shifted_CSR_vector_mul_mini_warp2<<<gb.g, gb.b>>>(n, min_w_size, alpha, beta, A->val, A->row, A->col, x->val, y->val, shift);
    }

    return y;
}

vector<vtype>* CSRVector_product_MPI(CSR* Alocal, vector<vtype>* x, int type)
{
    assert(Alocal->on_the_device);
    assert(x->on_the_device);

    if (type == 0) {

        // everyone gets all
        vector<vtype>* out = Vector::init<vtype>(x->n, true, true);
        Vector::fillWithValue(out, 0.);

        CSRm::shifted_CSRVector_product_adaptive_miniwarp(Alocal, x, out, Alocal->row_shift);

        vector<vtype>* h_out = Vector::copyToHost(out);
        vector<vtype>* h_full_out = Vector::init<vtype>(x->n, true, false);

        CHECK_MPI(MPI_Allreduce(
            h_out->val,
            h_full_out->val,
            h_full_out->n * sizeof(vtype),
            MPI_DOUBLE,
            MPI_SUM,
            MPI_COMM_WORLD));

        Vector::free(out);
        Vector::free(h_out);

        return h_full_out;

    } else if (type == 1) {

        // local vector outputs
        vector<vtype>* out = Vector::init<vtype>(Alocal->n, true, true);
        CSRm::shifted_CSRVector_product_adaptive_miniwarp(Alocal, x, out, 0);
        return out;

    } else {
        assert(false);
        return NULL;
    }
}

__global__ void _getDiagonal_warp(itype n, int MINI_WARP_SIZE, vtype* A_val, itype* A_col, itype* A_row, vtype* D)
{

    itype tid = blockDim.x * blockIdx.x + threadIdx.x;

    int warp = tid / MINI_WARP_SIZE;

    if (warp >= n) {
        return;
    }

    int lane = tid % MINI_WARP_SIZE;
    int mask_id = (tid % FULL_WARP) / MINI_WARP_SIZE;
    int warp_mask = getMaskByWarpID(MINI_WARP_SIZE, mask_id);

    itype j_start = A_row[warp];
    itype j_stop = A_row[warp + 1];

    int j_d = WARP_SIZE, j;

    for (j = j_start + lane;; j += MINI_WARP_SIZE) {
        int is_diag = __ballot_sync(warp_mask, ((j < j_stop) && (A_col[j] == warp)));
        j_d = __clz(is_diag);
        if (j_d != MINI_WARP_SIZE) {
            break;
        }
    }
}

// SUPER temp kernel
__global__ void _getDiagonal(itype n, vtype* val, itype* col, itype* row, vtype* D, itype row_shift)
{
    itype i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= n) {
        return;
    }

    itype r = i;
    itype j_start = row[i];
    itype j_stop = row[i + 1];

    int j;
    for (j = j_start; j < j_stop; j++) {
        itype c = col[j];

        // if is a diagonal element
        if (c == (r /* + row_shift */)) {
            D[i] = val[j];
            break;
        }
    }
}

// get a copy of the diagonal
vector<vtype>* CSRm::diag(CSR* A)
{
    vector<vtype>* D = Vector::init<vtype>(A->n, true, true);
    GridBlock gb = gb1d(D->n, BLOCKSIZE);
    _getDiagonal<<<gb.g, gb.b>>>(D->n, A->val, A->col, A->row, D->val, A->row_shift);
    return D;
}

__global__ void _row_sum_2(itype n, vtype* A_val, itype* A_row, itype* A_col, vtype* sum)
{

    itype i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= n) {
        return;
    }

    vtype local_sum = 0.;

    int j;
    for (j = A_row[i]; j < A_row[i + 1]; j++) {
        local_sum += fabs(A_val[j]);
    }

    sum[i] = local_sum;
}

vector<vtype>* CSRm::absoluteRowSum(CSR* A, vector<vtype>* sum)
{
    _MPI_ENV;

    assert(A->on_the_device);

    if (sum == NULL) {
        sum = Vector::init<vtype>(A->n, true, true);
    } else {
        assert(sum->on_the_device);
    }

    GridBlock gb = gb1d(A->n, BLOCKSIZE, false);
    _row_sum_2<<<gb.g, gb.b>>>(A->n, A->val, A->row, A->col, sum->val);

    return sum;
}

__global__ void _CSR_vector_mul_prolongator(itype n, vtype* A_val, itype* A_row, itype* A_col, vtype* x, vtype* y)
{

    itype tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n) {
        return;
    }

    itype j = A_row[tid];
    y[tid] += A_val[j] * __ldg(&x[A_col[j]]);
}

vector<vtype>* CSRm::CSRVector_product_prolungator(CSR* A, vector<vtype>* x, vector<vtype>* y)
{
    itype n = A->n;

    assert(A->on_the_device);
    assert(x->on_the_device);

    GridBlock gb = gb1d(n, BLOCKSIZE);

    _CSR_vector_mul_prolongator<<<gb.g, gb.b>>>(n, A->val, A->row, A->col, x->val, y->val);

    return y;
}

// checks if the colmuns are in the correct order
void CSRm::checkColumnsOrder(CSR* A_)
{

    CSR* A;
    if (A_->on_the_device) {
        A = CSRm::copyToHost(A_);
    } else {
        A = A_;
    }

    for (int i = 0; i < A->n; i++) {
        itype _c = -1;
        for (int j = A->row[i]; j < A->row[i + 1]; j++) {
            itype c = A->col[j];

            if (c < _c) {
                printf("WRONG ORDER COLUMNS: %d %d-%d\n", i, c, _c);
                exit(1);
            }
            if (c > _c) {
                _c = c;
            }
            if (c > A->m - 1) {
                printf("WRONG COLUMN TO BIG: %d %d-%d\n", i, c, _c);
                exit(1);
            }
        }
    }
    if (A_->on_the_device) {
        CSRm::free(A);
    }
}

#define MY_EPSILON 0.0001
void CSRm::checkMatrix(CSR* A_, bool check_diagonal)
{
    _MPI_ENV;
    CSR* A = NULL;

    if (A_->on_the_device) {
        A = CSRm::copyToHost(A_);
    } else {
        A = A_;
    }

    for (int i = 0; i < A->n; i++) {
        for (int j = A->row[i]; j < A->row[i + 1]; j++) {
            int c = A->col[j];
            double v = A->val[j];
            int found = 0;
            for (int jj = A->row[c]; jj < A->row[c + 1]; jj++) {
                if (A->col[jj] == i) {
                    found = 1;
                    vtype diff = abs(v - A->val[jj]);
                    if (A->val[jj] != v && diff >= MY_EPSILON) {
                        printf("\n\nNONSYM %lf %lf %lf\n\n", v, A->val[jj], diff);
                        exit(1);
                    }
                    break;
                }
            }
            if (!found) {
                printf("BAD[%d]: %d %d\n", myid, i, c);
                exit(1);
            }
        }
    }

    checkColumnsOrder(A);

    if (check_diagonal) {
        printf("CHECKING DIAGONAL\n");
        for (int i = 0; i < A->n; i++) {
            bool found = false;
            for (int j = A->row[i]; j < A->row[i + 1]; j++) {
                int c = A->col[j];
                vtype v = A->val[j];
                if (c == i && v > 0.) {
                    found = true;
                }
            }
            if (!found) {
                printf("MISSING ELEMENT DIAG %d\n", i);
                exit(1);
            }
        }
        if (A_->on_the_device) {
            CSRm::free(A);
        }
    }
}

/**
 * CUDA kernel.
 * Scans a matrix in CSR format and collects non zero items in ret.
 * Should be invoked using 1 (mini)warp per row.
 *
 * @param row CSR matrix row indexes
 * @param col CSR matrix column indexes
 * @param val CSR matrix (non zero) values
 * @param nrows number of rows
 * @param ret returned array
 */
__global__ void _combineRowAndCol(itype* row, itype* col, vtype* val,
    gstype row_shift, gsstype col_shift, stype nrows, int warpSize,
    matrixItem_t* ret)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int irow = tid / warpSize;
    int lane = tid % warpSize;
    int rstart, rend;
    if (irow < nrows) {
        rstart = row[irow] + lane;
        rend = row[irow + 1];
        for (int we = rstart; we < rend; we += warpSize) {
            ret[we].row = irow + row_shift;
            ret[we].col = col[we] - col_shift;
            ret[we].val = val[we];
        }
    }
}

matrixItem_t* CSRm::collectMatrixItems(CSR* dlA, FILE* debug, bool useColShift)
{
    // Allocate vector to collect non-zero items in dlA
    // ---------------------------------------------------------------------------
    matrixItem_t* d_nnzItems = CUDA_MALLOC(matrixItem_t, dlA->nnz, true);

    // Collect items
    // ---------------------------------------------------------------------------
    int warpSize = CSRm::choose_mini_warp_size(dlA);
    GridBlock gb = getKernelParams(dlA->n * warpSize); // One mini-warp per row
    _combineRowAndCol<<<gb.g, gb.b>>>(dlA->row, dlA->col, dlA->val, dlA->row_shift, useColShift ? dlA->col_shifted : 0, dlA->n, warpSize, d_nnzItems);
    hipError_t err = hipDeviceSynchronize();
    CHECK_DEVICE(err);

    if (debug) {
        debugMatrixItems("nnzItems", d_nnzItems, dlA->nnz, true, debug);
    }

    return d_nnzItems;
}

matrixItem_t* CSRm::collectMatrixItems_nogpu(CSR* dlA, FILE* debug, bool useColShift)
{
    // Allocate vector to collect non-zero items in dlA
    // ---------------------------------------------------------------------------
    matrixItem_t* d_nnzItems = MALLOC(matrixItem_t, dlA->nnz, true);

    // Collect items
    // ---------------------------------------------------------------------------
    itype col_shift = useColShift ? dlA->col_shifted : 0;
    for (itype irow = 0; irow < dlA->n; irow++) {
        for (int we = dlA->row[irow]; we < dlA->row[irow + 1]; we += 1) {
            d_nnzItems[we].row = irow + dlA->row_shift;
            d_nnzItems[we].col = dlA->col[we] - col_shift;
            d_nnzItems[we].val = dlA->val[we];
        }
    }

    if (debug) {
        debugMatrixItems("nnzItems", d_nnzItems, dlA->nnz, false, debug);
    }

    return d_nnzItems;
}

/**
 * @param dlA device local A
 * @param f process-specific log file
 */
CSR* CSRm::transpose(CSR* dlA, FILE* f, const char* shape)
{
    assert(dlA->on_the_device);

    _MPI_ENV;

    if (f) {
        fprintf(f, "[Process %d] n (rows) : %d\n" , myid, dlA->n);
        fprintf(f, "[Process %d] m (cols) : %lu\n", myid, dlA->m);
        fprintf(f, "[Process %d] nnz      : %d\n" , myid, dlA->nnz);
        fprintf(f, "[Process %d] row shift: %lu\n", myid, dlA->row_shift);
    }

    // Register custom MPI datatypes
    // ---------------------------------------------------------------------------
    registerMatrixItemMpiDatatypes();

    // Collect non-zero items in dlA
    // ---------------------------------------------------------------------------
    matrixItem_t* d_nnzItems = collectMatrixItems(dlA, f, true);

    // Identify the items to be sent: they are the ones whose column
    // index is before the first row index assigned to the process or
    // after the last row assigned to the process.
    // ---------------------------------------------------------------------------

    size_t nnzItemsToBeSentSize = 0;
    matrixItem_t* d_nnzItemsToBeSent = devicePartition(
        d_nnzItems,
        dlA->nnz,
        MatrixItemColumnIndexOutOfBoundsSelector(
            dlA->row_shift,
            dlA->row_shift + dlA->n - 1),
        &nnzItemsToBeSentSize);

    if (f) {
        fprintf(f, "nnzItemsToBeSent effective size: %zu\n", nnzItemsToBeSentSize);
        debugMatrixItems("nnzItemsToBeSent", d_nnzItemsToBeSent, nnzItemsToBeSentSize, true, f);
    }

    // Identify the items not to be requested: they are the ones whose column
    // index is between the first row index and the last row index
    // assigned to the process.
    // ---------------------------------------------------------------------------
    matrixItem_t* d_nnzItemsNotToBeSent = d_nnzItemsToBeSent + nnzItemsToBeSentSize;
    size_t nnzItemsNotToBeSentSize = dlA->nnz - nnzItemsToBeSentSize;

    if (f) {
        fprintf(f, "nnzItemsNotToBeSent effective size: %zu\n", nnzItemsNotToBeSentSize);
        debugMatrixItems("nnzItemsNotToBeSent", d_nnzItemsNotToBeSent, nnzItemsNotToBeSentSize, true, f);
    }

    // Release memory
    // ---------------------------------------------------------------------------
    CUDA_FREE(d_nnzItems);

    // Copy data to host in order to perform MPI communication
    // ---------------------------------------------------------------------------
    matrixItem_t* h_nnzItemsToBeSent = copyArrayToHost(d_nnzItemsToBeSent, nnzItemsToBeSentSize);

    // Exchange data with other processes
    // ---------------------------------------------------------------------------
    ProcessSelector processSelector(dlA, f);
    MatrixItemSender itemSender(&processSelector, f);
    MpiBuffer<matrixItem_t> sendBuffer;
    MpiBuffer<matrixItem_t> rcvBuffer;
    itemSender.send(h_nnzItemsToBeSent, nnzItemsToBeSentSize,
        &sendBuffer, &rcvBuffer);

    // Now we have all the initially missing values in rcv_buffer and all the
    // initially interesting values in nnzItemsNotToBeSent. We need
    // to construct a new (transposed) matrix from all those values.
    // ---------------------------------------------------------------------------
    size_t concatenatedSize = rcvBuffer.size + nnzItemsNotToBeSentSize;

    matrixItem_t* d_concatenated = concatArrays<matrixItem_t>(
        rcvBuffer.buffer, // arr1
        rcvBuffer.size, // len1
        false, // onDevice1
        d_nnzItemsToBeSent + nnzItemsToBeSentSize, // arr2
        nnzItemsNotToBeSentSize, // len2
        true, // onDevice2
        true // retOnDevice
    );

    if (f) {
        fprintf(f, "concatenatedItems effective size: %zu\n", concatenatedSize);
        debugMatrixItems("concatenatedItems", d_concatenated, concatenatedSize, true, f);
    }

    // Release memory
    // ---------------------------------------------------------------------------
    CUDA_FREE(d_nnzItemsToBeSent);
    FREE(h_nnzItemsToBeSent);

    // Sort items by col, row (pratically: already transposed)
    // ---------------------------------------------------------------------------
    deviceSort<matrixItem_t, gstype, MatrixItemTransposedComparator>(d_concatenated, concatenatedSize, MatrixItemTransposedComparator(dlA->full_n));

    if (f) {
        debugMatrixItems("sortedItems", d_concatenated, concatenatedSize, true, f);
    }

    if (!concatenatedSize) {
        fprintf(f ? f : stderr, "concatenatedSize in process %d is 0. Row shift: %ld, n: %d\n", myid, dlA->row_shift, dlA->n);
    }

    // Create new CSR matrix
    // ---------------------------------------------------------------------------
    bool is_rectangular = !strncasecmp("R", shape, 1);
    CSR* d_transposed = CSRm::init(
        is_rectangular ? dlA->m : dlA->n, // Nr of rows,
        is_rectangular ? dlA->full_n : dlA->full_n, // Nr of columns,
        concatenatedSize, // nnz
        true, // Allocate memory
        true, // On the device
        false, // Is symmetric?
        is_rectangular ? dlA->m : dlA->full_n,
        dlA->row_shift);

    // Fill CSR
    // ---------------------------------------------------------------------------
    fillCsrFromMatrixItems(
        d_concatenated,
        concatenatedSize,
        d_transposed->n,
        d_transposed->row_shift,
        &(d_transposed->row),
        &(d_transposed->col),
        &(d_transposed->val),
        true, // Transposed
        false // Allocate memory
    );

    if (f) {
        debugArray("d_transposed->row[%d] = %d\n", d_transposed->row, d_transposed->n + 1, true, f);
        debugArray("d_transposed->col[%d] = %d\n", d_transposed->col, d_transposed->nnz, true, f);
        debugArray("d_transposed->val[%d] = %lf\n", d_transposed->val, d_transposed->nnz, true, f);
    }

    CUDA_FREE(d_concatenated);

    if (d_transposed->row_shift) {
        CSRm::shift_cols(d_transposed, -d_transposed->row_shift);
        d_transposed->col_shifted = -d_transposed->row_shift;
    }

    return d_transposed;
}

/**
 * @param dlA device local A
 * @param f process-specific log file
 */
CSR* CSRm::Transpose_local(CSR* dlA, FILE* f)
{
    assert(dlA->on_the_device);

    _MPI_ENV;

    if (f) {
        fprintf(f, "n (rows) : %d\n", dlA->n);
        fprintf(f, "m (cols) : %lu\n", dlA->m);
        fprintf(f, "nnz      : %d\n", dlA->nnz);
        fprintf(f, "row shift: %lu\n", dlA->row_shift);
    }

    // Register custom MPI datatypes
    // ---------------------------------------------------------------------------
    registerMatrixItemMpiDatatypes();

    // Collect non-zero items in dlA
    // ---------------------------------------------------------------------------
    matrixItem_t* d_nnzItems = collectMatrixItems(dlA, f, true);

    // Sort items by col, row (pratically: already transposed)
    // ---------------------------------------------------------------------------
    deviceSort<matrixItem_t, gstype, MatrixItemTransposedComparator>(d_nnzItems, dlA->nnz, MatrixItemTransposedComparator(dlA->n));

    if (f) {
        debugMatrixItems("sortedItems", d_nnzItems, dlA->nnz, true, f);
    }

    // Create new CSR matrix
    // ---------------------------------------------------------------------------
    CSR* d_transposed = CSRm::init(
        dlA->m, // Nr of rows,
        dlA->full_n, // Nr of columns,
        dlA->nnz, // nnz
        true, // Allocate memory
        true, // On the device
        false, // Is symmetric?
        dlA->m,
        0); // row shift

    // Fill CSR
    // ---------------------------------------------------------------------------
    fillCsrFromMatrixItems(
        d_nnzItems,
        dlA->nnz,
        d_transposed->n,
        d_transposed->row_shift,
        &(d_transposed->row),
        &(d_transposed->col),
        &(d_transposed->val),
        true, // Transposed
        false // Allocate memory
    );

    if (f) {
        debugArray("d_transposed->row[%d] = %d\n", d_transposed->row, d_transposed->n + 1, true, f);
        debugArray("d_transposed->col[%d] = %d\n", d_transposed->col, d_transposed->nnz, true, f);
        debugArray("d_transposed->val[%d] = %lf\n", d_transposed->val, d_transposed->nnz, true, f);
    }

    // Release memory
    // ---------------------------------------------------------------------------
    CUDA_FREE(d_nnzItems);

    if (dlA->row_shift) {
        CSRm::shift_cols(d_transposed, -dlA->row_shift);
    }

    return d_transposed;
}

/**
 * CUDA kernel.
 * Scans a matrix in CSR format, counts non zero items in each requested row,
 * and returns the result in ret.
 * Should be invoked using 1 thread per requested row.
 *
 * @param row CSR matrix row indexes
 * @param col CSR matrix column indexes
 * @param val CSR matrix (non zero) values
 * @param nrows number of rows
 * @param ret returned array
 */
__global__ void CSRm::countNnzPerRow(itype* row,
    itype row_shift,
    itype* requestedRowIndexes,
    itype requestedRowIndexesSize,
    itype* ret)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < requestedRowIndexesSize) {
        itype irow = requestedRowIndexes[tid] - row_shift;
        itype rstart = row[irow];
        itype rend = row[irow + 1];
        ret[tid] = rend - rstart;
    }
}

/**
 * CUDA kernel.
 * Scans a matrix in CSR format and collects non zero items in ret.
 * Should be invoked using 1 (mini)warp per requested row index.
 *
 * @param row CSR matrix row indexes
 * @param col CSR matrix column indexes
 * @param val CSR matrix (non zero) values
 * @param row_shift Distributed CSR matrix row shift
 * @param n Number of requested rows
 * @param requestedRowIndexes Requested row indexes
 * @param counter Number of nnz per requested row index
 * @param offset Offset of each requested row with respect to the return buffer
 * @param ret returned buffer
 */
__global__ void CSRm::collectNnzPerRow(
    int warpSize,
    itype* row,
    itype* col,
    vtype* val,
    itype row_shift,
    itype n,
    itype* requestedRowIndexes,
    itype* counter,
    itype* offset,
    matrixItem_t* ret)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int index = tid / warpSize;
    int lane = tid % warpSize;
    if (index < n) {
        int off = offset[index];
        int irow = requestedRowIndexes[index] - row_shift;
        int rstart = row[irow];
        int rend = row[irow + 1];
        for (int we = rstart + lane; we < rend; we += warpSize) {
            int ibuf = off + we - rstart;
            ret[ibuf].row = irow + row_shift;
            ret[ibuf].col = col[we];
            ret[ibuf].val = val[we];
        }
    }
}

void check_and_fix_order(CSR* A)
{

    itype* Arow = A->row;
    itype* Acol = A->col;
    vtype* Aval = A->val;
    itype prev;
    int wrongo;
    for (int i = 0; i < A->n; i++) {
        wrongo = 0;
        prev = A->col[Arow[i]];
        for (int j = Arow[i] + 1; j < Arow[i + 1]; j++) {
            if (A->col[j] < prev) {
                wrongo = 1;
                break;
            } else {
                prev = A->col[j];
            }
        }
        if (wrongo) {
            bubbleSort(&Acol[Arow[i]], &Aval[Arow[i]], (Arow[i + 1] - Arow[i]));
        }
    }
}

void swap(itype* xcol, itype* ycol, vtype* xval, vtype* yval)
{
    itype temp = *xcol;
    vtype tempf = *xval;
    *xcol = *ycol;
    *xval = *yval;
    *ycol = temp;
    *yval = tempf;
}

void bubbleSort(itype arr[], vtype val[], itype n)
{
    itype i, j;
    for (i = 0; i < n - 1; i++) {

        // Last i elements are already in place
        for (j = 0; j < n - i - 1; j++) {
            if (arr[j] > arr[j + 1]) {
                swap(&arr[j], &arr[j + 1], &val[j], &val[j + 1]);
            }
        }
    }
}

CSR* read_matrix_from_file(const char* matrix_path, int m_type, bool loadOnDevice)
{

    CSR* A_host = NULL;

    switch (m_type) {
    case 0:
        A_host = readMTXDouble(matrix_path);
        break;
    case 1:
        A_host = readMTX2Double(matrix_path);
        break;
    default:
        std::cout << "You need to specify an input matrix type with the argument -F/--inputype\n";
        exit(1);
    }

    assert(A_host != NULL);

    if (loadOnDevice) {
        CSR* A = CSRm::copyToDevice(A_host);
        CSRm::free(A_host);
        return A;
    }

    return A_host;
}

// stolen from BootCMatch CPU
CSR* readMTXDouble(const char* file_name)
{
    FILE* fp;
    char banner[64], mtx[64], crd[64], data_type[64], storage_scheme[64];
    char buffer[BUFSIZE + 1];
    double *matrix_value, *matrix_data, val;
    unsigned long int *matrix_cooi, *matrix_i;
    unsigned long int *matrix_cooj, *matrix_j;
    unsigned long int num_rows, num_cols, ri, cj;
    unsigned long int fr_nonzeros, allc_nonzeros;
    unsigned long int num_nonzeros;
    unsigned long int max_col = 0, is_general = 0, is_symmetric = 0;
    unsigned long int row_shift = 0;
    unsigned long int full_n = 0;
    unsigned long int col_shifted = 0;

    int file_base = 1;

    long int i, j, k, k0, iad;
    double x;

    /*----------------------------------------------------------
     * Read in the data (matrix in MM format)
     *----------------------------------------------------------*/

    fp = fopen(file_name, "r");
    if (fp == NULL) {
        fprintf(stdout, "Error opening file %s, errno = %d: %s\n", file_name, errno, strerror(errno));
        exit(1);
    }

    fscanf(fp, "%s %s %s %s %s\n", banner, mtx, crd, data_type, storage_scheme);
    fgets(buffer, BUFSIZE, fp);
    for (; buffer[0] == '%'; fgets(buffer, BUFSIZE, fp))
        ;

    int readParams = sscanf(buffer, "%lu %lu %lu "
                                    "%lu %lu %lu",
        &num_rows, &num_cols, &fr_nonzeros,
        &row_shift, &full_n, &col_shifted);
    
    assert(readParams == 3 || readParams == 6);
    if (readParams == 3) {
        row_shift = 0;
        full_n = num_rows;
        col_shifted = 0;
    }
    
    if (strcmp(data_type, "real") != 0) {
        fprintf(stderr, "Error: we only read real matrices, not '%s'\n", data_type);
        fclose(fp);
        return (NULL);
    }

    if (strcmp(storage_scheme, "general") == 0) {
        allc_nonzeros = fr_nonzeros;
        is_general = 1;
    } else if (strcmp(storage_scheme, "symmetric") == 0) {
        allc_nonzeros = 2 * fr_nonzeros;
        is_symmetric = 1;
    } else {
        fprintf(stderr, "Error: unhandled storage scheme '%s'\n", storage_scheme);
        fclose(fp);
        return (NULL);
    }

    matrix_cooi = MALLOC(unsigned long int, allc_nonzeros, true);
    matrix_cooj = MALLOC(unsigned long int, allc_nonzeros, true);
    matrix_value = MALLOC(double, allc_nonzeros, true);
    if (is_general) {
        num_nonzeros = fr_nonzeros;
        for (j = 0; j < fr_nonzeros; j++) {
            if (fgets(buffer, BUFSIZE, fp) != NULL) {
                sscanf(buffer, "%lu %lu %le", &matrix_cooi[j], &matrix_cooj[j], &matrix_value[j]);
                matrix_cooi[j] -= file_base + row_shift;
                matrix_cooj[j] -= file_base;
                if (matrix_cooj[j] > max_col) {
                    max_col = matrix_cooj[j];
                }
            } else {
                fprintf(stderr, "Reading from MatrixMarket file failed\n");
                fprintf(stderr, "Error while trying to read record %ld of %lu from file %s\n",
                    j, fr_nonzeros, file_name);
                exit(-1);
            }
        }
    } else if (is_symmetric) {
        k = 0;
        for (j = 0; j < fr_nonzeros; j++) {
            if (fgets(buffer, BUFSIZE, fp) != NULL) {
                sscanf(buffer, "%lu %lu %le", &ri, &cj, &val);
                ri -= file_base;
                cj -= file_base;
                if (cj > max_col) {
                    max_col = cj;
                }
                matrix_cooi[k] = ri;
                matrix_cooj[k] = cj;
                matrix_value[k] = val;
                k++;
                if (ri != cj) {
                    matrix_cooi[k] = cj;
                    matrix_cooj[k] = ri;
                    matrix_value[k] = val;
                    k++;
                }
            } else {
                fprintf(stderr, "Reading from MatrixMarket file failed\n");
                fprintf(stderr, "Error while trying to read record %ld of %lu from file %s\n",
                    j, fr_nonzeros, file_name);
                fclose(fp);
                return (NULL);
            }
        }
        num_nonzeros = k;
    } else {
        fprintf(stderr, "Internal error: neither symmetric nor general ? \n");
        fclose(fp);
        return (NULL);
    }
    /*----------------------------------------------------------
     * Transform matrix from COO to CSR format
     *----------------------------------------------------------*/

    matrix_i = MALLOC(unsigned long, num_rows + 1, true);

    /* determine row lenght */
    for (j = 0; j < num_nonzeros; j++) {
        if (matrix_cooi[j] < num_rows) {
            matrix_i[matrix_cooi[j]] = matrix_i[matrix_cooi[j]] + 1;
        } else {
            fprintf(stderr, "Wrong row index %lu at position %ld\n", matrix_cooi[j], j);
        }
    }

    /* starting position of each row */
    k = 0;
    for (j = 0; j <= num_rows; j++) {
        k0 = matrix_i[j];
        matrix_i[j] = k;
        k = k + k0;
    }
    matrix_j = MALLOC(unsigned long int, num_nonzeros, true);
    matrix_data = MALLOC(double, num_nonzeros, true);

    /* go through the structure once more. Fill in output matrix */
    for (k = 0; k < num_nonzeros; k++) {
        i = matrix_cooi[k];
        j = matrix_cooj[k];
        x = matrix_value[k];
        iad = matrix_i[i];
        matrix_data[iad] = x;
        matrix_j[iad] = j;
        matrix_i[i] = iad + 1;
    }
    /* shift back matrix_i */
    for (j = num_rows - 1; j >= 0; j--) {
        matrix_i[j + 1] = matrix_i[j];
    }
    matrix_i[0] = 0;

    // assert(num_rows > 0 && num_cols > 0 && num_nonzeros >= 0);
    CSR* A = CSRm::init(num_rows, num_cols, num_nonzeros, true, false, false, num_rows, row_shift);
    FREE(A->val);
    A->val = matrix_data;

    for (j = 0; j <= num_rows; j++) {
        A->row[j] = matrix_i[j];
    }
    for (k = 0; k < num_nonzeros; k++) {
        A->col[k] = matrix_j[k];
    }

    FREE(matrix_cooi);
    FREE(matrix_cooj);
    FREE(matrix_value);
    fclose(fp);

    return A;
}

CSR* readMTX2Double(const char* file_name)
{

    FILE* fp;

    double *matrix_value, *matrix_data;
    int *matrix_cooi, *matrix_i;
    int *matrix_cooj, *matrix_j;
    int num_rows;
    int num_nonzeros;
    int max_col = 0;

    int file_base = 1;

    int i, j, k, k0, iad;
    double x;

    /*----------------------------------------------------------
     * Read in the data (matrix in COO format)
     *----------------------------------------------------------*/

    fp = fopen(file_name, "r");
    if (fp == NULL) {
        fprintf(stdout, "Error opening file %s, errno = %d: %s\n", file_name, errno, strerror(errno));
        printf("FILE NOT FOUND!\n");
        exit(1);
    }

    fscanf(fp, "%d", &num_rows);
    fscanf(fp, "%d", &num_nonzeros);

    matrix_cooi = MALLOC(int, num_nonzeros, true);
    for (j = 0; j < num_nonzeros; j++) {
        fscanf(fp, "%d", &matrix_cooi[j]);
        matrix_cooi[j] -= file_base;
    }
    matrix_cooj = MALLOC(int, num_nonzeros, true);
    for (j = 0; j < num_nonzeros; j++) {
        fscanf(fp, "%d", &matrix_cooj[j]);
        matrix_cooj[j] -= file_base;
        if (matrix_cooj[j] > max_col) {
            max_col = matrix_cooj[j];
        }
    }
    matrix_value = MALLOC(double, num_nonzeros, true);
    for (j = 0; j < num_nonzeros; j++) {
        fscanf(fp, "%le", &matrix_value[j]);
    }

    /*----------------------------------------------------------
     * Transform matrix from COO to CSR format
     *----------------------------------------------------------*/

    matrix_i = MALLOC(int, num_rows + 1, true);

    /* determine row lenght */
    for (j = 0; j < num_nonzeros; j++) {
        matrix_i[matrix_cooi[j]] = matrix_i[matrix_cooi[j]] + 1;
    }

    /* starting position of each row */
    k = 0;
    for (j = 0; j <= num_rows; j++) {
        k0 = matrix_i[j];
        matrix_i[j] = k;
        k = k + k0;
    }
    matrix_j = MALLOC(int, num_nonzeros, true);
    matrix_data = MALLOC(double, num_nonzeros, true);

    /* go through the structure once more. Fill in output matrix */
    for (k = 0; k < num_nonzeros; k++) {
        i = matrix_cooi[k];
        j = matrix_cooj[k];
        x = matrix_value[k];
        iad = matrix_i[i];
        matrix_data[iad] = x;
        matrix_j[iad] = j;
        matrix_i[i] = iad + 1;
    }
    /* shift back matrix_i */
    for (j = num_rows - 1; j >= 0; j--) {
        matrix_i[j + 1] = matrix_i[j];
    }
    matrix_i[0] = 0;

    assert(num_rows > 0 && num_rows > 0 && num_nonzeros >= 0);
    CSR* A = CSRm::init(num_rows, num_rows, num_nonzeros, false, false, false, num_rows);
    A->val = matrix_data;
    A->row = matrix_i;
    A->col = matrix_j;

    FREE(matrix_cooi);
    FREE(matrix_cooj);
    FREE(matrix_value);
    fclose(fp);

    return A;
}

void CSRMatrixPrintMM(CSR* A_, const char* file_name)
{
    CSR* A = NULL;
    if (A_->on_the_device) {
        A = CSRm::copyToHost(A_);
    } else {
        A = A_;
    }

    FILE* fp;

    double* matrix_data;
    int* matrix_i;
    int* matrix_j;
    int num_rows;
    int num_cols, nnz;

    int file_base = 1;

    int i, j;

    matrix_data = A->val;
    matrix_i = A->row;
    matrix_j = A->col;
    num_rows = A->n;
    num_cols = A->m;
    nnz = A->nnz;

    fp = fopen(file_name, "w");
    fprintf(fp, "%s\n", "%%MatrixMarket matrix coordinate real general");

    fprintf(fp, "%d  %d %d \n", num_rows, num_cols, nnz);

    for (i = 0; i < num_rows; i++) {
        for (j = matrix_i[i]; j < matrix_i[i + 1]; j++) {
            fprintf(fp, "%d   %d  %lg\n", i + file_base, matrix_j[j] + file_base, matrix_data[j]);
        }
    }
    fclose(fp);
}

void CSRm::printInfo(CSR* A, FILE* fp)
{
    _MPI_ENV;

    fprintf(fp, "nnz                   : %d\n", A->nnz);
    fprintf(fp, "n                     : %d\n", A->n);
    fprintf(fp, "m                     : %lu\n", A->m);
    fprintf(fp, "shrinked_m            : %d\n", A->shrinked_m);
    fprintf(fp, "full_n                : %lu\n", A->full_n);
    fprintf(fp, "full_m                : %lu\n", A->full_m);
    fprintf(fp, "on_the_device         : %d\n", A->on_the_device);
    fprintf(fp, "is_symmetric          : %d\n", A->is_symmetric);
    fprintf(fp, "shrinked_flag         : %d\n", A->shrinked_flag);
    fprintf(fp, "custom_alloced        : %d\n", A->custom_alloced);
    fprintf(fp, "col_shifted           : %ld\n", A->col_shifted);
    fprintf(fp, "shrinked_firstrow     : %lu\n", A->shrinked_firstrow);
    fprintf(fp, "shrinked_lastrow      : %lu\n", A->shrinked_lastrow);
    fprintf(fp, "row_shift             : %lu\n", A->row_shift);
    fprintf(fp, "bitcolsize            : %d\n", A->bitcolsize);
    fprintf(fp, "post_local            : %d\n", A->post_local);

    // fprintf(fp, "val                   : 0x%X\n", A->val);
    // fprintf(fp, "col                   : 0x%X\n", A->col);
    // fprintf(fp, "row                   : 0x%X\n", A->row);
    // fprintf(fp, "shrinked_col          : 0x%X\n", A->shrinked_col);
    // fprintf(fp, "bitcol                : 0x%X\n", A->bitcol);

    fprintf(fp, "halo.init             : %d\n", A->halo.init);
    fprintf(fp, "halo.to_receive_n     : %d\n", A->halo.to_receive_n);
    if (A->halo.to_receive) {
        debugArray("halo.to_receive[%d]: %d\n", A->halo.to_receive->val, A->halo.to_receive->n, A->halo.to_receive->on_the_device, fp);
    }
    if (A->halo.to_receive_d) {
        debugArray("halo.to_receive_d[%d]: %d\n", A->halo.to_receive_d->val, A->halo.to_receive_d->n, A->halo.to_receive_d->on_the_device, fp);
    }
    if (A->halo.to_receive_counts) {
        debugArray("halo.to_receive_counts[%d]: %d\n", A->halo.to_receive_counts, nprocs, false, fp);
    }
    if (A->halo.to_receive_spls) {
        debugArray("halo.to_receive_spls[%d]: %d\n", A->halo.to_receive_spls, nprocs, false, fp);
    }
    if (A->halo.what_to_receive) {
        debugArray("halo.what_to_receive[%d]: %d\n", A->halo.what_to_receive, A->halo.to_receive_n, false, fp);
    }
    if (A->halo.what_to_receive_d) {
        debugArray("halo.what_to_receive_d[%d]: %d\n", A->halo.what_to_receive_d, A->halo.to_receive_n, true, fp);
    }

    fprintf(fp, "halo.to_send_n        : %d\n", A->halo.to_send_n);
    if (A->halo.to_send) {
        debugArray("halo.to_send[%d]: %d\n", A->halo.to_send->val, A->halo.to_send->n, A->halo.to_send->on_the_device, fp);
    }
    if (A->halo.to_send_d) {
        debugArray("halo.to_send_d[%d]: %d\n", A->halo.to_send_d->val, A->halo.to_send_d->n, A->halo.to_send_d->on_the_device, fp);
    }
    if (A->halo.to_send_counts) {
        debugArray("halo.to_send_counts[%d]: %d\n", A->halo.to_send_counts, nprocs, false, fp);
    }
    if (A->halo.to_send_spls) {
        debugArray("halo.to_send_spls[%d]: %d\n", A->halo.to_send_spls, nprocs, false, fp);
    }
    if (A->halo.what_to_send) {
        debugArray("halo.what_to_send[%d]: %d\n", A->halo.what_to_send, A->halo.to_send_n, false, fp);
    }
    if (A->halo.what_to_send_d) {
        debugArray("halo.what_to_send_d[%d]: %d\n", A->halo.what_to_send_d, A->halo.to_send_n, true, fp);
    }

    fprintf(fp, "os.loc_n              : %d\n", A->os.loc_n);
    if (A->os.loc_rows) {
        debugArray("os.loc_rows[%d]: %d\n", A->os.loc_rows->val, A->os.loc_rows->n, A->os.loc_rows->on_the_device, fp);
    }

    fprintf(fp, "os.needy_n            : %d\n", A->os.needy_n);
    if (A->os.needy_rows) {
        debugArray("os.needy_rows[%d]: %d\n", A->os.needy_rows->val, A->os.needy_rows->n, A->os.needy_rows->on_the_device, fp);
    }

    fprintf(fp, "\n");
}
