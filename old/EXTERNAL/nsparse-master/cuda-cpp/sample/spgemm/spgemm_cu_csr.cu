#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>

#include <nsparse.hpp>
#include <CSR.hpp>
#include <SpGEMM.hpp>

typedef int IT;
#ifdef FLOAT
typedef float VT;
#else
typedef double VT;
#endif

template <class idType, class valType>
void spgemm_cu_csr(CSR<idType, valType> a, CSR<idType, valType> b, CSR<idType, valType> &c)
{

    idType i;
  
    long long int flop_count;
    hipEvent_t event[2];
    float msec, ave_msec, flops;
  
    hipsparseHandle_t cusparseHandle;
    hipsparseMatDescr_t descr_a, descr_b, descr_c;
    hipsparseOperation_t trans_a, trans_b;

    for (i = 0; i < 2; i++) {
        hipEventCreate(&(event[i]));
    }
    trans_a = trans_b = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  
    /* Memcpy A and B from Host to Device */
    a.memcpyHtD();
    b.memcpyHtD();
  
    /* Count flop of SpGEMM computation */
    get_spgemm_flop(a, b, flop_count);

    /* Set up cuSPARSE Library */
    hipsparseCreate(&cusparseHandle);
    hipsparseCreateMatDescr(&descr_a);
    hipsparseCreateMatDescr(&descr_b);
    hipsparseCreateMatDescr(&descr_c);
    hipsparseSetMatType(descr_a, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descr_b, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descr_c, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr_a, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descr_b, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descr_c, HIPSPARSE_INDEX_BASE_ZERO);
  
    /* Execution of SpGEMM on Device */
    ave_msec = 0;
    for (i = 0; i < SpGEMM_TRI_NUM; i++) {
        if (i > 0) {
            c.release_csr();
        }
        hipEventRecord(event[0], 0);
        SpGEMM_cuSPARSE_kernel(a, b, c, cusparseHandle, trans_a, trans_b, descr_a, descr_b, descr_c);
        hipEventRecord(event[1], 0);
        hipDeviceSynchronize();
        hipEventElapsedTime(&msec, event[0], event[1]);
    
        if (i > 0) {
            ave_msec += msec;
        }
    }
    ave_msec /= SpGEMM_TRI_NUM - 1;

    flops = (float)(flop_count) / 1000 / 1000 / ave_msec;
    printf("SpGEMM using CSR format (cuSPARSE): %f[GFLOPS], %f[ms]\n", flops, ave_msec);

    c.memcpyDtH();

    a.release_csr();
    b.release_csr();
    c.release_csr();

    hipsparseDestroy(cusparseHandle);
    for (i = 0; i < 2; i++) {
        hipEventDestroy(event[i]);
    }
}

/*Main Function*/
int main(int argc, char *argv[])
{
    CSR<IT, VT> a, b, c;

    /* Set CSR reding from MM file or generating random matrix */
    cout << "Initialize Matrix A" << endl;
    cout << "Read matrix data from " << argv[1] << endl;
    a.init_data_from_mtx(argv[1]);

    cout << "Initialize Matrix B" << endl;
    cout << "Read matrix data from " << argv[1] << endl;
    b.init_data_from_mtx(argv[1]);
  
    /* Execution of SpGEMM on GPU */
    spgemm_cu_csr(a, b, c);
    
    a.release_cpu_csr();
    b.release_cpu_csr();
    c.release_cpu_csr();
  
    return 0;

}

