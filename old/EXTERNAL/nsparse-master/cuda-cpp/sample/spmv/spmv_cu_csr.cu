#include <iostream>
#include <cfloat>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>

#include <CSR.hpp>
#include <nsparse.hpp>

typedef int IT;
#ifdef FLOAT
typedef float VT;
#else
typedef double VT;
#endif

template <class idType, class valType>
void spmv_cu_csr(CSR<idType, valType> &mat, const valType *x, valType *y)
{
    idType i;
    valType *d_x, *d_y;

    hipEvent_t event[2];
    float exe_msec, ave_msec, flops;

    const valType alpha = 1.0;
    const valType beta = 0.0;
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseMatDescr_t descr = 0;

    for (i = 0; i < 2; i++) {
        hipEventCreate(&(event[i]));
    }
  
    /* Malloc and memcpy HtoD */
    mat.memcpyHtD();
  
    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(valType) * mat.ncolumn));
    checkCudaErrors(hipMalloc((void **)&d_y, sizeof(valType) * mat.nrow));
    checkCudaErrors(hipMemcpy(d_x, x, sizeof(valType) * mat.ncolumn, hipMemcpyHostToDevice));

    /* Set up of cuSPARSE */
    hipsparseCreate(&cusparseHandle);
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
  
    /* Execution of SpMV on Device */
    ave_msec = 0;
    for (i = 0; i < TRI_NUM; i++) {
        hipEventRecord(event[0], 0);
#ifdef FLOAT
        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                       mat.nrow, mat.ncolumn, mat.nnz,
                       &alpha, descr, mat.d_values, mat.d_rpt, mat.d_colids,
                       d_x, &beta, d_y);
#else
        hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                       mat.nrow, mat.ncolumn, mat.nnz,
                       &alpha, descr, mat.d_values, mat.d_rpt, mat.d_colids,
                       d_x, &beta, d_y);
#endif
        hipEventRecord(event[1], 0);
    
        hipDeviceSynchronize();
    
        hipEventElapsedTime(&exe_msec, event[0], event[1]);
        if (i > 0) {
            ave_msec += exe_msec;
        }
    }
    ave_msec /= TRI_NUM - 1;
  
    checkCudaErrors(hipMemcpy(y, d_y, sizeof(valType) * mat.nrow, hipMemcpyDeviceToHost));

    flops = (float)(mat.nnz) * 2 / 1000 / 1000 / ave_msec;
    printf("SpMV using CSR format (cuSPARSE): %f[GFLOPS], %f[ms]\n", flops, ave_msec);

    /* Release memory object*/
    hipFree(d_x);
    hipFree(d_y);
    mat.release_csr();
    hipsparseDestroy(cusparseHandle);

}

/*Main Function*/
int main(int argc, char *argv[])
{
    CSR<IT, VT> mat;
    VT *x, *y;

    /* Set CSR reding from MM file or generating random matrix */
    cout << "Read matrix data from " << argv[1] << endl;
    mat.init_data_from_mtx(argv[1]);
  
    /* Init vectors on CPU */
    x = new VT[mat.ncolumn];
    y = new VT[mat.nrow];
    
    init_vector<IT, VT>(x, mat.ncolumn);
    
    /* Execution of SpMV on GPU */
    spmv_cu_csr<IT, VT>(mat, x, y);

#ifdef sfDEBUG
    /* Execution of SpMV on CPU */
    VT *ans_y = new VT[mat.nrow];
    mat.spmv_cpu(x, ans_y);
    check_answer<IT, VT>(ans_y, y, mat.nrow);
    delete[] ans_y;
#endif

    delete[] x;
    delete[] y;
    mat.release_cpu_csr();
  
    return 0;

}

