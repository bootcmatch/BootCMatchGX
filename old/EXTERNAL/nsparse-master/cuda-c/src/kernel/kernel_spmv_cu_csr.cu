#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>

#include <nsparse.h>

void sf_spmv_cu_csr(real *d_y, sfCSR *mat, real *d_x,
                    hipsparseHandle_t *cusparseHandle,
                    hipsparseMatDescr_t *descr) {
  
    const real alpha = 1.0;
    const real beta = 0.0;

#ifdef FLOAT
    hipsparseScsrmv(*cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   mat->M, mat->N, mat->nnz,
                   &alpha, *descr, mat->d_val, mat->d_rpt, mat->d_col,
                   d_x, &beta, d_y);
#else
    hipsparseDcsrmv(*cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   mat->M, mat->N, mat->nnz,
                   &alpha, *descr, mat->d_val, mat->d_rpt, mat->d_col,
                   d_x, &beta, d_y);
#endif

    hipDeviceSynchronize();
      
}

